#include "hip/hip_runtime.h"
/* 
 * vegasp3mgpu.byte.2dtex.cu
 * VEGAS Priority 3 Mode - Stand-Alone GPU Implementation
 *
 * Created by Jayanth Chennamangalam on 2011.06.13
 */

#include "vegasp3mgpu.byte.2dtex.h"

#define BENCHMARKING    0

int g_iIsDone = FALSE;

int g_iMaxThreadsPerBlock = 0;

BYTE *g_pbInBuf = NULL;
BYTE *g_pbInBufRead = NULL;
int g_iReadCount = 0;
int g_iNumReads = 0;

hipArray* g_pcuabData_d;   /* raw data, LEN_DATA * g_iNFFT */
texture<signed char, 2, hipReadModeNormalizedFloat> g_stTexData;
hipChannelFormatDesc g_stChanDescData;

int g_iPFBReadIdx = 0;
int g_iPFBWriteIdx = 0;

int g_iNFFT = DEF_LEN_SPEC;

dim3 g_dimBlockPFB(1, 1, 1);
dim3 g_dimGridPFB(1, 1);
dim3 g_dimBlockCopy(1, 1, 1);
dim3 g_dimGridCopy(1, 1);
dim3 g_dimBlockAccum(1, 1, 1);
dim3 g_dimGridAccum(1, 1);

hipfftComplex *g_pccFFTInX = NULL;
hipfftComplex *g_pccFFTInX_d = NULL;
hipfftComplex *g_pccFFTOutX = NULL;
hipfftComplex *g_pccFFTOutX_d = NULL;
hipfftHandle g_stPlanX = {0};
hipfftComplex *g_pccFFTInY = NULL;
hipfftComplex *g_pccFFTInY_d = NULL;
hipfftComplex *g_pccFFTOutY = NULL;
hipfftComplex *g_pccFFTOutY_d = NULL;
hipfftHandle g_stPlanY = {0};

float *g_pfSumPowX = NULL;
float *g_pfSumPowY = NULL;
float *g_pfSumStokesRe = NULL;
float *g_pfSumStokesIm = NULL;

#if GPUACCUM
float *g_pfSumPowX_d = NULL;
float *g_pfSumPowY_d = NULL;
float *g_pfSumStokesRe_d = NULL;
float *g_pfSumStokesIm_d = NULL;
#endif

int g_iIsPFBOn = DEF_PFB_ON;
int g_iNTaps = 1;                       /* 1 if no PFB, NUM_TAPS if PFB */
int g_iFileCoeff = 0;
char g_acFileCoeff[256] = {0};
signed char (*g_pacPFBCoeff)[][NUM_BYTES_PER_SAMP] = NULL;
hipArray* g_pcuabPFBCoeff_d;
texture<signed char, 2, hipReadModeNormalizedFloat> g_stTexPFBCoeff;
hipChannelFormatDesc g_stChanDescPFBCoeff;


int g_iFileData = 0;
char g_acFileData[256] = {0};

/* PGPLOT global */
float *g_pfFreq = NULL;
float g_fFSamp = 1.0;                   /* 1 [frequency] */

#if BENCHMARKING
    float g_fTimeCpIn = 0.0;
    float g_fAvgCpIn = 0.0;
    float g_fTimeUnpack = 0.0;
    float g_fAvgUnpack = 0.0;
    hipEvent_t g_cuStart;
    hipEvent_t g_cuStop;
    int g_iCount = 0;
#endif

int main(int argc, char *argv[])
{
    int iRet = GUPPI_OK;
#if (!GPUACCUM)
    int i = 0;
#endif
    int iTime = 0;
    int iAcc = DEF_ACC;
#if BENCHMARKING
    float fTimePFB = 0.0;
    float fAvgPFB = 0.0;
    float fTimeCpInFFT = 0.0;
    float fAvgCpInFFT = 0.0;
    float fTimeFFT = 0.0;
    float fAvgFFT = 0.0;
    float fTimeCpOut = 0.0;
    float fAvgCpOut = 0.0;
    float fTimeAccum = 0.0;
    float fAvgAccum = 0.0;
    float fAvgTotal = 0.0;
#else
    struct timeval stStart = {0};
    struct timeval stStop = {0};
#endif
    const char *pcProgName = NULL;
    int iNextOpt = 0;
    /* valid short options */
    const char* const pcOptsShort = "hn:pa:s:";
    /* valid long options */
    const struct option stOptsLong[] = {
        { "help",           0, NULL, 'h' },
        { "nfft",           1, NULL, 'n' },
        { "pfb",            0, NULL, 'p' },
        { "nacc",           1, NULL, 'a' },
        { "fsamp",          1, NULL, 's' },
        { NULL,             0, NULL, 0   }
    };

    /* get the filename of the program from the argument list */
    pcProgName = argv[0];

    /* parse the input */
    do
    {
        iNextOpt = getopt_long(argc, argv, pcOptsShort, stOptsLong, NULL);
        switch (iNextOpt)
        {
            case 'h':   /* -h or --help */
                /* print usage info and terminate */
                PrintUsage(pcProgName);
                return EXIT_SUCCESS;

            case 'n':   /* -n or --nfft */
                /* set option */
                g_iNFFT = (int) atoi(optarg);
                break;

            case 'p':   /* -p or --pfb */
                /* set option */
                g_iIsPFBOn = TRUE;
                break;

            case 'a':   /* -a or --nacc */
                /* set option */
                iAcc = (int) atoi(optarg);
                break;

            case 's':   /* -s or --fsamp */
                /* set option */
                g_fFSamp = (float) atof(optarg);
                break;

            case '?':   /* user specified an invalid option */
                /* print usage info and terminate with error */
                (void) fprintf(stderr, "ERROR: Invalid option!\n");
                PrintUsage(pcProgName);
                return EXIT_FAILURE;

            case -1:    /* done with options */
                break;

            default:    /* unexpected */
                assert(0);
        }
    } while (iNextOpt != -1);

    /* no arguments */
    if (argc <= optind)
    {
        (void) fprintf(stderr, "ERROR: Data file not specified!\n");
        PrintUsage(pcProgName);
        return GUPPI_ERR_GEN;
    }

    (void) strncpy(g_acFileData, argv[optind], 256);
    g_acFileData[255] = '\0';

    /* initialise */
    iRet = Init();
    if (iRet != GUPPI_OK)
    {
        (void) fprintf(stderr, "ERROR! Init failed!\n");
        CleanUp();
        return GUPPI_ERR_GEN;
    }

#if BENCHMARKING
    (void) printf("* Benchmarking run commencing...\n");
    VEGASCUDASafeCall(hipEventCreate(&g_cuStart));
    VEGASCUDASafeCall(hipEventCreate(&g_cuStop));
    (void) printf("* Events created.\n");
#else
    (void) gettimeofday(&stStart, NULL);
#endif
    while (IsRunning())
    {
#if BENCHMARKING
        ++g_iCount;
#endif
        if (g_iIsPFBOn)
        {
            /* do pfb */
#if BENCHMARKING
            VEGASCUDASafeCall(hipEventRecord(g_cuStart, 0));
            VEGASCUDASafeCall(hipEventSynchronize(g_cuStart));
#endif
            DoPFB<<<g_dimGridPFB, g_dimBlockPFB>>>(g_iPFBReadIdx,
                                                   g_iNTaps,
                                                   g_pccFFTInX_d,
                                                   g_pccFFTInY_d);
            VEGASCUDASafeCall(hipDeviceSynchronize());
#if BENCHMARKING
            VEGASCUDASafeCall(hipEventRecord(g_cuStop, 0));
            VEGASCUDASafeCall(hipEventSynchronize(g_cuStop));
            VEGASCUDASafeCall(hipEventElapsedTime(&fTimePFB, g_cuStart, g_cuStop));
            fAvgPFB = (fTimePFB + ((g_iCount - 1) * fAvgPFB)) / g_iCount;
#endif
        }
        else
        {
#if BENCHMARKING
            VEGASCUDASafeCall(hipEventRecord(g_cuStart, 0));
            VEGASCUDASafeCall(hipEventSynchronize(g_cuStart));
#endif
            #if 0
            CopyDataForFFT<<<g_dimGridCopy, g_dimBlockCopy>>>(g_pccFFTInX_d,
                                                              g_pccFFTInY_d);
            #else
            CopyDataForFFT<<<g_dimGridCopy, g_dimBlockCopy>>>(g_pcuabData_d,
                                                              g_pccFFTInX_d,
                                                              g_pccFFTInY_d);
            #endif
            VEGASCUDASafeCall(hipDeviceSynchronize());
#if BENCHMARKING
            VEGASCUDASafeCall(hipEventRecord(g_cuStop, 0));
            VEGASCUDASafeCall(hipEventSynchronize(g_cuStop));
            VEGASCUDASafeCall(hipEventElapsedTime(&fTimeCpInFFT, g_cuStart, g_cuStop));
            fAvgCpInFFT = (fTimeCpInFFT + ((g_iCount - 1) * fAvgCpInFFT)) / g_iCount;
#endif
        }

        /* do fft */
#if BENCHMARKING
        VEGASCUDASafeCall(hipEventRecord(g_cuStart, 0));
        VEGASCUDASafeCall(hipEventSynchronize(g_cuStart));
#endif
        (void) DoFFT();

#if BENCHMARKING
        VEGASCUDASafeCall(hipEventRecord(g_cuStop, 0));
        VEGASCUDASafeCall(hipEventSynchronize(g_cuStop));
        VEGASCUDASafeCall(hipEventElapsedTime(&fTimeFFT, g_cuStart, g_cuStop));
        fAvgFFT = (fTimeFFT + ((g_iCount - 1) * fAvgFFT)) / g_iCount;
#endif

#if (!GPUACCUM)
#if BENCHMARKING
        VEGASCUDASafeCall(hipEventRecord(g_cuStart, 0));
        VEGASCUDASafeCall(hipEventSynchronize(g_cuStart));
#endif
        VEGASCUDASafeCall(hipMemcpy(g_pccFFTOutX,
                   g_pccFFTOutX_d,
                   g_iNFFT * sizeof(hipfftComplex),
                   hipMemcpyDeviceToHost));
        VEGASCUDASafeCall(hipMemcpy(g_pccFFTOutY,
                   g_pccFFTOutY_d,
                   g_iNFFT * sizeof(hipfftComplex),
                   hipMemcpyDeviceToHost));
#if BENCHMARKING
        VEGASCUDASafeCall(hipEventRecord(g_cuStop, 0));
        VEGASCUDASafeCall(hipEventSynchronize(g_cuStop));
        VEGASCUDASafeCall(hipEventElapsedTime(&fTimeCpOut, g_cuStart, g_cuStop));
        fAvgCpOut = (fTimeCpOut + ((g_iCount - 1) * fAvgCpOut)) / g_iCount;
#endif
#endif

        /* accumulate power x, power y, stokes, if the blanking bit is
           not set */
        if (!IsBlankingSet())
        {
            if (0/* blanking to non-blanking */)
            {
                /* TODO: when blanking is unset, start accumulating */
                /* reset time */
                iTime = 0;
                /* zero accumulators */
                (void) memset(g_pfSumPowX, '\0', g_iNFFT * sizeof(float));
                (void) memset(g_pfSumPowY, '\0', g_iNFFT * sizeof(float));
                (void) memset(g_pfSumStokesRe, '\0', g_iNFFT * sizeof(float));
                (void) memset(g_pfSumStokesIm, '\0', g_iNFFT * sizeof(float));
            }
            else
            {
#if BENCHMARKING
                VEGASCUDASafeCall(hipEventRecord(g_cuStart, 0));
                VEGASCUDASafeCall(hipEventSynchronize(g_cuStart));
#endif
                #if GPUACCUM
                Accumulate<<<g_dimGridAccum, g_dimBlockAccum>>>(g_pccFFTOutX_d,
                                                                g_pccFFTOutY_d,
                                                                g_pfSumPowX_d,
                                                                g_pfSumPowY_d,
                                                                g_pfSumStokesRe_d,
                                                                g_pfSumStokesIm_d);
                VEGASCUDASafeCall(hipDeviceSynchronize());
                #else
                for (i = 0; i < g_iNFFT; ++i)
                {
                    /* Re(X)^2 + Im(X)^2 */
                    g_pfSumPowX[i] += (g_pccFFTOutX[i].x * g_pccFFTOutX[i].x)
                                      + (g_pccFFTOutX[i].y * g_pccFFTOutX[i].y);
                    /* Re(Y)^2 + Im(Y)^2 */
                    g_pfSumPowY[i] += (g_pccFFTOutY[i].x * g_pccFFTOutY[i].x)
                                      + (g_pccFFTOutY[i].y * g_pccFFTOutY[i].y);
                    /* Re(XY*) */
                    g_pfSumStokesRe[i] += (g_pccFFTOutX[i].x * g_pccFFTOutY[i].x)
                                          + (g_pccFFTOutX[i].y * g_pccFFTOutY[i].y);
                    /* Im(XY*) */
                    g_pfSumStokesIm[i] += (g_pccFFTOutX[i].y * g_pccFFTOutY[i].x)
                                          - (g_pccFFTOutX[i].x * g_pccFFTOutY[i].y);
                }
                #endif
#if BENCHMARKING
                VEGASCUDASafeCall(hipEventRecord(g_cuStop, 0));
                VEGASCUDASafeCall(hipEventSynchronize(g_cuStop));
                VEGASCUDASafeCall(hipEventElapsedTime(&fTimeAccum, g_cuStart, g_cuStop));
                fAvgAccum = (fTimeAccum + ((g_iCount - 1) * fAvgAccum)) / g_iCount;
#endif
                ++iTime;
                if (iTime == iAcc)
                {
                    #if PLOT
                    /* NOTE: Plot() will modify data! */
                    Plot();
                    usleep(500000);
                    #endif

                    /* dump to buffer */
                    #if GPUACCUM
#if BENCHMARKING
                    VEGASCUDASafeCall(hipEventRecord(g_cuStart, 0));
                    VEGASCUDASafeCall(hipEventSynchronize(g_cuStart));
#endif
                    VEGASCUDASafeCall(hipMemcpy(g_pfSumPowX,
                                                 g_pfSumPowX_d,
                                                 g_iNFFT * sizeof(float),
                                                 hipMemcpyDeviceToHost));
                    VEGASCUDASafeCall(hipMemcpy(g_pfSumPowY,
                                                 g_pfSumPowY_d,
                                                 g_iNFFT * sizeof(float),
                                                 hipMemcpyDeviceToHost));
                    VEGASCUDASafeCall(hipMemcpy(g_pfSumStokesRe,
                                                 g_pfSumStokesRe_d,
                                                 g_iNFFT * sizeof(float),
                                                 hipMemcpyDeviceToHost));
                    VEGASCUDASafeCall(hipMemcpy(g_pfSumStokesIm,
                                                 g_pfSumStokesIm_d,
                                                 g_iNFFT * sizeof(float),
                                                 hipMemcpyDeviceToHost));
#if BENCHMARKING
                    VEGASCUDASafeCall(hipEventRecord(g_cuStop, 0));
                    VEGASCUDASafeCall(hipEventSynchronize(g_cuStop));
                    VEGASCUDASafeCall(hipEventElapsedTime(&fTimeCpOut, g_cuStart, g_cuStop));
                    fAvgCpOut = (fTimeCpOut + ((g_iCount - 1) * fAvgCpOut)) / g_iCount;
#endif
                    #endif

                    /* reset time */
                    iTime = 0;
                    /* zero accumulators */
                    #if GPUACCUM
                    VEGASCUDASafeCall(hipMemset(g_pfSumPowX_d, '\0', g_iNFFT * sizeof(float)));
                    VEGASCUDASafeCall(hipMemset(g_pfSumPowY_d, '\0', g_iNFFT * sizeof(float)));
                    VEGASCUDASafeCall(hipMemset(g_pfSumStokesRe_d, '\0', g_iNFFT * sizeof(float)));
                    VEGASCUDASafeCall(hipMemset(g_pfSumStokesIm_d, '\0', g_iNFFT * sizeof(float)));
                    #else
                    (void) memset(g_pfSumPowX, '\0', g_iNFFT * sizeof(float));
                    (void) memset(g_pfSumPowY, '\0', g_iNFFT * sizeof(float));
                    (void) memset(g_pfSumStokesRe, '\0', g_iNFFT * sizeof(float));
                    (void) memset(g_pfSumStokesIm, '\0', g_iNFFT * sizeof(float));
                    #endif
                }
            }
        }
        else
        {
            /* TODO: */
            if (1/* non-blanking to blanking */)
            {
                /* write status, dump data to disk buffer */
            }
            else
            {
                /* do nothing, wait for blanking to stop */
            }
        }

        /* read data from input buffer, convert 8_7 to float */
        iRet = ReadData();
        if (iRet != GUPPI_OK)
        {
            (void) fprintf(stderr, "ERROR: Data reading failed!\n");
            break;
        }
    }
#if (!BENCHMARKING)
    (void) gettimeofday(&stStop, NULL);
    (void) printf("Time taken (barring Init()): %gs\n",
                  ((stStop.tv_sec + (stStop.tv_usec * USEC2SEC))
                   - (stStart.tv_sec + (stStart.tv_usec * USEC2SEC))));
#endif

    CleanUp();

#if BENCHMARKING
    fAvgTotal = g_fAvgCpIn + g_fAvgUnpack + fAvgPFB + fAvgCpInFFT + fAvgFFT + fAvgAccum + fAvgCpOut;
    (void) printf("    Average elapsed time for %d\n", g_iCount);
    (void) printf("        calls to hipMemcpy(Host2Device)          : %5.3fms, %2d%%\n",
                  g_fAvgCpIn,
                  (int) ((g_fAvgCpIn / fAvgTotal) * 100));
    (void) printf("        calls to Unpack()                         : %5.3fms, %2d%%\n",
                  g_fAvgUnpack,
                  (int) ((g_fAvgUnpack / fAvgTotal) * 100));
    if (g_iIsPFBOn)
    {
        (void) printf("        calls to DoPFB()                          : %5.3fms, %2d%%\n",
                      fAvgPFB,
                      (int) ((fAvgPFB / fAvgTotal) * 100));
    }
    else
    {
        (void) printf("        calls to CopyDataForFFT()                 : %5.3fms, %2d%%\n",
                      fAvgCpInFFT,
                      (int) ((fAvgCpInFFT / fAvgTotal) * 100));
    }
    (void) printf("        calls to DoFFT()                          : %5.3fms, %2d%%\n",
                  fAvgFFT,
                  (int) ((fAvgFFT / fAvgTotal) * 100));
    (void) printf("        calls to Accumulate()/accumulation loop   : %5.3fms, %2d%%\n",
                  fAvgAccum,
                  (int) ((fAvgAccum / fAvgTotal) * 100));
#if GPUACCUM
    (void) printf("        x4 calls to hipMemcpy(Device2Host)       : %5.3fms, %2d%%\n",
                  fAvgCpOut,
                  (int) ((fAvgCpOut / fAvgTotal) * 100));
#else
    (void) printf("        x2 calls to hipMemcpy(Device2Host)       : %5.3fms, %2d%%\n",
                  fAvgCpOut,
                  (int) ((fAvgCpOut / fAvgTotal) * 100));
#endif
    VEGASCUDASafeCall(hipEventDestroy(g_cuStart));
    VEGASCUDASafeCall(hipEventDestroy(g_cuStop));
    (void) printf("* Events destroyed.\n");
    (void) printf("* Benchmarking run completed.\n");
#endif

    return GUPPI_OK;
}

/* function that creates the FFT plan, allocates memory, initialises counters,
   etc. */
int Init()
{
    int i = 0;
    int j = 0;
    int iDevCount = 0;
    hipDeviceProp_t stDevProp = {0};
    int iRet = GUPPI_OK;

    iRet = RegisterSignalHandlers();
    if (iRet != GUPPI_OK)
    {
        (void) fprintf(stderr, "ERROR: Signal-handler registration failed!\n");
        return GUPPI_ERR_GEN;
    }

    VEGASCUDASafeCall(hipGetDeviceCount(&iDevCount));
    if (0 == iDevCount)
    {
        (void) fprintf(stderr, "ERROR: No CUDA-capable device found!\n");
        return EXIT_FAILURE;
    }
    else if (iDevCount > 1)
    {
        /* TODO: figure this out */
        (void) fprintf(stderr,
                       "ERROR: More than one CUDA-capable device "
                       "found! Don't know how to proceed!\n");
        return EXIT_FAILURE;
    }

    /* TODO: make it automagic */
    VEGASCUDASafeCall(hipSetDevice(0));

    VEGASCUDASafeCall(hipGetDeviceProperties(&stDevProp, 0));
    g_iMaxThreadsPerBlock = stDevProp.maxThreadsPerBlock;

    if (g_iIsPFBOn)
    {
        /* set number of taps to NUM_TAPS if PFB is on, else number of
           taps = 1 */
        g_iNTaps = NUM_TAPS;

        g_pacPFBCoeff = (signed char(*) [][NUM_BYTES_PER_SAMP]) malloc(NUM_BYTES_PER_SAMP * g_iNTaps * g_iNFFT * sizeof(signed char));
        if (NULL == g_pacPFBCoeff)
        {
            (void) fprintf(stderr,
                           "ERROR: Memory allocation failed! %s.\n",
                           strerror(errno));
            return GUPPI_ERR_GEN;
        }

        /* allocate memory for the filter coefficient array on the device */
        g_stChanDescPFBCoeff = hipCreateChannelDesc<signed char>();
        VEGASCUDASafeCall(hipMallocArray(&g_pcuabPFBCoeff_d,
                                          &g_stChanDescPFBCoeff,
                                          NUM_BYTES_PER_SAMP,
                                          (g_iNTaps * g_iNFFT * sizeof(signed char))));

        /* read filter coefficients */
        /* build file name */
        (void) sprintf(g_acFileCoeff,
                       "%s%d_%d%s",
                       FILE_COEFF_PREFIX,
                       g_iNTaps,
                       g_iNFFT,
                       FILE_COEFF_SUFFIX);
        g_iFileCoeff = open(g_acFileCoeff, O_RDONLY);
        if (GUPPI_ERR_GEN == g_iFileCoeff)
        {
            (void) fprintf(stderr,
                           "ERROR: Opening filter coefficients file %s failed! %s.\n",
                           g_acFileCoeff,
                           strerror(errno));
            return GUPPI_ERR_GEN;
        }

        for (i = 0; i < (g_iNTaps * g_iNFFT); ++i)
        {
            for (j = 0; j < NUM_BYTES_PER_SAMP; ++j)
            {
                iRet = read(g_iFileCoeff,
                            &((*g_pacPFBCoeff)[i][j]),
                            sizeof(signed char));
                if (GUPPI_ERR_GEN == iRet)
                {
                    (void) fprintf(stderr,
                                   "ERROR: Reading filter coefficients failed! %s.\n",
                                   strerror(errno));
                    return GUPPI_ERR_GEN;
                }
            }
        }
        (void) close(g_iFileCoeff);

        /* copy filter coefficients to the device */
        VEGASCUDASafeCall(hipMemcpy2DToArray(g_pcuabPFBCoeff_d,
                                              0,
                                              0,
                                              g_pacPFBCoeff,
                                              NUM_BYTES_PER_SAMP * sizeof(signed char),
                                              NUM_BYTES_PER_SAMP * sizeof(signed char),
                                              g_iNTaps * g_iNFFT,
                                              hipMemcpyHostToDevice));
    }

    /* allocate memory for data array contents */
    g_stChanDescData = hipCreateChannelDesc<signed char>();
    VEGASCUDASafeCall(hipMallocArray(&g_pcuabData_d,
                                      &g_stChanDescData,
                                      NUM_BYTES_PER_SAMP,
                                      (g_iNTaps * g_iNFFT * sizeof(BYTE))));

    /* temporarily read a file, instead of input buffer */
    g_iFileData = open(g_acFileData, O_RDONLY);
    if (GUPPI_ERR_GEN == g_iFileData)
    {
        (void) fprintf(stderr,
                       "ERROR! Opening data file %s failed! %s.\n",
                       g_acFileData,
                       strerror(errno));
        return GUPPI_ERR_GEN;
    }

    /* load data into memory */
    iRet = LoadData();
    if (iRet != GUPPI_OK)
    {
        (void) fprintf(stderr,
                       "ERROR! Data loading failed!\n");
        return GUPPI_ERR_GEN;
    }

    /* calculate kernel parameters */
    if (g_iNFFT < g_iMaxThreadsPerBlock)
    {
        g_dimBlockPFB.x = g_iNFFT;
        #if GPUACCUM
        g_dimBlockAccum.x = g_iNFFT;
        #endif
    }
    else
    {
        g_dimBlockPFB.x = g_iMaxThreadsPerBlock;
        #if GPUACCUM
        g_dimBlockAccum.x = g_iMaxThreadsPerBlock;
        #endif
    }
    g_dimGridPFB.x = (int) ceilf(((float) g_iNFFT) / g_iMaxThreadsPerBlock);

    g_dimBlockCopy.x = NUM_BYTES_PER_SAMP;
    g_dimBlockCopy.y = (int) (((float) g_iMaxThreadsPerBlock) / NUM_BYTES_PER_SAMP);
    g_dimBlockCopy.z = 1;
    g_dimGridCopy.x = (int) (((float) g_iNFFT) / g_dimBlockCopy.y);
    g_dimGridCopy.y = 1;
    #if GPUACCUM
    g_dimGridAccum.x = (int) ceilf(((float) g_iNFFT) / g_iMaxThreadsPerBlock);
    #endif
    g_pbInBufRead = g_pbInBuf;
    VEGASCUDASafeCall(hipMemcpy2DToArray(g_pcuabData_d,
                                          0,
                                          0,
                                          g_pbInBufRead,
                                          NUM_BYTES_PER_SAMP * sizeof(BYTE),
                                          NUM_BYTES_PER_SAMP * sizeof(BYTE),
                                          g_iNTaps * g_iNFFT,
                                          hipMemcpyHostToDevice));
    g_pbInBufRead += g_iNTaps * LEN_DATA;
    g_iReadCount += g_iNTaps;
    if (g_iReadCount == g_iNumReads)
    {
        (void) printf("Data read done!\n");
        g_iIsDone = TRUE;
    }

    /* bind texture to memory */
    VEGASCUDASafeCall(hipBindTextureToArray(&g_stTexData,
                                             g_pcuabData_d,
                                             &g_stChanDescData));
    if (g_iIsPFBOn)
    {
        VEGASCUDASafeCall(hipBindTextureToArray(&g_stTexPFBCoeff,
                                                 g_pcuabPFBCoeff_d,
                                                 &g_stChanDescPFBCoeff));
    }

    g_iPFBWriteIdx = 0;     /* next write into the first buffer */
    g_iPFBReadIdx = 0;      /* PFB to be performed from first buffer */

    g_pccFFTInX = (hipfftComplex *) malloc(g_iNFFT * sizeof(hipfftComplex));
    if (NULL == g_pccFFTInX)
    {
        (void) fprintf(stderr,
                       "ERROR: Memory allocation failed! %s.\n",
                       strerror(errno));
        return GUPPI_ERR_GEN;
    }
    VEGASCUDASafeCall(hipMalloc((void **) &g_pccFFTInX_d,
                          g_iNFFT * sizeof(hipfftComplex)));
    g_pccFFTInY = (hipfftComplex *) malloc(g_iNFFT * sizeof(hipfftComplex));
    if (NULL == g_pccFFTInY)
    {
        (void) fprintf(stderr,
                       "ERROR: Memory allocation failed! %s.\n",
                       strerror(errno));
        return GUPPI_ERR_GEN;
    }
    VEGASCUDASafeCall(hipMalloc((void **) &g_pccFFTInY_d,
                      g_iNFFT * sizeof(hipfftComplex)));
    g_pccFFTOutX = (hipfftComplex *) malloc(g_iNFFT * sizeof(hipfftComplex));
    if (NULL == g_pccFFTOutX)
    {
        (void) fprintf(stderr,
                       "ERROR: Memory allocation failed! %s.\n",
                       strerror(errno));
        return GUPPI_ERR_GEN;
    }
    VEGASCUDASafeCall(hipMalloc((void **) &g_pccFFTOutX_d,
                      g_iNFFT * sizeof(hipfftComplex)));
    g_pccFFTOutY = (hipfftComplex *) malloc(g_iNFFT * sizeof(hipfftComplex));
    if (NULL == g_pccFFTOutY)
    {
        (void) fprintf(stderr,
                       "ERROR: Memory allocation failed! %s.\n",
                       strerror(errno));
        return GUPPI_ERR_GEN;
    }
    VEGASCUDASafeCall(hipMalloc((void **) &g_pccFFTOutY_d,
                      g_iNFFT * sizeof(hipfftComplex)));

    g_pfSumPowX = (float *) calloc(g_iNFFT, sizeof(float));
    if (NULL == g_pfSumPowX)
    {
        (void) fprintf(stderr,
                       "ERROR: Memory allocation failed! %s.\n",
                       strerror(errno));
        return GUPPI_ERR_GEN;
    }
    g_pfSumPowY = (float *) calloc(g_iNFFT, sizeof(float));
    if (NULL == g_pfSumPowY)
    {
        (void) fprintf(stderr,
                       "ERROR: Memory allocation failed! %s.\n",
                       strerror(errno));
        return GUPPI_ERR_GEN;
    }
    g_pfSumStokesRe = (float *) calloc(g_iNFFT, sizeof(float));
    if (NULL == g_pfSumStokesRe)
    {
        (void) fprintf(stderr,
                       "ERROR: Memory allocation failed! %s.\n",
                       strerror(errno));
        return GUPPI_ERR_GEN;
    }
    g_pfSumStokesIm = (float *) calloc(g_iNFFT, sizeof(float));
    if (NULL == g_pfSumStokesIm)
    {
        (void) fprintf(stderr,
                       "ERROR: Memory allocation failed! %s.\n",
                       strerror(errno));
        return GUPPI_ERR_GEN;
    }
#if GPUACCUM
    VEGASCUDASafeCall(hipMalloc((void **) &g_pfSumPowX_d, g_iNFFT * sizeof(float)));
    VEGASCUDASafeCall(hipMemset(g_pfSumPowX_d, '\0', g_iNFFT * sizeof(float)));
    VEGASCUDASafeCall(hipMalloc((void **) &g_pfSumPowY_d, g_iNFFT * sizeof(float)));
    VEGASCUDASafeCall(hipMemset(g_pfSumPowY_d, '\0', g_iNFFT * sizeof(float)));
    VEGASCUDASafeCall(hipMalloc((void **) &g_pfSumStokesRe_d, g_iNFFT * sizeof(float)));
    VEGASCUDASafeCall(hipMemset(g_pfSumStokesRe_d, '\0', g_iNFFT * sizeof(float)));
    VEGASCUDASafeCall(hipMalloc((void **) &g_pfSumStokesIm_d, g_iNFFT * sizeof(float)));
    VEGASCUDASafeCall(hipMemset(g_pfSumStokesIm_d, '\0', g_iNFFT * sizeof(float)));
#endif

    /* create plans */
    (void) hipfftPlan1d(&g_stPlanX, g_iNFFT, HIPFFT_C2C, 1);
    (void) hipfftPlan1d(&g_stPlanY, g_iNFFT, HIPFFT_C2C, 1);

#if PLOT
    /* just for plotting */
    InitPlot();
#endif

    return GUPPI_OK;
}

/* function that reads data from the data file and loads it into memory during
   initialisation */
int LoadData()
{
    struct stat stFileStats = {0};
    int iRet = GUPPI_OK;

    iRet = stat(g_acFileData, &stFileStats);
    if (iRet != GUPPI_OK)
    {
        (void) fprintf(stderr,
                       "ERROR: Failed to stat %s: %s!\n",
                       g_acFileData,
                       strerror(errno));
        return GUPPI_ERR_GEN;
    }

    g_pbInBuf = (BYTE *) malloc(stFileStats.st_size * sizeof(BYTE));
    if (NULL == g_pbInBuf)
    {
        (void) fprintf(stderr,
                       "ERROR: Memory allocation failed! %s.\n",
                       strerror(errno));
        return GUPPI_ERR_GEN;
    }

    iRet = read(g_iFileData, g_pbInBuf, stFileStats.st_size);
    if (GUPPI_ERR_GEN == iRet)
    {
        (void) fprintf(stderr,
                       "ERROR: Data reading failed! %s.\n",
                       strerror(errno));
        return GUPPI_ERR_GEN;
    }
    else if (iRet != stFileStats.st_size)
    {
        (void) printf("File read done!\n");
    }

    /* calculate the number of reads required */
    g_iNumReads = stFileStats.st_size / LEN_DATA;

    return GUPPI_OK;
}

/* function that reads data from input buffer */
int ReadData()
{
    /* write new data to the write buffer */
#if BENCHMARKING
    VEGASCUDASafeCall(hipEventRecord(g_cuStart, 0));
    VEGASCUDASafeCall(hipEventSynchronize(g_cuStart));
#endif
    VEGASCUDASafeCall(hipMemcpy2DToArray(g_pcuabData_d,
                                          0,
                                          g_iPFBWriteIdx * g_iNFFT,
                                          g_pbInBufRead,
                                          NUM_BYTES_PER_SAMP * sizeof(BYTE),
                                          NUM_BYTES_PER_SAMP * sizeof(BYTE),
                                          g_iNFFT,
                                          hipMemcpyHostToDevice));
#if BENCHMARKING
    VEGASCUDASafeCall(hipEventRecord(g_cuStop, 0));
    VEGASCUDASafeCall(hipEventSynchronize(g_cuStop));
    VEGASCUDASafeCall(hipEventElapsedTime(&g_fTimeCpIn, g_cuStart, g_cuStop));
    g_fAvgCpIn = (g_fTimeCpIn + ((g_iCount - 1) * g_fAvgCpIn)) / g_iCount;
#endif
    g_pbInBufRead += LEN_DATA;
    ++g_iReadCount;
    if (g_iReadCount == g_iNumReads)
    {
        (void) printf("Data read done!\n");
        g_iIsDone = TRUE;
    }

    if (g_iPFBWriteIdx != (g_iNTaps - 1))
    {
        ++g_iPFBWriteIdx;
    }
    else
    {
        g_iPFBWriteIdx = 0;
    }
    if (g_iPFBReadIdx != (g_iNTaps - 1))
    {
        ++g_iPFBReadIdx;
    }
    else
    {
        g_iPFBReadIdx = 0;
    }

    return GUPPI_OK;
}

/* function that performs the PFB */
__global__ void DoPFB(int iPFBReadIdx,
                      int iNTaps,
                      hipfftComplex *pccFFTInX,
                      hipfftComplex *pccFFTInY)
{
#if 0
    int i = iPFBReadIdx;
    int j = 0;
    int k = (blockIdx.x * blockDim.x) + threadIdx.x;
    int iNFFT = gridDim.x * blockDim.x;
    hipfftComplex ccAccumX;
    hipfftComplex ccAccumY;
    signed char (*pabData)[][NUM_BYTES_PER_SAMP] = (signed char(*) [][NUM_BYTES_PER_SAMP]) pbData;

    ccAccumX.x = 0.0;
    ccAccumX.y = 0.0;
    ccAccumY.x = 0.0;
    ccAccumY.y = 0.0;

    for (j = 0; j < iNTaps; ++j)
    {
        ccAccumX.x += (*pabData)[(i * iNFFT) + k][0] * tex2D(g_stTexPFBCoeff, 0, (j * iNFFT) + k);
        ccAccumX.y += (*pabData)[(i * iNFFT) + k][1] * tex2D(g_stTexPFBCoeff, 1, (j * iNFFT) + k);
        ccAccumY.x += (*pabData)[(i * iNFFT) + k][2] * tex2D(g_stTexPFBCoeff, 2, (j * iNFFT) + k);
        ccAccumY.y += (*pabData)[(i * iNFFT) + k][3] * tex2D(g_stTexPFBCoeff, 3, (j * iNFFT) + k);
        if (i != (iNTaps - 1))
        {
            ++i;
        }
        else
        {
            i = 0;
        }
    }

    pccFFTInX[k] = ccAccumX;
    pccFFTInY[k] = ccAccumY;
#endif
    return;
}

#if 0
__global__ void DoPFB(signed char *pbDataX,
                      signed char *pbDataY,
                      int iPFBReadIdx,
                      int iNTaps,
                      signed char *pcPFBCoeff,
                      hipfftComplex *pccFFTInX,
                      hipfftComplex *pccFFTInY)
{
    int i = iPFBReadIdx;
    int j = 0;
    int k = (blockIdx.x * blockDim.x) + threadIdx.x;
    int iNFFT = gridDim.x * blockDim.x;
    hipfftComplex ccAccumX;
    hipfftComplex ccAccumY;
    signed char (*pabDataX)[][2] = (signed char(*) [][2]) pbDataX;
    signed char (*pabDataY)[][2] = (signed char(*) [][2]) pbDataY;

    ccAccumX.x = 0.0;
    ccAccumX.y = 0.0;
    ccAccumY.x = 0.0;
    ccAccumY.y = 0.0;

    for (j = 0; j < iNTaps; ++j)
    {
        ccAccumX.x += (*pabDataX)[(i * iNFFT) + k][0] * pcPFBCoeff[(j * iNFFT) + k];
        ccAccumX.y += (*pabDataX)[(i * iNFFT) + k][1] * pcPFBCoeff[(j * iNFFT) + k];
        ccAccumY.x += (*pabDataY)[(i * iNFFT) + k][0] * pcPFBCoeff[(j * iNFFT) + k];
        ccAccumY.y += (*pabDataY)[(i * iNFFT) + k][1] * pcPFBCoeff[(j * iNFFT) + k];
        if (i != (iNTaps - 1))
        {
            ++i;
        }
        else
        {
            i = 0;
        }
    }

    pccFFTInX[k] = ccAccumX;
    pccFFTInY[k] = ccAccumY;

    return;
}
#endif

#if 0
__global__ void CopyDataForFFT(hipfftComplex *pccFFTInX,
                               hipfftComplex *pccFFTInY)
{
    int i = (blockIdx.x * blockDim.y) + threadIdx.y;
    int x = threadIdx.x;
    float f = 0.0;

    f = tex2D(g_stTexData, x, i);
    switch (x)
    {
        case 0: pccFFTInX[i].x = f; break;
        case 1: pccFFTInX[i].y = f; break;
        case 2: pccFFTInY[i].x = f; break;
        case 3: pccFFTInY[i].y = f; break;
    }

    return;
}
#else
__global__ void CopyDataForFFT(hipArray *pcuabData,
                               hipfftComplex *pccFFTInX,
                               hipfftComplex *pccFFTInY)
{
    int i = (blockIdx.x * blockDim.y) + threadIdx.y;
    int x = threadIdx.x;
    signed char (*pabData)[][4] = (signed char(*) [][4]) pcuabData;
    __shared__ float afTile[128][4];

    afTile[i][x] = (*pabData)[i][x];

    __syncthreads();

    switch (x)
    {
        case 0: pccFFTInX[i].x = afTile[i][x]; break;
        case 1: pccFFTInX[i].y = afTile[i][x]; break;
        case 2: pccFFTInY[i].x = afTile[i][x]; break;
        case 3: pccFFTInY[i].y = afTile[i][x]; break;
    }

    return;
}
#endif

/* function that performs the FFT */
int DoFFT()
{
    /* execute plan */
    (void) hipfftExecC2C(g_stPlanX, g_pccFFTInX_d, g_pccFFTOutX_d, HIPFFT_FORWARD);
    (void) hipfftExecC2C(g_stPlanY, g_pccFFTInY_d, g_pccFFTOutY_d, HIPFFT_FORWARD);

    return GUPPI_OK;
}

#if GPUACCUM
__global__ void Accumulate(hipfftComplex *pccFFTOutX,
                           hipfftComplex *pccFFTOutY,
                           float *pfSumPowX,
                           float *pfSumPowY,
                           float *pfSumStokesRe,
                           float *pfSumStokesIm)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    /* Re(X)^2 + Im(X)^2 */
    pfSumPowX[i] += (pccFFTOutX[i].x * pccFFTOutX[i].x)
                    + (pccFFTOutX[i].y * pccFFTOutX[i].y);
    /* Re(Y)^2 + Im(Y)^2 */
    pfSumPowY[i] += (pccFFTOutY[i].x * pccFFTOutY[i].x)
                    + (pccFFTOutY[i].y * pccFFTOutY[i].y);
    /* Re(XY*) */
    pfSumStokesRe[i] += (pccFFTOutX[i].x * pccFFTOutY[i].x)
                        + (pccFFTOutX[i].y * pccFFTOutY[i].y);
    /* Im(XY*) */
    pfSumStokesIm[i] += (pccFFTOutX[i].y * pccFFTOutY[i].x)
                        - (pccFFTOutX[i].x * pccFFTOutY[i].y);

    return;
}
#endif

int IsRunning()
{
    return (!g_iIsDone);
}

int IsBlankingSet()
{
    /* check for status and return TRUE or FALSE */
    return FALSE;
}

/* function that frees resources */
void CleanUp()
{
    /* free resources */
    free(g_pbInBuf);

    (void) hipFreeArray(g_pcuabData_d);
    free(g_pccFFTInX);
    (void) hipFree(g_pccFFTInX_d);
    free(g_pccFFTInY);
    (void) hipFree(g_pccFFTInY_d);
    free(g_pccFFTOutX);
    (void) hipFree(g_pccFFTOutX_d);
    free(g_pccFFTOutY);
    (void) hipFree(g_pccFFTOutY_d);

    free(g_pacPFBCoeff);
    (void) hipFreeArray(g_pcuabPFBCoeff_d);

    free(g_pfSumPowX);
    free(g_pfSumPowY);
    free(g_pfSumStokesRe);
    free(g_pfSumStokesIm);

    /* destroy plans */
    (void) hipfftDestroy(g_stPlanX);
    (void) hipfftDestroy(g_stPlanY);

    (void) close(g_iFileData);

#if PLOT
    /* for plotting */
    free(g_pfFreq);
    cpgclos();
#endif

    return;
}

#if PLOT
void InitPlot()
{
    int iRet = GUPPI_OK;
    int i = 0;

    iRet = cpgopen(PG_DEV);
    if (iRet <= 0)
    {
        (void) fprintf(stderr,
                       "ERROR: Opening graphics device %s failed!\n",
                       PG_DEV);
        return;
    }

    cpgsch(2);
    cpgsubp(1, 4);

    g_pfFreq = (float *) malloc(g_iNFFT * sizeof(float));
    if (NULL == g_pfFreq)
    {
        (void) fprintf(stderr,
                       "ERROR: Memory allocation failed! %s.\n",
                       strerror(errno));
        return;
    }

    /* load the frequency axis */
    for (i = 0; i < g_iNFFT; ++i)
    {
        g_pfFreq[i] = ((float) i * g_fFSamp) / g_iNFFT;
    }

    return;
}

void Plot()
{
    float fMinFreq = g_pfFreq[0];
    float fMaxFreq = g_pfFreq[g_iNFFT-1];
    float fMinY = FLT_MAX;
    float fMaxY = -(FLT_MAX);
    int i = 0;

    /* take log10 of data */
    for (i = 0; i < g_iNFFT; ++i)
    {
        g_pfSumPowX[i] = 10 * log10f(g_pfSumPowX[i]);
        g_pfSumPowY[i] = 10 * log10f(g_pfSumPowY[i]);
        g_pfSumStokesRe[i] = log10f(g_pfSumStokesRe[i]);
        g_pfSumStokesIm[i] = log10f(g_pfSumStokesIm[i]);
    }

    /* plot g_pfSumPowX */
    for (i = 0; i < g_iNFFT; ++i)
    {
        if (g_pfSumPowX[i] > fMaxY)
        {
            fMaxY = g_pfSumPowX[i];
        }
        if (g_pfSumPowX[i] < fMinY)
        {
            fMinY = g_pfSumPowX[i];
        }
    }
    cpgpanl(1, 1);
    cpgeras();
    cpgsvp(PG_VP_ML, PG_VP_MR, PG_VP_MB, PG_VP_MT);
    cpgswin(fMinFreq, fMaxFreq, fMinY, fMaxY);
    cpglab("Bin Number",
           "",
           "SumPowX");
    cpgbox("BCNST", 0.0, 0, "BCNST", 0.0, 0);
    cpgsci(PG_CI_PLOT);
    cpgline(g_iNFFT, g_pfFreq, g_pfSumPowX);
    cpgsci(PG_CI_DEF);

    /* plot g_pfSumPowY */
    fMinY = FLT_MAX;
    fMaxY = -(FLT_MAX);
    for (i = 0; i < g_iNFFT; ++i)
    {
        if (g_pfSumPowY[i] > fMaxY)
        {
            fMaxY = g_pfSumPowY[i];
        }
        if (g_pfSumPowY[i] < fMinY)
        {
            fMinY = g_pfSumPowY[i];
        }
    }
    for (i = 0; i < g_iNFFT; ++i)
    {
        g_pfSumPowY[i] -= fMaxY;
        //printf("%g\n", g_pfSumPowY[i]);
    }
    fMinY -= fMaxY;
    fMaxY = 0;
    //printf("********************************\n");
    cpgpanl(1, 2);
    cpgeras();
    cpgsvp(PG_VP_ML, PG_VP_MR, PG_VP_MB, PG_VP_MT);
    cpgswin(fMinFreq, fMaxFreq, fMinY, fMaxY);
    cpglab("Bin Number",
           "",
           "SumPowY");
    cpgbox("BCNST", 0.0, 0, "BCNST", 0.0, 0);
    cpgsci(PG_CI_PLOT);
    cpgline(g_iNFFT, g_pfFreq, g_pfSumPowY);
    cpgsci(PG_CI_DEF);

    /* plot g_pfSumStokesRe */
    fMinY = FLT_MAX;
    fMaxY = -(FLT_MAX);
    for (i = 0; i < g_iNFFT; ++i)
    {
        if (g_pfSumStokesRe[i] > fMaxY)
        {
            fMaxY = g_pfSumStokesRe[i];
        }
        if (g_pfSumStokesRe[i] < fMinY)
        {
            fMinY = g_pfSumStokesRe[i];
        }
    }
    cpgpanl(1, 3);
    cpgeras();
    cpgsvp(PG_VP_ML, PG_VP_MR, PG_VP_MB, PG_VP_MT);
    cpgswin(fMinFreq, fMaxFreq, fMinY, fMaxY);
    cpglab("Bin Number",
           "",
           "SumStokesRe");
    cpgbox("BCNST", 0.0, 0, "BCNST", 0.0, 0);
    cpgsci(PG_CI_PLOT);
    cpgline(g_iNFFT, g_pfFreq, g_pfSumStokesRe);
    cpgsci(PG_CI_DEF);

    /* plot g_pfSumStokesIm */
    fMinY = FLT_MAX;
    fMaxY = -(FLT_MAX);
    for (i = 0; i < g_iNFFT; ++i)
    {
        if (g_pfSumStokesIm[i] > fMaxY)
        {
            fMaxY = g_pfSumStokesIm[i];
        }
        if (g_pfSumStokesIm[i] < fMinY)
        {
            fMinY = g_pfSumStokesIm[i];
        }
    }
    cpgpanl(1, 4);
    cpgeras();
    cpgsvp(PG_VP_ML, PG_VP_MR, PG_VP_MB, PG_VP_MT);
    cpgswin(fMinFreq, fMaxFreq, fMinY, fMaxY);
    cpglab("Bin Number",
           "",
           "SumStokesIm");
    cpgbox("BCNST", 0.0, 0, "BCNST", 0.0, 0);
    cpgsci(PG_CI_PLOT);
    cpgline(g_iNFFT, g_pfFreq, g_pfSumStokesIm);
    cpgsci(PG_CI_DEF);

    return;
}
#endif

/*
 * Registers handlers for SIGTERM and CTRL+C
 */
int RegisterSignalHandlers()
{
    struct sigaction stSigHandler = {{0}};
    int iRet = GUPPI_OK;

    /* register the CTRL+C-handling function */
    stSigHandler.sa_handler = HandleStopSignals;
    iRet = sigaction(SIGINT, &stSigHandler, NULL);
    if (iRet != GUPPI_OK)
    {
        (void) fprintf(stderr,
                       "ERROR: Handler registration failed for signal %d!\n",
                       SIGINT);
        return GUPPI_ERR_GEN;
    }

    /* register the SIGTERM-handling function */
    stSigHandler.sa_handler = HandleStopSignals;
    iRet = sigaction(SIGTERM, &stSigHandler, NULL);
    if (iRet != GUPPI_OK)
    {
        (void) fprintf(stderr,
                       "ERROR: Handler registration failed for signal %d!\n",
                       SIGTERM);
        return GUPPI_ERR_GEN;
    }

    return GUPPI_OK;
}

/*
 * Catches SIGTERM and CTRL+C and cleans up before exiting
 */
void HandleStopSignals(int iSigNo)
{
    /* clean up */
    CleanUp();

    /* exit */
    exit(GUPPI_OK);

    /* never reached */
    return;
}

void __VEGASCUDASafeCall(hipError_t iRet,
                         const char* pcFile,
                         const int iLine,
                         void (*pCleanUp)(void))
{
    if (iRet != hipSuccess)
    {
        (void) fprintf(stderr,
                       "ERROR: File <%s>, Line %d: %s\n",
                       pcFile,
                       iLine,
                       hipGetErrorString(iRet));
        /* free resources */
        (*pCleanUp)();
        exit(GUPPI_ERR_GEN);
    }

    return;
}

/*
 * Prints usage information
 */
void PrintUsage(const char *pcProgName)
{
    (void) printf("Usage: %s [options] <data-file>\n",
                  pcProgName);
    (void) printf("    -h  --help                           ");
    (void) printf("Display this usage information\n");
    (void) printf("    -n  --nfft <value>                   ");
    (void) printf("Number of points in FFT\n");
    (void) printf("    -p  --pfb                            ");
    (void) printf("Enable PFB\n");
    (void) printf("    -a  --nacc <value>                   ");
    (void) printf("Number of spectra to add\n");
#if PLOT
    (void) printf("    -s  --fsamp <value>                  ");
    (void) printf("Sampling frequency\n");
#endif

    return;
}

