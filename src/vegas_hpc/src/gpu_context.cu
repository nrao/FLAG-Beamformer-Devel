#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include "gpu_context.h"
#include "pfb_gpu.h"

// Ugly, but so much depends upon it
extern int run;

GpuContext::GpuContext() :
        _pf4FFTIn_d(0),
        _pf4FFTOut_d(0),
        _pc4InBuf(0),
        _pc4Data_d(0),
        _pc4DataRead_d(0),
        _dimBPFB(),
        _dimGPFB(),
        _dimBAccum(),
        _dimGAccum(),
        _pfPFBCoeff(0),
        _pfPFBCoeff_d(0),
        _pf4SumStokes_d(0),
        _nchan(0),
        _nsubband(0),
        _blanker()
{
    memset(&_stPlan, 0, sizeof(_stPlan));
    memset(&_first_time_heap_in_accum, 0, sizeof(_first_time_heap_in_accum));    
}

GpuContext::GpuContext(GpuContext *p, int nsubband, int nchan, int in_blok_siz, int out_blok_siz)
{
    _blanker.reset();
    memset(&_first_time_heap_in_accum, 0, sizeof(_first_time_heap_in_accum));
    if (p != 0)
    {
        // Move resources from p into this object and null out p's reference
        _pf4FFTIn_d    = p->_pf4FFTIn_d;     p->_pf4FFTIn_d = 0;
        _pf4FFTOut_d   = p->_pf4FFTOut_d;    p->_pf4FFTOut_d = 0;
        _pc4InBuf      = p->_pc4InBuf;       p->_pc4InBuf = 0;
        _pc4Data_d     = p->_pc4Data_d;      p->_pc4Data_d = 0;
        _pc4DataRead_d = p->_pc4DataRead_d;  p->_pc4DataRead_d = 0;
        _dimBPFB       = p->_dimBPFB;
        _dimGPFB       = p->_dimGPFB;
        _dimBAccum     = p->_dimBAccum;
        _pfPFBCoeff    = p->_pfPFBCoeff;     p->_pfPFBCoeff = 0;
        _pfPFBCoeff_d  = p->_pfPFBCoeff_d;   p->_pfPFBCoeff_d = 0;
        _pf4SumStokes_d= p->_pf4SumStokes_d; p->_pf4SumStokes_d = 0;
        _stPlan        = p->_stPlan;         p->_stPlan = 0;
        _nsubband      = p->_nsubband;       p->_nsubband = 0;
        _nchan         = p->_nchan;          p->_nchan   = 0;    
    }
    else
    {
        // If we have no object to consume, initialize everything to nil
        _pf4FFTIn_d    = 0;
        _pf4FFTOut_d   = 0;
        _pc4InBuf      = 0;
        _pc4Data_d     = 0;
        _pc4DataRead_d = 0;
        _pfPFBCoeff    = 0;
        _pfPFBCoeff_d  = 0;
        _pf4SumStokes_d= 0;
        _stPlan        = 0;
        _nchan         = 0;
        _nsubband      = 0;    
    }
    
    // Do we have new buffer geometry?
    if (_nsubband == nsubband &&
        _nchan    == nchan &&
        _in_block_size == in_blok_siz &&
        _out_block_size == out_blok_siz)
    {
        // Nothing changed, so we should be done
        printf("### No GPU reallocations necessary\n");
        return;
    }
    else
    {
        release_resources();       
        // setup the new configuration
        _nsubband = nsubband;
        _nchan = nchan;
        _in_block_size = in_blok_siz;
        _out_block_size = out_blok_siz;
        // Now allocate new resources for the new configuration
        init_resources();
    }
}

bool
GpuContext::verify_setup(int nsubband, int nchan, int in_block_size, int out_block_size)
{
    _blanker.reset();
    memset(&_first_time_heap_in_accum, 0, sizeof(_first_time_heap_in_accum));
    _first_time_heap_mjd = 0.0;
    _first_time_heap_in_accum_status_bits = 0; 
       
    // Does the setup match?
    if (_nsubband == nsubband &&
        _nchan    == nchan &&
        _in_block_size  == in_block_size &&
        _out_block_size == out_block_size)
        return true;
    return false;
}

int GpuContext::init_resources()
{
    int iDevCount = 0;
    hipDeviceProp_t stDevProp = {0};
    hipfftResult iCUFFTRet = HIPFFT_SUCCESS;
    int iRet = EXIT_SUCCESS;
    int iMaxThreadsPerBlock = 0;
    size_t buf_in_block_size;
    int iFileCoeff = 0;
    char acFileCoeff[256] = {0};

    
    buf_in_block_size    = _in_block_size;
    _first_time_heap_in_accum_status_bits = 0;

    /* since CUDASafeCall() calls hipGetErrorString(),
       it should not be used here - will cause crash if no CUDA device is
       found */
    (void) hipGetDeviceCount(&iDevCount);
    if (0 == iDevCount)
    {
        (void) fprintf(stderr, "ERROR: No CUDA-capable device found!\n");
        run = 0;
        return EXIT_FAILURE;
    }

    /* just use the first device */
    printf("pfb_gpu.cu: CUDA_SAFE_CALL(hipSetDevice(0))\n");
    CUDA_SAFE_CALL(hipSetDevice(0));

    CUDA_SAFE_CALL(hipGetDeviceProperties(&stDevProp, 0));
    iMaxThreadsPerBlock = stDevProp.maxThreadsPerBlock;
    printf("pfb_gpu.cu: iMaxThreadsPerBlock = %i\n", iMaxThreadsPerBlock);

    _pfPFBCoeff = (float *) malloc(_nsubband
                                   * VEGAS_NUM_TAPS
                                   * _nchan
                                   * sizeof(float));
    if (NULL == _pfPFBCoeff)
    {
        (void) fprintf(stderr,
                       "ERROR: GpuContext Memory allocation failed! %s.\n",
                       strerror(errno));
        return EXIT_FAILURE;
    }

    /* allocate memory for the filter coefficient array on the device */

    printf("pfb_gpu.cu: before CUDA_SAFE_CALL(hipFree(0))\n");
    CUDA_SAFE_CALL(hipFree(0));
    printf("pfb_gpu.cu: after CUDA_SAFE_CALL(hipFree(0))\n");

    printf("pfb_gpu.cu:  before CUDA_SAFE_CALL(hipMalloc((void...\n");
    printf("subbands=%i, taps=%i, nchan=%i, floatsize=%i\n", _nsubband, VEGAS_NUM_TAPS, _nchan, sizeof(float));
    CUDA_SAFE_CALL(hipMalloc((void **) &_pfPFBCoeff_d,
                                       _nsubband
                                       * VEGAS_NUM_TAPS
                                       * _nchan
                                       * sizeof(float)));
    printf("pfb_gpu.cu:  CUDA_SAFE_CALL(hipMalloc((void...\n");

    /* read filter coefficients */
    /* Locate the coefficient directory.  This searches for the configuration
     * directory in one of YGOR_TELESCOPE, VEGAS_DIR or CONFIG_DIR
       If none of the environment variables above are specified
       then we punt and use the current working directory.
     */
     
     
    char *ygor_root = getenv("YGOR_TELESCOPE");
    char *vdir_root = getenv("VEGAS_DIR");
    char *config_root = getenv("CONFIG_DIR");
    char conf_dir_root[128];
    
    if (ygor_root)
    {
        /* Use YGOR_TELESCOPE if available */
        snprintf(conf_dir_root, sizeof(conf_dir_root), "%s/etc/config", ygor_root);
    }
    else if (config_root)
    {
        snprintf(conf_dir_root, sizeof(conf_dir_root), "%s", config_root);
    }
    else if (vdir_root)
    {
        snprintf(conf_dir_root, sizeof(conf_dir_root), "%s", vdir_root);
    }
    else
    {
        snprintf(conf_dir_root, sizeof(conf_dir_root), ".");
    }

    /* build file name */
    (void) snprintf(acFileCoeff, sizeof(acFileCoeff),
                   "%s/%s_%s_%d_%d_%d%s",
                   conf_dir_root,                   
                   FILE_COEFF_PREFIX,
                   FILE_COEFF_DATATYPE,
                   VEGAS_NUM_TAPS,
                   _nchan,
                   _nsubband,
                   FILE_COEFF_SUFFIX);

    iFileCoeff = open(acFileCoeff, O_RDONLY);
    if (iFileCoeff < EXIT_SUCCESS)
    {
        (void) fprintf(stderr,
                       "ERROR: Opening filter coefficients file %s "
                       "failed! %s.\n",
                       acFileCoeff,
                       strerror(errno));
        return EXIT_FAILURE;
    }

    iRet = read(iFileCoeff,
                _pfPFBCoeff,
                _nsubband * VEGAS_NUM_TAPS * _nchan * sizeof(float));
    if (iRet != (_nsubband * VEGAS_NUM_TAPS * _nchan * sizeof(float)))
    {
        (void) fprintf(stderr,
                       "ERROR: Reading filter coefficients failed! %s.\n",
                       strerror(errno));
        return EXIT_FAILURE;
    }
    (void) close(iFileCoeff);

    /* copy filter coefficients to the device */
    CUDA_SAFE_CALL(hipMemcpy(_pfPFBCoeff_d,
                              _pfPFBCoeff,
                              _nsubband * VEGAS_NUM_TAPS * _nchan * sizeof(float),
                              hipMemcpyHostToDevice));

    /* allocate memory for data array - 32MB is the block size for the VEGAS
       input buffer, allocate enough to hold two entire data blocks.
     */
    CUDA_SAFE_CALL(hipMalloc((void **) &_pc4Data_d,
                                       (buf_in_block_size * 2)
                                        ));
    printf("pfb_gpu.cu: CUDA_SAFE_CALL(hipMalloc((void...)\n");
    _pc4DataRead_d = _pc4Data_d;
    
    /* calculate kernel parameters */
    /* ASSUMPTION: gpuCtx._nchan >= iMaxThreadsPerBlock */
    _dimBPFB.x =   iMaxThreadsPerBlock;
    _dimBAccum.x = iMaxThreadsPerBlock;
    _dimGPFB.x =   (_nsubband * _nchan) / iMaxThreadsPerBlock;
    _dimGAccum.x = (_nsubband * _nchan) / iMaxThreadsPerBlock;

    CUDA_SAFE_CALL(hipMalloc((void **) &_pf4FFTIn_d,
                                 _nsubband * _nchan * sizeof(float4)));
    CUDA_SAFE_CALL(hipMalloc((void **) &_pf4FFTOut_d,
                                 _nsubband * _nchan * sizeof(float4)));
    CUDA_SAFE_CALL(hipMalloc((void **) &_pf4SumStokes_d,
                                 _nsubband * _nchan * sizeof(float4)));
    CUDA_SAFE_CALL(hipMemset(_pf4SumStokes_d,
                              0,
                              _nsubband * _nchan * sizeof(float4)));

    printf("pfb_gpu.cu: 4 CUDA_SAFE_CALL(hipMalloc...) calls\n");

    /* create plan */
    iCUFFTRet = hipfftPlanMany(&_stPlan,
                              FFTPLAN_RANK,
                              &_nchan,
                              &_nchan,
                              fft_in_stride(),
                              FFTPLAN_IDIST,
                              &_nchan,
                              fft_in_stride(),
                              FFTPLAN_ODIST,
                              HIPFFT_C2C,
                              fft_batch() );
    if (iCUFFTRet != HIPFFT_SUCCESS)
    {
        (void) fprintf(stderr, "ERROR: Plan creation failed!\n");
        run = 0;
        return EXIT_FAILURE;
    }
    printf("GPU resources resized for %d subbands and %d channels\n", _nsubband, _nchan);
    printf("#################### GPU RE-INIT COMPLETE ####################\n");
    return EXIT_SUCCESS;
}

void
GpuContext::blanking_inputs(int status)
{
    _blanker.new_input(status);
}

int
GpuContext::blank_current_fft()
{
    return _blanker.blank_current_fft();
}

int
GpuContext::needs_flush()
{
    return _blanker.needs_flush();
}

void
GpuContext::release_resources()
{
    // Free existing resources
    printf("Releasing GPU resources \n");
    
    if (_pc4InBuf != NULL)
    {
        free(_pc4InBuf);
        _pc4InBuf = NULL;
    }
    if (_pc4Data_d != NULL)
    {
        (void) hipFree(_pc4Data_d);
        _pc4Data_d = NULL;
    }
    if (_pf4FFTIn_d != NULL)
    {
        (void) hipFree(_pf4FFTIn_d);
        _pf4FFTIn_d = NULL;
    }
    if (_pf4FFTOut_d != NULL)
    {
        (void) hipFree(_pf4FFTOut_d);
        _pf4FFTOut_d = NULL;
    }
    if (_pf4SumStokes_d != NULL)
    {
        (void) hipFree(_pf4SumStokes_d);
        _pf4SumStokes_d = NULL;
    }

    /* destroy plan */
    /* TODO: check if plan exists */
    if (_stPlan)
    {
        (void) hipfftDestroy(_stPlan);
        _stPlan = NULL;
    }
}

