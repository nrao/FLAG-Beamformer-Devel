#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <pthread.h>
#include <errno.h>
#include <hipfft/hipfft.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>

#include "fitshead.h"
#include "vegas_error.h"
#ifdef __cplusplus
extern "C" {
#endif
#include "vegas_status.h"
#include "vegas_databuf.h"
#ifdef __cplusplus
}
#endif
#include "vegas_defines.h"
#include "pfb_gpu.h"
#include "pfb_gpu_kernels.h"
#include "spead_heap.h"

#define STATUS_KEY "GPUSTAT"

/* ASSUMPTIONS: 1. All blocks contain the same number of heaps. */

extern int run;

/**
 * Global variables: maybe move this to a struct that is passed to each function?
 */
 
class GpuContext
{
public:
    // stuff associated with gpu
    GpuContext();
    hipfftHandle _stPlan;
    float4* _pf4FFTIn_d;
    float4* _pf4FFTOut_d;
    char4*  _pc4InBuf;
    char4*  _pc4Data_d;              /* raw data starting address */
    char4*  _pc4DataRead_d;          /* raw data read pointer */
    dim3    _dimBPFB;
    dim3    _dimGPFB;
    dim3    _dimBAccum;
    dim3    _dimGAccum;
    float * _pfPFBCoeff;
    float * _pfPFBCoeff_d;
    float4* _pf4SumStokes_d;
    
    int     _nchan;
    int     _nsubband;
    
    int fft_in_stride()  { return 2*_nsubband; };
    int fft_out_stride() { return 2*_nsubband; };
    int fft_batch()      { return 2*_nsubband; };
    int accumulate();
    int do_fft();
    void zero_accumulator();
    int get_accumulated_spectrum_from_device(char *h_out);

};

GpuContext::GpuContext() :
        _pf4FFTIn_d(0),
        _pf4FFTOut_d(0),
        _pc4InBuf(0),
        _pc4Data_d(0),
        _pc4DataRead_d(0),
        _dimBPFB(),
        _dimGPFB(),
        _dimBAccum(),
        _dimGAccum(),
        _pfPFBCoeff(0),
        _pfPFBCoeff_d(0),
        _pf4SumStokes_d(0),
        _nchan(0),
        _nsubband(0)
{
    memset(&_stPlan, 0, sizeof(_stPlan));    
}


// Make the damn object global until we complete refactoring ...
GpuContext gpuCtx;

static size_t g_buf_out_block_size;
// static int g_nchan;

// static hipfftHandle g_stPlan = {0};
// static float4* g_pf4FFTIn_d = NULL;
// static float4* g_pf4FFTOut_d = NULL;
// static char4* g_pc4InBuf = NULL;
// static char4* g_pc4Data_d = NULL;              /* raw data starting address */
// static char4* g_pc4DataRead_d = NULL;          /* raw data read pointer */
// static dim3 g_dimBPFB(1, 1, 1);
// static dim3 g_dimGPFB(1, 1);
// static dim3 g_dimBAccum(1, 1, 1);
// static dim3 g_dimGAccum(1, 1);
// static float4* g_pf4SumStokes_d = NULL;
// static int g_iNumSubBands = 0;
// static float *g_pfPFBCoeff = NULL;
// static float *g_pfPFBCoeff_d = NULL;
static unsigned int g_iPrevBlankingState = FALSE;
static int g_iTotHeapOut = 0;
static int g_iMaxNumHeapOut = 0;
// static int g_iPFBCurBlockOut = 0;
static int g_iHeapOut = 0;
// static int g_iBlockInDataSize = 0;
/* these arrays need to be only a little longer than MAX_HEAPS_PER_BLK, but
   since we don't know the exact length, just allocate twice that value */
static unsigned int g_auiStatusBits[2*MAX_HEAPS_PER_BLK] = {0};
static unsigned int g_auiHeapValid[2*MAX_HEAPS_PER_BLK] = {0};
static int g_iFirstHeapIn = 0;
static int g_iSpecPerAcc = 0;

void __CUDASafeCall(hipError_t iCUDARet,
                               const char* pcFile,
                               const int iLine,
                               void (*pCleanUp)(void));

#define CUDASafeCall(iRet)   __CUDASafeCall(iRet,       \
                                                                  __FILE__,   \
                                                                  __LINE__,   \
                                                                  &cleanup_gpu)

#define CUDA_SAFE_CALL(call) \
do { \
    hipError_t err = call; \
    if (hipSuccess != err) { \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.", \
                 __FILE__, __LINE__, hipGetErrorString(err) ); \
        exit(EXIT_FAILURE); \
    } \
} while (0)


extern "C"
int init_cuda_context(void)
{
    int iDevCount = 0;

    /* since CUDASafeCall() calls hipGetErrorString(),
       it should not be used here - will cause crash if no CUDA device is
       found */
    (void) hipGetDeviceCount(&iDevCount);
    if (0 == iDevCount)
    {
        (void) fprintf(stderr, "ERROR: No CUDA-capable device found!\n");
        return EXIT_FAILURE;
    }

    /* just use the first device */
    printf("pfb_gpu.cu: CUDA_SAFE_CALL(hipSetDevice(0))\n");
    CUDA_SAFE_CALL(hipSetDevice(0));
    printf("pfb_gpu.cu: CUDA_SAFE_CALL(hipFree(0)\n");
    CUDA_SAFE_CALL(hipFree(0));
    printf("#################### GPU CONTEXT INITIALIZED ####################\n");
    return EXIT_SUCCESS;
}


/* Initialize all necessary memory, etc for doing PFB
 * at the given params.
 */
extern "C"
int init_gpu(size_t input_block_sz, size_t output_block_sz, int num_subbands, int num_chans)
{
    int iDevCount = 0;
    hipDeviceProp_t stDevProp = {0};
    hipfftResult iCUFFTRet = HIPFFT_SUCCESS;
    int iRet = EXIT_SUCCESS;
    int iMaxThreadsPerBlock = 0;
    size_t buf_in_block_size;
    int iFileCoeff = 0;
    char acFileCoeff[256] = {0};

    
    buf_in_block_size = input_block_sz;
    g_buf_out_block_size = output_block_sz;
    gpuCtx._nchan = num_chans;
    gpuCtx._nsubband = num_subbands;

    /* since CUDASafeCall() calls hipGetErrorString(),
       it should not be used here - will cause crash if no CUDA device is
       found */
    (void) hipGetDeviceCount(&iDevCount);
    if (0 == iDevCount)
    {
        (void) fprintf(stderr, "ERROR: No CUDA-capable device found!\n");
        run = 0;
        return EXIT_FAILURE;
    }

    /* just use the first device */
    printf("pfb_gpu.cu: CUDA_SAFE_CALL(hipSetDevice(0))\n");
    CUDA_SAFE_CALL(hipSetDevice(0));

    CUDA_SAFE_CALL(hipGetDeviceProperties(&stDevProp, 0));
    iMaxThreadsPerBlock = stDevProp.maxThreadsPerBlock;
    printf("pfb_gpu.cu: iMaxThreadsPerBlock = %i\n", iMaxThreadsPerBlock);

    gpuCtx._pfPFBCoeff = (float *) malloc(gpuCtx._nsubband
                                          * VEGAS_NUM_TAPS
                                          * gpuCtx._nchan
                                          * sizeof(float));
    if (NULL == gpuCtx._pfPFBCoeff)
    {
        (void) fprintf(stderr,
                       "ERROR: Memory allocation failed! %s.\n",
                       strerror(errno));
        return EXIT_FAILURE;
    }

    /* allocate memory for the filter coefficient array on the device */

    printf("pfb_gpu.cu: before CUDA_SAFE_CALL(hipFree(0))\n");
    CUDA_SAFE_CALL(hipFree(0));
    printf("pfb_gpu.cu: after CUDA_SAFE_CALL(hipFree(0))\n");

    printf("pfb_gpu.cu:  before CUDA_SAFE_CALL(hipMalloc((void...\n");
    printf("%i, %i, %i, %i\n", gpuCtx._nsubband, VEGAS_NUM_TAPS,  gpuCtx._nchan, sizeof(float));
    CUDA_SAFE_CALL(hipMalloc((void **) &gpuCtx._pfPFBCoeff_d,
                                       gpuCtx._nsubband
                                       * VEGAS_NUM_TAPS
                                       * gpuCtx._nchan
                                       * sizeof(float)));
    printf("pfb_gpu.cu:  CUDA_SAFE_CALL(hipMalloc((void...\n");

    /* read filter coefficients */
    /* Locate the coefficient directory.  This searches for the configuration
     * directory in one of YGOR_TELESCOPE, VEGAS_DIR or CONFIG_DIR
       If none of the environment variables above are specified
       then we punt and use the current working directory.
     */
     
     
    char *ygor_root = getenv("YGOR_TELESCOPE");
    char *vdir_root = getenv("VEGAS_DIR");
    char *config_root = getenv("CONFIG_DIR");
    char conf_dir_root[128];
    
    if (ygor_root)
    {
        /* Use YGOR_TELESCOPE if available */
        snprintf(conf_dir_root, sizeof(conf_dir_root), "%s/etc/config", ygor_root);
    }
    else if (config_root)
    {
        snprintf(conf_dir_root, sizeof(conf_dir_root), "%s", config_root);
    }
    else if (vdir_root)
    {
        snprintf(conf_dir_root, sizeof(conf_dir_root), "%s", vdir_root);
    }
    else
    {
        snprintf(conf_dir_root, sizeof(conf_dir_root), ".");
    }

    /* build file name */
    (void) snprintf(acFileCoeff, sizeof(acFileCoeff),
                   "%s/%s_%s_%d_%d_%d%s",
                   conf_dir_root,                   
                   FILE_COEFF_PREFIX,
                   FILE_COEFF_DATATYPE,
                   VEGAS_NUM_TAPS,
                   gpuCtx._nchan,
                   gpuCtx._nsubband,
                   FILE_COEFF_SUFFIX);

    iFileCoeff = open(acFileCoeff, O_RDONLY);
    if (iFileCoeff < EXIT_SUCCESS)
    {
        (void) fprintf(stderr,
                       "ERROR: Opening filter coefficients file %s "
                       "failed! %s.\n",
                       acFileCoeff,
                       strerror(errno));
        return EXIT_FAILURE;
    }

    iRet = read(iFileCoeff,
                gpuCtx._pfPFBCoeff,
                gpuCtx._nsubband * VEGAS_NUM_TAPS * gpuCtx._nchan * sizeof(float));
    if (iRet != (gpuCtx._nsubband * VEGAS_NUM_TAPS * gpuCtx._nchan * sizeof(float)))
    {
        (void) fprintf(stderr,
                       "ERROR: Reading filter coefficients failed! %s.\n",
                       strerror(errno));
        return EXIT_FAILURE;
    }
    (void) close(iFileCoeff);

    /* copy filter coefficients to the device */
    CUDA_SAFE_CALL(hipMemcpy(gpuCtx._pfPFBCoeff_d,
               gpuCtx._pfPFBCoeff,
               gpuCtx._nsubband * VEGAS_NUM_TAPS * gpuCtx._nchan * sizeof(float),
               hipMemcpyHostToDevice));

    /* allocate memory for data array - 32MB is the block size for the VEGAS
       input buffer, allocate 32MB + space for (VEGAS_NUM_TAPS - 1) blocks of
       data
       NOTE: the actual data in a 32MB block will be only
       (num_heaps * heap_size), but since we don't know that value until data
       starts flowing, allocate the maximum possible size */
    CUDA_SAFE_CALL(hipMalloc((void **) &gpuCtx._pc4Data_d,
                                       (buf_in_block_size
                                        + ((VEGAS_NUM_TAPS - 1)
                                           * gpuCtx._nsubband
                                           * gpuCtx._nchan
                                           * sizeof(char4)))));
    printf("pfb_gpu.cu: CUDA_SAFE_CALL(hipMalloc((void...)\n");
    gpuCtx._pc4DataRead_d = gpuCtx._pc4Data_d;
    
    g_iPrevBlankingState = TRUE;
    g_iTotHeapOut = 0;
    g_iHeapOut = 0;
    g_iSpecPerAcc = 0;

    /* calculate kernel parameters */
    /* ASSUMPTION: gpuCtx._nchan >= iMaxThreadsPerBlock */
    gpuCtx._dimBPFB.x = iMaxThreadsPerBlock;
    gpuCtx._dimBAccum.x = iMaxThreadsPerBlock;
    gpuCtx._dimGPFB.x = (gpuCtx._nsubband * gpuCtx._nchan) / iMaxThreadsPerBlock;
    gpuCtx._dimGAccum.x = (gpuCtx._nsubband * gpuCtx._nchan) / iMaxThreadsPerBlock;

    CUDA_SAFE_CALL(hipMalloc((void **) &gpuCtx._pf4FFTIn_d,
                                 gpuCtx._nsubband * gpuCtx._nchan * sizeof(float4)));
    CUDA_SAFE_CALL(hipMalloc((void **) &gpuCtx._pf4FFTOut_d,
                                 gpuCtx._nsubband * gpuCtx._nchan * sizeof(float4)));
    CUDA_SAFE_CALL(hipMalloc((void **) &gpuCtx._pf4SumStokes_d,
                                 gpuCtx._nsubband * gpuCtx._nchan * sizeof(float4)));
    CUDA_SAFE_CALL(hipMemset(gpuCtx._pf4SumStokes_d,
                                 '\0',
                                 gpuCtx._nsubband * gpuCtx._nchan * sizeof(float4)));

    printf("pfb_gpu.cu: 4 CUDA_SAFE_CALL(hipMalloc...) calls\n");

    /* create plan */
    iCUFFTRet = hipfftPlanMany(&gpuCtx._stPlan,
                              FFTPLAN_RANK,
                              &gpuCtx._nchan,
                              &gpuCtx._nchan,
                              gpuCtx.fft_in_stride(),
                              FFTPLAN_IDIST,
                              &gpuCtx._nchan,
                              gpuCtx.fft_in_stride(),
                              FFTPLAN_ODIST,
                              HIPFFT_C2C,
                              gpuCtx.fft_batch() );
    if (iCUFFTRet != HIPFFT_SUCCESS)
    {
        (void) fprintf(stderr, "ERROR: Plan creation failed!\n");
        run = 0;
        return EXIT_FAILURE;
    }

    printf("#################### GPU RE-INIT COMPLETE ####################\n");
    return EXIT_SUCCESS;
}

struct freq_spead_heap *
frequency_heap(struct vegas_databuf *db, int iblk, int iHeap)
{
    struct freq_spead_heap *freq_heap;
    char *ptr;
    ptr = (char *)(vegas_databuf_data(db, iblk) +
                   sizeof(struct freq_spead_heap) * iHeap);
    freq_heap = (struct freq_spead_heap*)ptr;                
    return freq_heap;
}

/* dump to buffer */
int dump_to_buffer(struct vegas_databuf *db_out,         // Output databuffer
                   int curblk_out,                       // Current output block
                   int iHeapOut,                         // output frequency heap number in current block
                   struct time_spead_heap *firsttimeheap,// first time sample of input 
                   int iTotHeapOut,                      // spectrum number/counter
                   int iSpecPerAcc,                      // GPU accumulations in this heap
                   double heap_mjd )                     // MJD from index_input
{
    struct freq_spead_heap *freq_heap_out;
    char * payload_addr_out;
    struct databuf_index *index_out;
    int rtn;
    
    freq_heap_out = frequency_heap(db_out, curblk_out, iHeapOut);
    index_out = (struct databuf_index*)vegas_databuf_index(db_out, curblk_out);
    
    payload_addr_out = (char*)(vegas_databuf_data(db_out, curblk_out) +
                        sizeof(struct freq_spead_heap) * MAX_HEAPS_PER_BLK +
                        (index_out->heap_size - sizeof(struct freq_spead_heap)) * iHeapOut);                        

    /* Write new heap header fields */
    freq_heap_out->time_cntr_id = 0x20;
    freq_heap_out->time_cntr_top8 = firsttimeheap->time_cntr_top8;
    freq_heap_out->time_cntr = firsttimeheap->time_cntr;
    freq_heap_out->spectrum_cntr_id = 0x21;
    freq_heap_out->spectrum_cntr = iTotHeapOut;
    freq_heap_out->integ_size_id = 0x22;
    freq_heap_out->integ_size = iSpecPerAcc;
    freq_heap_out->mode_id = 0x23;
    freq_heap_out->mode = firsttimeheap->mode;
    freq_heap_out->status_bits_id = 0x24;
    freq_heap_out->status_bits = firsttimeheap->status_bits;
    freq_heap_out->payload_data_off_addr_mode = 0;
    freq_heap_out->payload_data_off_id = 0x25;
    freq_heap_out->payload_data_off = 0;
    
/////DEBUG
    memset(firsttimeheap, 0, sizeof(struct time_spead_heap));

    /* Update output index */
    index_out->cpu_gpu_buf[iHeapOut].heap_valid = 1;
    index_out->cpu_gpu_buf[iHeapOut].heap_cntr = iTotHeapOut;
    index_out->cpu_gpu_buf[iHeapOut].heap_rcvd_mjd = heap_mjd;

    /* copy out GPU data into buffer */
    rtn = gpuCtx.get_accumulated_spectrum_from_device(payload_addr_out);
    index_out->num_heaps += (rtn == VEGAS_OK ? 1 : 0);
    return rtn;
}

/* Actually do the PFB by calling CUDA kernels */
extern "C"
void do_pfb(struct vegas_databuf *db_in,
            int curblock_in,
            struct vegas_databuf *db_out,
            int *curblock_out,
            int first,
            struct vegas_status st,
            int acc_len)
{
    /* Declare local variables */
    char *hdr_out = NULL;
    struct databuf_index *index_in = NULL;
    struct databuf_index *index_out = NULL;
    int heap_in = 0;
    char *heap_addr_in = NULL;
    struct time_spead_heap first_time_heap_in_accum;
    int iProcData = 0;
    hipError_t iCUDARet = hipSuccess;
    int iRet = VEGAS_OK;
    char* payload_addr_in = NULL;
    int num_in_heaps_per_proc = 0;
    int pfb_count = 0;
    int num_in_heaps_gpu_buffer = 0;
    int num_in_heaps_tail = 0;
    int i = 0;
    int iBlockInDataSize;

    /* Setup input and first output data block stuff */
    index_in = (struct databuf_index*)vegas_databuf_index(db_in, curblock_in);
    /* Get the number of heaps per block of data that will be processed by the GPU */
    num_in_heaps_per_proc = (gpuCtx._nsubband * gpuCtx._nchan * sizeof(char4)) / (index_in->heap_size - sizeof(struct time_spead_heap));
    iBlockInDataSize = (index_in->num_heaps * index_in->heap_size) - (index_in->num_heaps * sizeof(struct time_spead_heap));

    num_in_heaps_tail = (((VEGAS_NUM_TAPS - 1) * gpuCtx._nsubband * gpuCtx._nchan * sizeof(char4))
                         / (index_in->heap_size - sizeof(struct time_spead_heap)));
    num_in_heaps_gpu_buffer = index_in->num_heaps + num_in_heaps_tail;

    /* Calculate the maximum number of output heaps per block */
    g_iMaxNumHeapOut = (g_buf_out_block_size - (sizeof(struct time_spead_heap) * MAX_HEAPS_PER_BLK)) / (gpuCtx._nsubband * gpuCtx._nchan * sizeof(float4));

    hdr_out = vegas_databuf_header(db_out, *curblock_out);
    index_out = (struct databuf_index*)vegas_databuf_index(db_out, *curblock_out);
    // index_out->num_heaps = 0;
    memcpy(hdr_out, vegas_databuf_header(db_in, curblock_in),
            VEGAS_STATUS_SIZE);

    /* Set basic params in output index */
    index_out->heap_size = sizeof(struct freq_spead_heap) + (gpuCtx._nsubband * gpuCtx._nchan * sizeof(float4));
    /* Read in heap from buffer */
    heap_addr_in = (char*)(vegas_databuf_data(db_in, curblock_in) +
                        sizeof(struct time_spead_heap) * heap_in);
    // first_time_heap_in_accum = (struct time_spead_heap*)(heap_addr_in);
    memcpy(&first_time_heap_in_accum, heap_addr_in, sizeof(first_time_heap_in_accum));
    if (first)
    {
        g_iFirstHeapIn = heap_in;
    }
    /* Here, the payload_addr_in is the start of the contiguous block of data that will be
       copied to the GPU (heap_in = 0) */
    payload_addr_in = (char*)(vegas_databuf_data(db_in, curblock_in) +
                        sizeof(struct time_spead_heap) * MAX_HEAPS_PER_BLK +
                        (index_in->heap_size - sizeof(struct time_spead_heap)) * heap_in );

    /* Copy data block to GPU */
    if (first)
    {
        /* Sanity check for the first iteration */
        if ((iBlockInDataSize % (gpuCtx._nsubband * gpuCtx._nchan * sizeof(char4))) != 0)
        {
            (void) fprintf(stderr, "ERROR: Data size mismatch! BlockInDataSize=%d NumSubBands=%d nchan=%d\n",
                                    iBlockInDataSize, gpuCtx._nsubband, gpuCtx._nchan);
            run = 0;
            return;
        }
        CUDA_SAFE_CALL(hipMemcpy(gpuCtx._pc4Data_d,
                                payload_addr_in,
                                iBlockInDataSize,
                                hipMemcpyHostToDevice));
        /* duplicate the last (VEGAS_NUM_TAPS - 1) segm                payload_addr_out = (char*)(vegas_databuf_data(db_out, g_iPFBCurBlockOut) +
                                sizeof(struct freq_spead_heap) * MAX_HEAPS_PER_BLK +
                                (index_out->heap_size - sizeof(struct freq_spead_heap)) * g_iHeapOut);
ents at the end for
           the next iteration */
        CUDA_SAFE_CALL(hipMemcpy(gpuCtx._pc4Data_d + (iBlockInDataSize / sizeof(char4)),
                                gpuCtx._pc4Data_d + (iBlockInDataSize / sizeof(char4)) - ((VEGAS_NUM_TAPS - 1) * gpuCtx._nsubband * gpuCtx._nchan),
                                ((VEGAS_NUM_TAPS - 1) * gpuCtx._nsubband * gpuCtx._nchan * sizeof(char4)),
                                hipMemcpyDeviceToDevice));

        /* copy the status bits and valid flags for all heaps to arrays separate
           from the index, so that it can be combined with the corresponding
           values from the previous block */
        struct time_spead_heap* time_heap = (struct time_spead_heap*) vegas_databuf_data(db_in, curblock_in);
        for (i = 0; i < index_in->num_heaps; ++i)
        {
            g_auiStatusBits[i] = time_heap->status_bits;
            g_auiHeapValid[i] = index_in->cpu_gpu_buf[i].heap_valid;
            ++time_heap;
        }
        /* duplicate the last (VEGAS_NUM_TAPS - 1) segments at the end for the
           next iteration */
        for ( ; i < index_in->num_heaps + num_in_heaps_tail; ++i)
        {
            g_auiStatusBits[i] = g_auiStatusBits[i-num_in_heaps_tail];
            g_auiHeapValid[i] = g_auiHeapValid[i-num_in_heaps_tail];
        }
    }
    else
    {
        /* If this is not the first run, need to handle block boundary, while doing the PFB */
        CUDA_SAFE_CALL(hipMemcpy(gpuCtx._pc4Data_d,
                                gpuCtx._pc4Data_d + (iBlockInDataSize / sizeof(char4)),
                                ((VEGAS_NUM_TAPS - 1) * gpuCtx._nsubband * gpuCtx._nchan * sizeof(char4)),
                                hipMemcpyDeviceToDevice));
        CUDA_SAFE_CALL(hipMemcpy(gpuCtx._pc4Data_d + ((VEGAS_NUM_TAPS - 1) * gpuCtx._nsubband * gpuCtx._nchan),
                                payload_addr_in,
                                iBlockInDataSize,
                                hipMemcpyHostToDevice));
        /* copy the status bits and valid flags for all heaps to arrays separate
           from the index, so that it can be combined with the corresponding
           values from the previous block */
        for (i = 0; i < num_in_heaps_tail; ++i)
        {
            g_auiStatusBits[i] = g_auiStatusBits[index_in->num_heaps+i];
            g_auiHeapValid[i] = g_auiHeapValid[index_in->num_heaps+i];
        }
        struct time_spead_heap* time_heap = (struct time_spead_heap*) vegas_databuf_data(db_in, curblock_in);
        for ( ; i < num_in_heaps_tail + index_in->num_heaps; ++i)
        {
            g_auiStatusBits[i] = time_heap->status_bits;
            g_auiHeapValid[i] = index_in->cpu_gpu_buf[i-num_in_heaps_tail].heap_valid;
            ++time_heap;
        }
    }

    gpuCtx._pc4DataRead_d = gpuCtx._pc4Data_d;
    iProcData = 0;
    while (iBlockInDataSize > iProcData)  /* loop till (num_heaps * heap_size) of data is processed */
    {
        if (0 == pfb_count)
        {
            /* Check if all heaps necessary for this PFB are valid */
            if (!(is_valid(heap_in, (VEGAS_NUM_TAPS * num_in_heaps_per_proc))))
            {
                /* Skip all heaps that go into this PFB if there is an invalid heap */
                iProcData += (VEGAS_NUM_TAPS * gpuCtx._nsubband * gpuCtx._nchan * sizeof(char4));
                /* update the data read pointer */
                gpuCtx._pc4DataRead_d += (VEGAS_NUM_TAPS * gpuCtx._nsubband * gpuCtx._nchan);
                if (iProcData >= iBlockInDataSize)
                {
                    break;
                }

                /* Calculate input heap addresses for the next round of processing */
                heap_in += (VEGAS_NUM_TAPS * num_in_heaps_per_proc);
                if (heap_in > num_in_heaps_gpu_buffer)
                {
                    /* This is not supposed to happen (but may happen if odd number of pkts are dropped
                       right at the end of the buffer, so we therefore do not exit) */
                    (void) fprintf(stdout,
                                   "WARNING: Heap count %d exceeds available number of heaps %d!\n",
                                   heap_in,
                                   num_in_heaps_gpu_buffer);
                }
                heap_addr_in = (char*)(vegas_databuf_data(db_in, curblock_in) +
                                    sizeof(struct time_spead_heap) * heap_in);
                memcpy(&first_time_heap_in_accum, heap_addr_in, sizeof(first_time_heap_in_accum));
                continue;
            }
        }

        /* Perform polyphase filtering */
        DoPFB<<<gpuCtx._dimGPFB, gpuCtx._dimBPFB>>>(gpuCtx._pc4DataRead_d,
                                                    gpuCtx._pf4FFTIn_d,
                                                    gpuCtx._pfPFBCoeff_d);
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        iCUDARet = hipGetLastError();
        if (iCUDARet != hipSuccess)
        {
            (void) fprintf(stdout,
                           "ERROR: File <%s>, Line %d: %s\n",
                           __FILE__,
                           __LINE__,
                           hipGetErrorString(iCUDARet));
            run = 0;
            break;
        }

        iRet = gpuCtx.do_fft();
        if (iRet != VEGAS_OK)
        {
            (void) fprintf(stdout, "ERROR: FFT failed!\n");
            run = 0;
            break;
        }

        /* Accumulate power x, power y, stokes real and imag, if the blanking
           bit is not set */
        if (!(is_blanked(heap_in, num_in_heaps_per_proc)))
        {
            iRet = gpuCtx.accumulate();
            if (iRet != VEGAS_OK)
            {
                (void) fprintf(stdout, "ERROR: Accumulation failed!\n");
                run = 0;
                break;
            }
            ++g_iSpecPerAcc;
            g_iPrevBlankingState = FALSE;
        }
        else
        {
            /* state just changed */
            if (FALSE == g_iPrevBlankingState)
            {
                /* dump to buffer */
                iRet = dump_to_buffer(db_out,             // Output databuffer
                                      *curblock_out,                       // Current output block
                                      g_iHeapOut,                         // output frequency heap number in current block
                                      &first_time_heap_in_accum,// first time sample of input 
                                      g_iTotHeapOut,                      // spectrum number/counter
                                      g_iSpecPerAcc,                      // GPU accumulations in this heap
                                      index_in->cpu_gpu_buf[g_iFirstHeapIn].heap_rcvd_mjd); // MJD from index_input
                                                                       
                if (iRet != VEGAS_OK)
                {
                    (void) fprintf(stdout, "ERROR: Getting accumulated spectrum failed (blank state changed)!\n");
                    run = 0;
                    break;
                }

                ++g_iHeapOut;
                ++g_iTotHeapOut;

                /* zero accumulators */
                gpuCtx.zero_accumulator();
                /* reset time */
                g_iSpecPerAcc = 0;
                g_iPrevBlankingState = TRUE;
            }
        }
        if (g_iSpecPerAcc == acc_len)
        {
            /* dump to buffer */
            iRet = dump_to_buffer(db_out,             
                                  *curblock_out,
                                  g_iHeapOut,
                                  &first_time_heap_in_accum,
                                  g_iTotHeapOut,
                                  g_iSpecPerAcc,
                                  index_in->cpu_gpu_buf[g_iFirstHeapIn].heap_rcvd_mjd);
                                  
            if (iRet != VEGAS_OK)
            {
                (void) fprintf(stdout, "ERROR: Getting accumulated spectrum failed!\n");
                run = 0;
                break;
            }                                  
            ++g_iHeapOut;
            ++g_iTotHeapOut;

            /* zero accumulators */
            gpuCtx.zero_accumulator();
            /* reset time */
            g_iSpecPerAcc = 0;
        }

        iProcData += (gpuCtx._nsubband * gpuCtx._nchan * sizeof(char4));
        /* update the data read pointer */
        gpuCtx._pc4DataRead_d += (gpuCtx._nsubband * gpuCtx._nchan);

        /* Calculate input heap addresses for the next round of processing */
        heap_in += num_in_heaps_per_proc;
        heap_addr_in = (char*)(vegas_databuf_data(db_in, curblock_in) +
                            sizeof(struct time_spead_heap) * heap_in);
        if (0 == g_iSpecPerAcc)
        {
            // first_time_heap_in_accum = (struct time_spead_heap*)(heap_addr_in);
            memcpy(&first_time_heap_in_accum, heap_addr_in, sizeof(first_time_heap_in_accum));
            g_iFirstHeapIn = heap_in;
        }

        /* if output block is full */
        if (g_iHeapOut == g_iMaxNumHeapOut)
        {
            /* Set the number of heaps written to this block */
            // JJB index_out->num_heaps = g_iHeapOut;
            // printf("gpu filled num_heaps=%d snum=%d\n", index_out->num_heaps, g_iTotHeapOut);

            /* Mark output buffer as filled */
            vegas_databuf_set_filled(db_out, *curblock_out);

            // printf("Debug: vegas_pfb_thread going to next output block\n");

            /* Note current output block */
            /* NOTE: vegas_status_lock_safe() and vegas_status_unlock_safe() are macros
               that have been explicitly expanded here, due to compilation issues */
            //vegas_status_lock_safe(&st);
                pthread_cleanup_push((void (*) (void *))&vegas_status_unlock, (void *) &st);
                vegas_status_lock(&st);
            hputi4(st.buf, "PFBBLKOU", *curblock_out);
            //vegas_status_unlock_safe(&st);
                vegas_status_unlock(&st);
                pthread_cleanup_pop(0);

            /*  Wait for next output block */
            *curblock_out = (*curblock_out + 1) % db_out->n_block;
            while ((vegas_databuf_wait_free(db_out, *curblock_out)!=0) && run) {
                //vegas_status_lock_safe(&st);
                    pthread_cleanup_push((void (*)(void *))&vegas_status_unlock, (void *) &st);
                    vegas_status_lock(&st);

                hputs(st.buf, STATUS_KEY, "blocked");
                //vegas_status_unlock_safe(&st);
                    vegas_status_unlock(&st);
                    pthread_cleanup_pop(0);
            }

            g_iHeapOut = 0;

            hdr_out = vegas_databuf_header(db_out, *curblock_out);
            index_out = (struct databuf_index*)vegas_databuf_index(db_out, *curblock_out);
            index_out->num_heaps = 0;
            memcpy(hdr_out, vegas_databuf_header(db_in, curblock_in),
                    VEGAS_STATUS_SIZE);

            /* Set basic params in output index */
            index_out->heap_size = sizeof(struct freq_spead_heap) + (gpuCtx._nsubband * gpuCtx._nchan * sizeof(float4));
        }

        pfb_count = (pfb_count + 1) % VEGAS_NUM_TAPS;
    }

    return;
}

/* function that performs the FFT */
int GpuContext::do_fft()
{
    hipfftResult iCUFFTRet = HIPFFT_SUCCESS;

    /* execute plan */
    iCUFFTRet = hipfftExecC2C(_stPlan,
                             (hipfftComplex*) _pf4FFTIn_d,
                             (hipfftComplex*) _pf4FFTOut_d,
                             HIPFFT_FORWARD);
    if (iCUFFTRet != HIPFFT_SUCCESS)
    {
        (void) fprintf(stderr, "ERROR: FFT failed!");
        run = 0;
        return VEGAS_ERR_GEN;
    }

    return VEGAS_OK;
}

int GpuContext::accumulate()
{
    hipError_t iCUDARet = hipSuccess;

    Accumulate<<<_dimGAccum, _dimBAccum>>>(_pf4FFTOut_d,
                                           _pf4SumStokes_d);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    iCUDARet = hipGetLastError();
    if (iCUDARet != hipSuccess)
    {
        (void) fprintf(stderr, hipGetErrorString(iCUDARet));
        run = 0;
        return VEGAS_ERR_GEN;
    }

    return VEGAS_OK;
}

void GpuContext::zero_accumulator()
{
    CUDA_SAFE_CALL(hipMemset(_pf4SumStokes_d,
                                       '\0',
                                       (_nsubband
                                       * _nchan
                                       * sizeof(float4))));

    return;
}

int GpuContext::get_accumulated_spectrum_from_device(char *out)
{
    /* copy the negative frequencies out first */
    CUDASafeCall(hipMemcpy(out,
                            _pf4SumStokes_d + (_nsubband * _nchan / 2),
                            (_nsubband
                             * (_nchan / 2)
                             * sizeof(float4)),
                            hipMemcpyDeviceToHost));
    /* copy the positive frequencies out */
    CUDASafeCall(hipMemcpy(out + (_nsubband * (_nchan / 2) * sizeof(float4)),
                            _pf4SumStokes_d,
                            (_nsubband
                             * (_nchan / 2)
                             * sizeof(float4)),
                            hipMemcpyDeviceToHost));
    return VEGAS_OK;
}

/*
 * function to be used to check if any heap within the current PFB is invalid,
 * in which case, the entire PFB should be discarded.
 * NOTE: this function does not check ALL heaps - it returns at the first
 * invalid heap.
 */
int is_valid(int heap_start, int num_heaps)
{
    for (int i = heap_start; i < (heap_start + num_heaps); ++i)
    {
        if (!g_auiHeapValid[i])
        {
            return FALSE;
        }
    }

    return TRUE;
}

/*
 * function that checks if blanking has started within this accumulation.
 * ASSUMPTION: the blanking bit does not toggle within this time interval.
 */
int is_blanked(int heap_start, int num_heaps)
{
    for (int i = heap_start; i < (heap_start + num_heaps); ++i)
    {
        if (g_auiStatusBits[i] & 0x08)
        {
            return TRUE;
        }
    }

    return FALSE;
}

void __CUDASafeCall(hipError_t iCUDARet,
                               const char* pcFile,
                               const int iLine,
                               void (*pCleanUp)(void))
{
    if (iCUDARet != hipSuccess)
    {
        (void) fprintf(stderr,
                       "ERROR: File <%s>, Line %d: %s\n",
                       pcFile,
                       iLine,
                       hipGetErrorString(iCUDARet));
        run = 0;
        return;
    }

    return;
}

/*
 * Frees up any allocated memory.
 */
extern "C"
void cleanup_gpu()
{
    /* free memory */
    if (gpuCtx._pc4InBuf != NULL)
    {
        free(gpuCtx._pc4InBuf);
        gpuCtx._pc4InBuf = NULL;
    }
    if (gpuCtx._pc4Data_d != NULL)
    {
        (void) hipFree(gpuCtx._pc4Data_d);
        gpuCtx._pc4Data_d = NULL;
    }
    if (gpuCtx._pf4FFTIn_d != NULL)
    {
        (void) hipFree(gpuCtx._pf4FFTIn_d);
        gpuCtx._pf4FFTIn_d = NULL;
    }
    if (gpuCtx._pf4FFTOut_d != NULL)
    {
        (void) hipFree(gpuCtx._pf4FFTOut_d);
        gpuCtx._pf4FFTOut_d = NULL;
    }
    if (gpuCtx._pf4SumStokes_d != NULL)
    {
        (void) hipFree(gpuCtx._pf4SumStokes_d);
        gpuCtx._pf4SumStokes_d = NULL;
    }

    /* destroy plan */
    /* TODO: check if plan exists */
    if (gpuCtx._stPlan)
    {
        (void) hipfftDestroy(gpuCtx._stPlan);
        gpuCtx._stPlan = NULL;
    }
    printf("#################### GPU CONTEXT CLEANED UP ####################\n");
 
    return;
}
