#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <pthread.h>
#include <errno.h>
#include <hipfft/hipfft.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>

#include "fitshead.h"
#include "vegas_error.h"
#ifdef __cplusplus
extern "C" {
#endif
#include "vegas_status.h"
#include "vegas_databuf.h"
#ifdef __cplusplus
}
#endif
#include "vegas_defines.h"
#include "pfb_gpu.h"
#include "pfb_gpu_kernels.h"
#include "spead_heap.h"

#include "BlankingStateMachine.h"

#define STATUS_KEY "GPUSTAT"

/* ASSUMPTIONS: 1. All blocks contain the same number of heaps. */

extern int run;

/**
 * Global variables: maybe move this to a struct that is passed to each function?
 */
 
class GpuContext
{
public:
    // stuff associated with gpu
    GpuContext();
    GpuContext(GpuContext *, int nchan, int nsubbands, int inblocksz, int outblksz);
    hipfftHandle _stPlan;
    float4* _pf4FFTIn_d;
    float4* _pf4FFTOut_d;
    char4*  _pc4InBuf;
    char4*  _pc4Data_d;              /* raw data starting address */
    char4*  _pc4DataRead_d;          /* raw data read pointer */
    dim3    _dimBPFB;
    dim3    _dimGPFB;
    dim3    _dimBAccum;
    dim3    _dimGAccum;
    float * _pfPFBCoeff;
    float * _pfPFBCoeff_d;
    float4* _pf4SumStokes_d;
    
    int     _nchan;
    int     _nsubband;
    int     _in_block_size;
    int     _out_block_size;
    int     _init_status;
    int     _first_time_heap_in_accum_status_bits;
    double  _first_time_heap_mjd;
    
    BlankingStateMachine _blanker;
    
    struct time_spead_heap _first_time_heap_in_accum;
    
    int fft_in_stride()  { return 2*_nsubband; };
    int fft_out_stride() { return 2*_nsubband; };
    int fft_batch()      { return 2*_nsubband; };
    int accumulate();
    int do_fft();
    void zero_accumulator();
    int get_accumulated_spectrum_from_device(char *h_out);
    int init_resources();
    void release_resources();
    int init_status()    { return _init_status; }
    bool verify_setup(int num_subbands, int num_chans, 
                      int input_block_sz, int output_block_sz);
    void blanking_inputs(int);
    int  blank_current_fft();
    int  needs_flush();
    int  sw_status_changed(int swstat) { return _blanker.sw_status_changed(swstat); }
    BlankingStateMachine * blanker() { return &_blanker; } // for debug only!!


};

GpuContext::GpuContext() :
        _pf4FFTIn_d(0),
        _pf4FFTOut_d(0),
        _pc4InBuf(0),
        _pc4Data_d(0),
        _pc4DataRead_d(0),
        _dimBPFB(),
        _dimGPFB(),
        _dimBAccum(),
        _dimGAccum(),
        _pfPFBCoeff(0),
        _pfPFBCoeff_d(0),
        _pf4SumStokes_d(0),
        _nchan(0),
        _nsubband(0),
        _blanker()
{
    memset(&_stPlan, 0, sizeof(_stPlan));
    memset(&_first_time_heap_in_accum, 0, sizeof(_first_time_heap_in_accum));    
}

GpuContext::GpuContext(GpuContext *p, int nsubband, int nchan, int in_blok_siz, int out_blok_siz)
{
    _blanker.reset();
    memset(&_first_time_heap_in_accum, 0, sizeof(_first_time_heap_in_accum));
    if (p != 0)
    {
        // Move resources from p into this object and null out p's reference
        _pf4FFTIn_d    = p->_pf4FFTIn_d;     p->_pf4FFTIn_d = 0;
        _pf4FFTOut_d   = p->_pf4FFTOut_d;    p->_pf4FFTOut_d = 0;
        _pc4InBuf      = p->_pc4InBuf;       p->_pc4InBuf = 0;
        _pc4Data_d     = p->_pc4Data_d;      p->_pc4Data_d = 0;
        _pc4DataRead_d = p->_pc4DataRead_d;  p->_pc4DataRead_d = 0;
        _dimBPFB       = p->_dimBPFB;
        _dimGPFB       = p->_dimGPFB;
        _dimBAccum     = p->_dimBAccum;
        _pfPFBCoeff    = p->_pfPFBCoeff;     p->_pfPFBCoeff = 0;
        _pfPFBCoeff_d  = p->_pfPFBCoeff_d;   p->_pfPFBCoeff_d = 0;
        _pf4SumStokes_d= p->_pf4SumStokes_d; p->_pf4SumStokes_d = 0;
        _stPlan        = p->_stPlan;         p->_stPlan = 0;
        _nsubband      = p->_nsubband;       p->_nsubband = 0;
        _nchan         = p->_nchan;          p->_nchan   = 0;    
    }
    else
    {
        // If we have no object to consume, initialize everything to nil
        _pf4FFTIn_d    = 0;
        _pf4FFTOut_d   = 0;
        _pc4InBuf      = 0;
        _pc4Data_d     = 0;
        _pc4DataRead_d = 0;
        _pfPFBCoeff    = 0;
        _pfPFBCoeff_d  = 0;
        _pf4SumStokes_d= 0;
        _stPlan        = 0;
        _nchan         = 0;
        _nsubband      = 0;    
    }
    
    // Do we have new buffer geometry?
    if (_nsubband == nsubband &&
        _nchan    == nchan &&
        _in_block_size == in_blok_siz &&
        _out_block_size == out_blok_siz)
    {
        // Nothing changed, so we should be done
        printf("### No GPU reallocations necessary\n");
        return;
    }
    else
    {
        release_resources();       
        // setup the new configuration
        _nsubband = nsubband;
        _nchan = nchan;
        _in_block_size = in_blok_siz;
        _out_block_size = out_blok_siz;
        // Now allocate new resources for the new configuration
        init_resources();
    }
}

bool
GpuContext::verify_setup(int nsubband, int nchan, int in_block_size, int out_block_size)
{
    _blanker.reset();
    memset(&_first_time_heap_in_accum, 0, sizeof(_first_time_heap_in_accum)); 
       
    // Does the setup match?
    if (_nsubband == nsubband &&
        _nchan    == nchan &&
        _in_block_size  == in_block_size &&
        _out_block_size == out_block_size)
        return true;
    return false;
}

void
GpuContext::blanking_inputs(int status)
{
    _blanker.new_input(status);
}

int
GpuContext::blank_current_fft()
{
    return _blanker.blank_current_fft();
}

int
GpuContext::needs_flush()
{
    return _blanker.needs_flush();
}

// Make the damn object global until we complete refactoring ...
GpuContext *gpuCtx = 0;

static size_t g_buf_out_block_size;
static int g_iTotHeapOut = 0;
static int g_iMaxNumHeapOut = 0;
static int g_iHeapOut = 0;

/* these arrays need to be only a little longer than MAX_HEAPS_PER_BLK, but
   since we don't know the exact length, just allocate twice that value */
static unsigned int g_auiStatusBits[2*MAX_HEAPS_PER_BLK] = {0};
static unsigned int g_auiHeapValid[2*MAX_HEAPS_PER_BLK] = {0};
static int g_iSpecPerAcc = 0;

void __CUDASafeCall(hipError_t iCUDARet,
                               const char* pcFile,
                               const int iLine,
                               void (*pCleanUp)(void));

#define CUDASafeCall(iRet)   __CUDASafeCall(iRet,       \
                                                                  __FILE__,   \
                                                                  __LINE__,   \
                                                                  &cleanup_gpu)

#define CUDA_SAFE_CALL(call) \
do { \
    hipError_t err = call; \
    if (hipSuccess != err) { \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.", \
                 __FILE__, __LINE__, hipGetErrorString(err) ); \
        exit(EXIT_FAILURE); \
    } \
} while (0)


extern "C"
int init_cuda_context(int subbands, int chans, int inBlokSz, int outBlokSz)
{
    int iDevCount = 0;

    /* since CUDASafeCall() calls hipGetErrorString(),
       it should not be used here - will cause crash if no CUDA device is
       found */
    (void) hipGetDeviceCount(&iDevCount);
    if (0 == iDevCount)
    {
        (void) fprintf(stderr, "ERROR: No CUDA-capable device found!\n");
        return EXIT_FAILURE;
    }        

    if (subbands == 0 || chans == 0)
    {

        /* just use the first device */
        printf("pfb_gpu.cu: CUDA_SAFE_CALL(hipSetDevice(0))\n");
        CUDA_SAFE_CALL(hipSetDevice(0));
        printf("pfb_gpu.cu: CUDA_SAFE_CALL(hipFree(0)\n");
        CUDA_SAFE_CALL(hipFree(0));
        printf("#################### GPU CONTEXT INITIALIZED ####################\n");
    }
    else
    {
        // Create a new mode specific set of resources.
        GpuContext *newctx, *oldctx;
        oldctx = gpuCtx;
        newctx = new GpuContext(oldctx, subbands, chans, inBlokSz, outBlokSz);
        gpuCtx = newctx;
        delete oldctx;
    }
    return EXIT_SUCCESS;
}


/* Initialize all necessary memory, etc for doing PFB
 * at the given params.
 */
extern "C" 
int reset_state(size_t input_block_sz, size_t output_block_sz, int num_subbands, int num_chans)
{
    g_iTotHeapOut = 0;
    g_iHeapOut = 0;
    g_iSpecPerAcc = 0;
    
    // Now verify we have the right setup
    if (gpuCtx == 0 || 
        true != gpuCtx->verify_setup(num_subbands, num_chans, input_block_sz, output_block_sz))
    {
        printf("Error: runtime and pre-init GPU setups didn't match\n");
        // For backward compatibility we try to initialize here.
        if (EXIT_SUCCESS != init_cuda_context(num_subbands, num_chans, input_block_sz, output_block_sz))
            return EXIT_FAILURE;
    }
    return EXIT_SUCCESS;
}

int GpuContext::init_resources()
{
    int iDevCount = 0;
    hipDeviceProp_t stDevProp = {0};
    hipfftResult iCUFFTRet = HIPFFT_SUCCESS;
    int iRet = EXIT_SUCCESS;
    int iMaxThreadsPerBlock = 0;
    size_t buf_in_block_size;
    int iFileCoeff = 0;
    char acFileCoeff[256] = {0};

    
    buf_in_block_size    = _in_block_size;
    g_buf_out_block_size = _out_block_size;
    _first_time_heap_in_accum_status_bits = 0;

    /* since CUDASafeCall() calls hipGetErrorString(),
       it should not be used here - will cause crash if no CUDA device is
       found */
    (void) hipGetDeviceCount(&iDevCount);
    if (0 == iDevCount)
    {
        (void) fprintf(stderr, "ERROR: No CUDA-capable device found!\n");
        run = 0;
        return EXIT_FAILURE;
    }

    /* just use the first device */
    printf("pfb_gpu.cu: CUDA_SAFE_CALL(hipSetDevice(0))\n");
    CUDA_SAFE_CALL(hipSetDevice(0));

    CUDA_SAFE_CALL(hipGetDeviceProperties(&stDevProp, 0));
    iMaxThreadsPerBlock = stDevProp.maxThreadsPerBlock;
    printf("pfb_gpu.cu: iMaxThreadsPerBlock = %i\n", iMaxThreadsPerBlock);

    _pfPFBCoeff = (float *) malloc(_nsubband
                                   * VEGAS_NUM_TAPS
                                   * _nchan
                                   * sizeof(float));
    if (NULL == _pfPFBCoeff)
    {
        (void) fprintf(stderr,
                       "ERROR: GpuContext Memory allocation failed! %s.\n",
                       strerror(errno));
        return EXIT_FAILURE;
    }

    /* allocate memory for the filter coefficient array on the device */

    printf("pfb_gpu.cu: before CUDA_SAFE_CALL(hipFree(0))\n");
    CUDA_SAFE_CALL(hipFree(0));
    printf("pfb_gpu.cu: after CUDA_SAFE_CALL(hipFree(0))\n");

    printf("pfb_gpu.cu:  before CUDA_SAFE_CALL(hipMalloc((void...\n");
    printf("subbands=%i, taps=%i, nchan=%i, floatsize=%i\n", _nsubband, VEGAS_NUM_TAPS, _nchan, sizeof(float));
    CUDA_SAFE_CALL(hipMalloc((void **) &_pfPFBCoeff_d,
                                       _nsubband
                                       * VEGAS_NUM_TAPS
                                       * _nchan
                                       * sizeof(float)));
    printf("pfb_gpu.cu:  CUDA_SAFE_CALL(hipMalloc((void...\n");

    /* read filter coefficients */
    /* Locate the coefficient directory.  This searches for the configuration
     * directory in one of YGOR_TELESCOPE, VEGAS_DIR or CONFIG_DIR
       If none of the environment variables above are specified
       then we punt and use the current working directory.
     */
     
     
    char *ygor_root = getenv("YGOR_TELESCOPE");
    char *vdir_root = getenv("VEGAS_DIR");
    char *config_root = getenv("CONFIG_DIR");
    char conf_dir_root[128];
    
    if (ygor_root)
    {
        /* Use YGOR_TELESCOPE if available */
        snprintf(conf_dir_root, sizeof(conf_dir_root), "%s/etc/config", ygor_root);
    }
    else if (config_root)
    {
        snprintf(conf_dir_root, sizeof(conf_dir_root), "%s", config_root);
    }
    else if (vdir_root)
    {
        snprintf(conf_dir_root, sizeof(conf_dir_root), "%s", vdir_root);
    }
    else
    {
        snprintf(conf_dir_root, sizeof(conf_dir_root), ".");
    }

    /* build file name */
    (void) snprintf(acFileCoeff, sizeof(acFileCoeff),
                   "%s/%s_%s_%d_%d_%d%s",
                   conf_dir_root,                   
                   FILE_COEFF_PREFIX,
                   FILE_COEFF_DATATYPE,
                   VEGAS_NUM_TAPS,
                   _nchan,
                   _nsubband,
                   FILE_COEFF_SUFFIX);

    iFileCoeff = open(acFileCoeff, O_RDONLY);
    if (iFileCoeff < EXIT_SUCCESS)
    {
        (void) fprintf(stderr,
                       "ERROR: Opening filter coefficients file %s "
                       "failed! %s.\n",
                       acFileCoeff,
                       strerror(errno));
        return EXIT_FAILURE;
    }

    iRet = read(iFileCoeff,
                _pfPFBCoeff,
                _nsubband * VEGAS_NUM_TAPS * _nchan * sizeof(float));
    if (iRet != (_nsubband * VEGAS_NUM_TAPS * _nchan * sizeof(float)))
    {
        (void) fprintf(stderr,
                       "ERROR: Reading filter coefficients failed! %s.\n",
                       strerror(errno));
        return EXIT_FAILURE;
    }
    (void) close(iFileCoeff);

    /* copy filter coefficients to the device */
    CUDA_SAFE_CALL(hipMemcpy(_pfPFBCoeff_d,
                              _pfPFBCoeff,
                              _nsubband * VEGAS_NUM_TAPS * _nchan * sizeof(float),
                              hipMemcpyHostToDevice));

    /* allocate memory for data array - 32MB is the block size for the VEGAS
       input buffer, allocate 32MB + space for (VEGAS_NUM_TAPS - 1) blocks of
       data
       NOTE: the actual data in a 32MB block will be only
       (num_heaps * heap_size), but since we don't know that value until data
       starts flowing, allocate the maximum possible size */
    CUDA_SAFE_CALL(hipMalloc((void **) &_pc4Data_d,
                                       (buf_in_block_size
                                        + ((VEGAS_NUM_TAPS - 1)
                                           * _nsubband
                                           * _nchan
                                           * sizeof(char4)))));
    printf("pfb_gpu.cu: CUDA_SAFE_CALL(hipMalloc((void...)\n");
    _pc4DataRead_d = _pc4Data_d;
    
    /* calculate kernel parameters */
    /* ASSUMPTION: gpuCtx._nchan >= iMaxThreadsPerBlock */
    _dimBPFB.x =   iMaxThreadsPerBlock;
    _dimBAccum.x = iMaxThreadsPerBlock;
    _dimGPFB.x =   (_nsubband * _nchan) / iMaxThreadsPerBlock;
    _dimGAccum.x = (_nsubband * _nchan) / iMaxThreadsPerBlock;

    CUDA_SAFE_CALL(hipMalloc((void **) &_pf4FFTIn_d,
                                 _nsubband * _nchan * sizeof(float4)));
    CUDA_SAFE_CALL(hipMalloc((void **) &_pf4FFTOut_d,
                                 _nsubband * _nchan * sizeof(float4)));
    CUDA_SAFE_CALL(hipMalloc((void **) &_pf4SumStokes_d,
                                 _nsubband * _nchan * sizeof(float4)));
    CUDA_SAFE_CALL(hipMemset(_pf4SumStokes_d,
                              0,
                              _nsubband * _nchan * sizeof(float4)));

    printf("pfb_gpu.cu: 4 CUDA_SAFE_CALL(hipMalloc...) calls\n");

    /* create plan */
    iCUFFTRet = hipfftPlanMany(&_stPlan,
                              FFTPLAN_RANK,
                              &_nchan,
                              &_nchan,
                              fft_in_stride(),
                              FFTPLAN_IDIST,
                              &_nchan,
                              fft_in_stride(),
                              FFTPLAN_ODIST,
                              HIPFFT_C2C,
                              fft_batch() );
    if (iCUFFTRet != HIPFFT_SUCCESS)
    {
        (void) fprintf(stderr, "ERROR: Plan creation failed!\n");
        run = 0;
        return EXIT_FAILURE;
    }
    printf("GPU resources resized for %d subbands and %d channels\n", _nsubband, _nchan);
    printf("#################### GPU RE-INIT COMPLETE ####################\n");
    return EXIT_SUCCESS;
}

void
GpuContext::release_resources()
{
    // Free existing resources
    printf("Releasing GPU resources \n");
    
    if (_pc4InBuf != NULL)
    {
        free(_pc4InBuf);
        _pc4InBuf = NULL;
    }
    if (_pc4Data_d != NULL)
    {
        (void) hipFree(_pc4Data_d);
        _pc4Data_d = NULL;
    }
    if (_pf4FFTIn_d != NULL)
    {
        (void) hipFree(_pf4FFTIn_d);
        _pf4FFTIn_d = NULL;
    }
    if (_pf4FFTOut_d != NULL)
    {
        (void) hipFree(_pf4FFTOut_d);
        _pf4FFTOut_d = NULL;
    }
    if (_pf4SumStokes_d != NULL)
    {
        (void) hipFree(_pf4SumStokes_d);
        _pf4SumStokes_d = NULL;
    }

    /* destroy plan */
    /* TODO: check if plan exists */
    if (_stPlan)
    {
        (void) hipfftDestroy(_stPlan);
        _stPlan = NULL;
    }
}

struct freq_spead_heap *
frequency_heap(struct vegas_databuf *db, int iblk, int iHeap)
{
    struct freq_spead_heap *freq_heap;
    char *ptr;
    ptr = (char *)(vegas_databuf_data(db, iblk) +
                   sizeof(struct freq_spead_heap) * iHeap);
    freq_heap = (struct freq_spead_heap*)ptr;                
    return freq_heap;
}

/* dump to buffer */
int dump_to_buffer(struct vegas_databuf *db_out,         // Output databuffer
                   int curblk_out,                       // Current output block
                   int iHeapOut,                         // output frequency heap number in current block
                   struct time_spead_heap *firsttimeheap,// first time sample of input 
                   int iTotHeapOut,                      // spectrum number/counter
                   int iSpecPerAcc,                      // GPU accumulations in this heap
                   double heap_mjd,                      // MJD from index_input
                   int first_t_series_status)            // switch state of first accumulation
{
    struct freq_spead_heap *freq_heap_out;
    char * payload_addr_out;
    struct databuf_index *index_out;
    int rtn;
    
    freq_heap_out = vegas_datablock_freq_heap_header(db_out, curblk_out, iHeapOut);
    index_out = (struct databuf_index*)vegas_databuf_index(db_out, curblk_out);
    
    payload_addr_out = vegas_datablock_freq_heap_data(db_out, curblk_out, iHeapOut);

    /* Write new heap header fields */
    freq_heap_out->time_cntr_id = 0x20;
    freq_heap_out->time_cntr_top8 = firsttimeheap->time_cntr_top8;
    freq_heap_out->time_cntr = firsttimeheap->time_cntr;
    freq_heap_out->spectrum_cntr_id = 0x21;
    freq_heap_out->spectrum_cntr = iTotHeapOut;
    freq_heap_out->integ_size_id = 0x22;
    freq_heap_out->integ_size = iSpecPerAcc;
    freq_heap_out->mode_id = 0x23;
    freq_heap_out->mode = firsttimeheap->mode;
    freq_heap_out->status_bits_id = 0x24;
    freq_heap_out->status_bits = first_t_series_status;
    freq_heap_out->payload_data_off_addr_mode = 0;
    freq_heap_out->payload_data_off_id = 0x25;
    freq_heap_out->payload_data_off = 0;
    
/////DEBUG
    memset(firsttimeheap, 0, sizeof(struct time_spead_heap));

    /* Update output index */
    index_out->cpu_gpu_buf[iHeapOut].heap_valid = 1;
    index_out->cpu_gpu_buf[iHeapOut].heap_cntr = iTotHeapOut;
    index_out->cpu_gpu_buf[iHeapOut].heap_rcvd_mjd = heap_mjd;

    /* copy out GPU data into buffer */
    rtn = gpuCtx->get_accumulated_spectrum_from_device(payload_addr_out);
    index_out->num_heaps += (rtn == VEGAS_OK ? 1 : 0);
    return rtn;
}
    
/* Actually do the PFB by calling CUDA kernels */
extern "C"
void do_pfb(struct vegas_databuf *db_in,
            int curblock_in,
            struct vegas_databuf *db_out,
            int *curblock_out,
            int first,
            struct vegas_status st,
            int acc_len)
{
    /* Declare local variables */
    char *hdr_out = NULL;
    struct databuf_index *index_in = NULL;
    struct databuf_index *index_out = NULL;
    int heap_in = 0;
    char *heap_addr_in = NULL;

    int iProcData = 0;
    hipError_t iCUDARet = hipSuccess;
    int iRet = VEGAS_OK;
    char* payload_addr_in = NULL;
    int num_in_heaps_per_proc = 0;
    int pfb_count = 0;
    int num_in_heaps_gpu_buffer = 0;
    int num_in_heaps_tail = 0;
    int i = 0;
    int iBlockInDataSize;
    int nsubband_x_nchan;
    size_t nsubband_x_nchan_fsize;
    size_t nsubband_x_nchan_csize;

    nsubband_x_nchan = gpuCtx->_nsubband * gpuCtx->_nchan;
    nsubband_x_nchan_fsize = nsubband_x_nchan * sizeof(float4);
    nsubband_x_nchan_csize = nsubband_x_nchan * sizeof(char4);
    
    /* Setup input and first output data block stuff */
    index_in = (struct databuf_index*)vegas_databuf_index(db_in, curblock_in);
    /* Get the number of heaps per block of data that will be processed by the GPU */
    num_in_heaps_per_proc = nsubband_x_nchan_csize / time_heap_datasize(index_in);
    iBlockInDataSize = index_in->num_heaps * time_heap_datasize(index_in);

    num_in_heaps_tail = ((VEGAS_NUM_TAPS - 1) * nsubband_x_nchan_csize)
                         / time_heap_datasize(index_in);
    num_in_heaps_gpu_buffer = index_in->num_heaps + num_in_heaps_tail;

    /* Calculate the maximum number of output heaps per block */
    // Seems like this should have freq_spead_heap not time_spead_heap??
    g_iMaxNumHeapOut = (g_buf_out_block_size - (sizeof(struct freq_spead_heap) * MAX_HEAPS_PER_BLK)) / nsubband_x_nchan_fsize;

    hdr_out = vegas_databuf_header(db_out, *curblock_out);
    index_out = (struct databuf_index*)vegas_databuf_index(db_out, *curblock_out);
    // index_out->num_heaps = 0;
    memcpy(hdr_out, vegas_databuf_header(db_in, curblock_in), VEGAS_STATUS_SIZE);

    /* Set basic params in output index */
    index_out->heap_size = sizeof(struct freq_spead_heap) + (nsubband_x_nchan_fsize);
    /* Read in heap from buffer */
    heap_addr_in = (char*)vegas_datablock_time_heap_header(db_in, curblock_in, heap_in);

    /* Here, the payload_addr_in is the start of the contiguous block of data that will be
       copied to the GPU (heap_in = 0) */
    payload_addr_in = vegas_datablock_time_heap_data(db_in, curblock_in, heap_in);

    /* Copy data block to GPU */
    if (first)
    {
        // bloksz replaces a calculated value which caused the check below to fail
        // in the presence of dropped packets. We used the blocksize from the data buffer
        // instead here to get things going. Not sure how dropped data at start of
        // scan should be treated.
        int bloksz;
        bloksz = db_in->block_size;
        // bloksz = (index_in->num_heaps * index_in->heap_size) - (index_in->num_heaps * sizeof(struct time_spead_heap));
        /* Sanity check for the first iteration */
        if ((bloksz % (nsubband_x_nchan_csize)) != 0)
        {
            (void) fprintf(stderr, "ERROR: Data size mismatch! BlockInDataSize=%d NumSubBands=%d nchan=%d\n",
                                    bloksz, gpuCtx->_nsubband, gpuCtx->_nchan);
            run = 0;
            return;
        }
        // Cuda Note: hipMemcpy host to device is asynchronous, be supposedly safe.
        CUDA_SAFE_CALL(hipMemcpy(gpuCtx->_pc4Data_d,
                                payload_addr_in,
                                bloksz,
                                hipMemcpyHostToDevice));
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        iCUDARet = hipGetLastError();
        if (iCUDARet != hipSuccess)
        {
            (void) fprintf(stderr, hipGetErrorString(iCUDARet));
        }
                                
        /* duplicate the last (VEGAS_NUM_TAPS - 1) segments at the end for 
           the next iteration */
        CUDA_SAFE_CALL(hipMemcpy(gpuCtx->_pc4Data_d + (bloksz / sizeof(char4)),
                                  gpuCtx->_pc4Data_d + (bloksz - ((VEGAS_NUM_TAPS - 1) * nsubband_x_nchan_csize))/sizeof(char4),
                                  ((VEGAS_NUM_TAPS - 1) * nsubband_x_nchan_csize),
                                  hipMemcpyDeviceToDevice));
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        iCUDARet = hipGetLastError();
        if (iCUDARet != hipSuccess)
        {
            (void) fprintf(stderr, hipGetErrorString(iCUDARet));
        }

        /* copy the status bits and valid flags for all heaps to arrays separate
           from the index, so that it can be combined with the corresponding
           values from the previous block */
        struct time_spead_heap* time_heap = (struct time_spead_heap*) vegas_databuf_data(db_in, curblock_in);
        for (i = 0; i < index_in->num_heaps; ++i)
        {
            g_auiStatusBits[i] = time_heap->status_bits;
            g_auiHeapValid[i] = index_in->cpu_gpu_buf[i].heap_valid;
            ++time_heap;
        }
        /* duplicate the last (VEGAS_NUM_TAPS - 1) segments at the end for the
           next iteration */
        for ( ; i < index_in->num_heaps + num_in_heaps_tail; ++i)
        {
            g_auiStatusBits[i] = g_auiStatusBits[i-num_in_heaps_tail];
            g_auiHeapValid[i] = g_auiHeapValid[i-num_in_heaps_tail];
        }
        // Zero out accumulators for 1st integration
        gpuCtx->zero_accumulator();
    }
    else
    {
        /* If this is not the first run, need to handle block boundary, while doing the PFB */
        CUDA_SAFE_CALL(hipMemcpy(gpuCtx->_pc4Data_d,
                                gpuCtx->_pc4Data_d + (iBlockInDataSize / sizeof(char4)),
                                ((VEGAS_NUM_TAPS - 1) * nsubband_x_nchan_csize),
                                hipMemcpyDeviceToDevice));
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        iCUDARet = hipGetLastError();
        if (iCUDARet != hipSuccess)
        {
            (void) fprintf(stderr, hipGetErrorString(iCUDARet));
        }
                                
        // Cuda Note: hipMemcpy host to device is asynchronous, be supposedly safe.                                
        CUDA_SAFE_CALL(hipMemcpy(gpuCtx->_pc4Data_d + ((VEGAS_NUM_TAPS - 1) * nsubband_x_nchan),
                                payload_addr_in,
                                iBlockInDataSize,
                                hipMemcpyHostToDevice));
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        iCUDARet = hipGetLastError();
        if (iCUDARet != hipSuccess)
        {
            (void) fprintf(stderr, hipGetErrorString(iCUDARet));
        }
                                
        /* copy the status bits and valid flags for all heaps to arrays separate
           from the index, so that it can be combined with the corresponding
           values from the previous block */
        for (i = 0; i < num_in_heaps_tail; ++i)
        {
            g_auiStatusBits[i] = g_auiStatusBits[index_in->num_heaps+i];
            g_auiHeapValid[i] = g_auiHeapValid[index_in->num_heaps+i];
        }
        struct time_spead_heap* time_heap = (struct time_spead_heap*) vegas_databuf_data(db_in, curblock_in);
        for ( ; i < num_in_heaps_tail + index_in->num_heaps; ++i)
        {
            g_auiStatusBits[i] = time_heap->status_bits;           
            g_auiHeapValid[i] = index_in->cpu_gpu_buf[i-num_in_heaps_tail].heap_valid;
            ++time_heap;
        }
    }

    gpuCtx->_pc4DataRead_d = gpuCtx->_pc4Data_d;
    iProcData = 0;
    while (iBlockInDataSize > iProcData)  /* loop till (num_heaps * heap_size) of data is processed */
    {
        if (0 == pfb_count)
        {
            /* Check if all heaps necessary for this PFB are valid */
            if (!(is_valid(heap_in, (VEGAS_NUM_TAPS * num_in_heaps_per_proc))))
            {
                /* Skip all heaps that go into this PFB if there is an invalid heap */
                iProcData += (VEGAS_NUM_TAPS * nsubband_x_nchan_csize);
                /* update the data read pointer */
                gpuCtx->_pc4DataRead_d += (VEGAS_NUM_TAPS * nsubband_x_nchan);
                if (iProcData >= iBlockInDataSize)
                {
                    break;
                }

                /* Calculate input heap addresses for the next round of processing */
                heap_in += (VEGAS_NUM_TAPS * num_in_heaps_per_proc);
                if (heap_in > num_in_heaps_gpu_buffer)
                {
                    /* This is not supposed to happen (but may happen if odd number of pkts are dropped
                       right at the end of the buffer, so we therefore do not exit) */
                    (void) fprintf(stdout,
                                   "WARNING: Heap count %d exceeds available number of heaps %d!\n",
                                   heap_in,
                                   num_in_heaps_gpu_buffer);
                }
                heap_addr_in = (char*)vegas_datablock_time_heap_header(db_in, curblock_in, heap_in);
                continue;
            }
        }
        /* Perform polyphase filtering */
        DoPFB<<<gpuCtx->_dimGPFB, gpuCtx->_dimBPFB>>>(gpuCtx->_pc4DataRead_d,
                                                      gpuCtx->_pf4FFTIn_d,
                                                      gpuCtx->_pfPFBCoeff_d);
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        iCUDARet = hipGetLastError();
        if (iCUDARet != hipSuccess)
        {
            (void) fprintf(stdout,
                           "ERROR: File <%s>, Line %d: %s\n",
                           __FILE__,
                           __LINE__,
                           hipGetErrorString(iCUDARet));
            run = 0;
            break;
        }

        iRet = gpuCtx->do_fft();
        if (iRet != VEGAS_OK)
        {
            (void) fprintf(stdout, "ERROR: FFT failed!\n");
            run = 0;
            break;
        }

        gpuCtx->blanking_inputs(is_blanked(heap_in, num_in_heaps_per_proc));
        ++g_iTotHeapOut; // unconditional spectrum counter
                                
        /* Accumulate power x, power y, stokes real and imag, if the blanking
           bit is not set */
        if (!(gpuCtx->blank_current_fft()))
        {
            iRet = gpuCtx->accumulate();
            if (iRet != VEGAS_OK)
            {
                (void) fprintf(stdout, "ERROR: Accumulation failed!\n");
                run = 0;
                break;
            }
            ++g_iSpecPerAcc;
            // record the first unblanked state in this accumulation sequence
            if (1 == g_iSpecPerAcc)
            {
                gpuCtx->_first_time_heap_in_accum_status_bits = g_auiStatusBits[heap_in];
                memcpy(&gpuCtx->_first_time_heap_in_accum, heap_addr_in, sizeof(gpuCtx->_first_time_heap_in_accum));
                gpuCtx->_first_time_heap_mjd = index_in->cpu_gpu_buf[heap_in].heap_rcvd_mjd;               
            }                    
        }
        
        if (g_iSpecPerAcc == acc_len || gpuCtx->needs_flush())
        {
            /* dump to buffer */
            iRet = dump_to_buffer(db_out,             
                                  *curblock_out,
                                  g_iHeapOut,
                                  &gpuCtx->_first_time_heap_in_accum,
                                  g_iTotHeapOut,
                                  g_iSpecPerAcc,
                                  gpuCtx->_first_time_heap_mjd,
                                  gpuCtx->_first_time_heap_in_accum_status_bits);
            
            if (iRet != VEGAS_OK)
            {
                (void) fprintf(stdout, "ERROR: Getting accumulated spectrum failed!\n");
                run = 0;
                break;
            }                                  
            ++g_iHeapOut;

            /* zero accumulators */
            gpuCtx->zero_accumulator();
            /* reset time */
            g_iSpecPerAcc = 0;
        }

        iProcData += nsubband_x_nchan_csize;
        /* update the data read pointer */
        gpuCtx->_pc4DataRead_d += (nsubband_x_nchan);

        /* Calculate input heap addresses for the next round of processing */
        heap_in += num_in_heaps_per_proc;
        heap_addr_in = (char*)vegas_datablock_time_heap_header(db_in, curblock_in, heap_in);

        /* if output block is full */
        if (g_iHeapOut == g_iMaxNumHeapOut)
        {
            /* Set the number of heaps written to this block */
            /* Mark output buffer as filled */
            vegas_databuf_set_filled(db_out, *curblock_out);

            // printf("Debug: vegas_pfb_thread going to next output block\n");

            /* Note current output block */
            /* NOTE: vegas_status_lock_safe() and vegas_status_unlock_safe() are macros
               that have been explicitly expanded here, due to compilation issues */
            //vegas_status_lock_safe(&st);
                pthread_cleanup_push((void (*) (void *))&vegas_status_unlock, (void *) &st);
                vegas_status_lock(&st);
            hputi4(st.buf, "PFBBLKOU", *curblock_out);
            //vegas_status_unlock_safe(&st);
                vegas_status_unlock(&st);
                pthread_cleanup_pop(0);

            /*  Wait for next output block */
            *curblock_out = (*curblock_out + 1) % db_out->n_block;
            while ((vegas_databuf_wait_free(db_out, *curblock_out)!=0) && run) {
                //vegas_status_lock_safe(&st);
                    pthread_cleanup_push((void (*)(void *))&vegas_status_unlock, (void *) &st);
                    vegas_status_lock(&st);

                hputs(st.buf, STATUS_KEY, "blocked");
                //vegas_status_unlock_safe(&st);
                    vegas_status_unlock(&st);
                    pthread_cleanup_pop(0);
            }

            g_iHeapOut = 0;

            hdr_out = vegas_databuf_header(db_out, *curblock_out);
            index_out = (struct databuf_index*)vegas_databuf_index(db_out, *curblock_out);
            index_out->num_heaps = 0;
            memcpy(hdr_out, vegas_databuf_header(db_in, curblock_in),
                    VEGAS_STATUS_SIZE);

            /* Set basic params in output index */
            index_out->heap_size = sizeof(struct freq_spead_heap) + (nsubband_x_nchan_fsize);
        }

        pfb_count = (pfb_count + 1) % VEGAS_NUM_TAPS;
    }

    return;
}

/* function that performs the FFT */
int GpuContext::do_fft()
{
    hipfftResult iCUFFTRet = HIPFFT_SUCCESS;
    hipError_t iCUDARet = hipSuccess;

    /* execute plan */
    iCUFFTRet = hipfftExecC2C(_stPlan,
                             (hipfftComplex*) _pf4FFTIn_d,
                             (hipfftComplex*) _pf4FFTOut_d,
                             HIPFFT_FORWARD);
    if (iCUFFTRet != HIPFFT_SUCCESS)
    {
        (void) fprintf(stderr, "ERROR: FFT failed!");
        iCUDARet = hipGetLastError();
        if (iCUDARet != hipSuccess)
        {
            (void) fprintf(stderr, hipGetErrorString(iCUDARet));
            run = 0;
            return VEGAS_ERR_GEN;
        }
    }
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    iCUDARet = hipGetLastError();
    if (iCUDARet != hipSuccess)
    {
        (void) fprintf(stderr, hipGetErrorString(iCUDARet));
        run = 0;
        return VEGAS_ERR_GEN;
    }

    return VEGAS_OK;
}

int GpuContext::accumulate()
{
    hipError_t iCUDARet = hipSuccess;

    Accumulate<<<_dimGAccum, _dimBAccum>>>(_pf4FFTOut_d,
                                           _pf4SumStokes_d);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    iCUDARet = hipGetLastError();
    if (iCUDARet != hipSuccess)
    {
        (void) fprintf(stderr, hipGetErrorString(iCUDARet));
        run = 0;
        return VEGAS_ERR_GEN;
    }

    return VEGAS_OK;
}

void GpuContext::zero_accumulator()
{
    hipError_t iCUDARet = hipSuccess;
    CUDA_SAFE_CALL(hipMemset(_pf4SumStokes_d,
                                       '\0',
                                       (_nsubband
                                       * _nchan
                                       * sizeof(float4))));
    CUDA_SAFE_CALL(hipDeviceSynchronize());                                       
    if (iCUDARet != hipSuccess)
    {
        (void) fprintf(stderr, hipGetErrorString(iCUDARet));
    }

    return;
}

int GpuContext::get_accumulated_spectrum_from_device(char *out)
{
    hipError_t iCUDARet = hipSuccess;
    // Cuda note: Device to host memcpy is always synchronous
    /* copy the negative frequencies out first */
    CUDASafeCall(hipMemcpy(out,
                            _pf4SumStokes_d + (_nsubband * _nchan / 2),
                            (_nsubband
                             * (_nchan / 2)
                             * sizeof(float4)),
                            hipMemcpyDeviceToHost));
    /* copy the positive frequencies out */
    CUDASafeCall(hipMemcpy(out + (_nsubband * (_nchan / 2) * sizeof(float4)),
                            _pf4SumStokes_d,
                            (_nsubband
                             * (_nchan / 2)
                             * sizeof(float4)),
                            hipMemcpyDeviceToHost));
    iCUDARet = hipGetLastError();
    if (iCUDARet != hipSuccess)
    {
        (void) fprintf(stderr, hipGetErrorString(iCUDARet));
    }

#ifdef DEBUG_ZERO_CHANNELS    
    // DEBUG check for near or zero channels X*X and Y*Y should never be =< 0.0                            
    int i, ndata, n_null = 0;
    ndata = _nsubband*_nchan;
    float4 *data = (float4 *)out;
    int first_bad = 0;
    
    for (i=0; i<ndata; ++i)
    {
        if (data[i].x <= 0.0 || data[i].y <= 0.0)
        {
            n_null++;
            first_bad=first_bad == 0 ? i : first_bad;
        }
    }
    if (n_null != 0)
    {
        printf("GPU: %d nil channels starting at %d\n", n_null, first_bad);
    }
#endif
    return VEGAS_OK;
}

/*
 * function to be used to check if any heap within the current PFB is invalid,
 * in which case, the entire PFB should be discarded.
 * NOTE: this function does not check ALL heaps - it returns at the first
 * invalid heap.
 */
int is_valid(int heap_start, int num_heaps)
{
    for (int i = heap_start; i < (heap_start + num_heaps); ++i)
    {
        if (!g_auiHeapValid[i])
        {
            return FALSE;
        }
    }

    return TRUE;
}

/*
A note about blanking:
The blanking status is copied from the time series input into the array
g_auiStatusBits, with time acending with index, like so:
    g_auiStatusBits[32] = t0
    g_auiStatusBits[33] = t0 + dt
    g_auiStatusBits[34] = t0 + dt + dt
    
So when we think about labeling the frequency heap outputs, the convention
is to use the '1st' non-blanked time-series (e.g. index 32 above) to fill 
in the timestamp, counter, mjd etc.

However, when we think about how to process blanking, we need to use the
most recent(e.g index 34 above), status to drive the blanking state machine.
Below, the check which sets 0x2 is taken from the most current time-series status.

 is_blanked(tail, length)
 * Check the input time series for blanking and encode
 * the result.
 * Return value: 
 *  - bit 0x4 -- indicates cal or sig/ref state changed during input
 *  - bit 0x2 -- indicates if the most recent time sample had blanking asserted
 *  - bit 0x1 -- indicates if any of the time samples had blanking asserted
 */
int is_blanked(int heap_start, int num_heaps)
{
    int state_changed = 0;
    int banked_at_start = (g_auiStatusBits[heap_start + num_heaps- 1] & 0x8)  ? 0x2 : 0x0;
    int is_blanked = (banked_at_start || (g_auiStatusBits[heap_start] & 0x8)) ? 0x1 : 0x0;
   
    for (int i = heap_start + 1; i < (heap_start + num_heaps); ++i)
    {
        if ((g_auiStatusBits[i] & 0x3) != (g_auiStatusBits[i-1] & 0x3))
        {
            state_changed = 0x4;
        }
        if (g_auiStatusBits[i] & 0x08)
        {
            is_blanked = 0x1;
        }
    }
    return (banked_at_start | state_changed | is_blanked);
}

void __CUDASafeCall(hipError_t iCUDARet,
                               const char* pcFile,
                               const int iLine,
                               void (*pCleanUp)(void))
{
    if (iCUDARet != hipSuccess)
    {
        (void) fprintf(stderr,
                       "ERROR: File <%s>, Line %d: %s\n",
                       pcFile,
                       iLine,
                       hipGetErrorString(iCUDARet));
        run = 0;
        return;
    }

    return;
}



/*
 * Frees up any allocated memory.
 */
extern "C"
void cleanup_gpu()
{
#if 0
    /* free memory */
    if (gpuCtx._pc4InBuf != NULL)
    {
        free(gpuCtx._pc4InBuf);
        gpuCtx._pc4InBuf = NULL;
    }
    if (gpuCtx._pc4Data_d != NULL)
    {
        (void) hipFree(gpuCtx._pc4Data_d);
        gpuCtx._pc4Data_d = NULL;
    }
    if (gpuCtx._pf4FFTIn_d != NULL)
    {
        (void) hipFree(gpuCtx._pf4FFTIn_d);
        gpuCtx._pf4FFTIn_d = NULL;
    }
    if (gpuCtx._pf4FFTOut_d != NULL)
    {
        (void) hipFree(gpuCtx._pf4FFTOut_d);
        gpuCtx._pf4FFTOut_d = NULL;
    }
    if (gpuCtx._pf4SumStokes_d != NULL)
    {
        (void) hipFree(gpuCtx._pf4SumStokes_d);
        gpuCtx._pf4SumStokes_d = NULL;
    }

    /* destroy plan */
    /* TODO: check if plan exists */
    if (gpuCtx._stPlan)
    {
        (void) hipfftDestroy(gpuCtx._stPlan);
        gpuCtx._stPlan = NULL;
    }
    printf("#################### GPU CONTEXT CLEANED UP ####################\n");
#endif 
    return;
}
