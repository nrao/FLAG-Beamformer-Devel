#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <string.h>
#include <pthread.h>
#include <errno.h>
#include <hipfft/hipfft.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>

#include "fitshead.h"
#include "vegas_error.h"
#ifdef __cplusplus
extern "C" {
#endif
#include "vegas_status.h"
#include "vegas_databuf.h"
#ifdef __cplusplus
}
#endif
#include "vegas_defines.h"
#include "pfb_gpu.h"
#include "pfb_gpu_kernels.h"
#include "spead_heap.h"

#include "BlankingStateMachine.h"
#include "gpu_context.h"

#define STATUS_KEY "GPUSTAT"

/* ASSUMPTIONS: 
   1. All blocks contain the same number of heaps. 
   2. All blocks contain a complete number of heaps 
      (i.e. no blocks with num_heaps < MAX_HEAPS_PER_BLOCK)
   3. Heaps which are prior to the start of scan are properly marked with the
      scan not started/blanking status bit.
   4. Packet loss will be indicated by setting the invalid index bit (or more consistenly)
      by setting the blanking status bit.
 */

extern int run;

/**
 * Global variables: maybe move this to a struct that is passed to each function?
 */
#include "gpu_context.h"

#include "DataBlockInfoCache.h"

// Make the damn object global until we complete refactoring ...
GpuContext *gpuCtx = 0;

// static size_t g_buf_out_block_size;
static int g_iTotHeapOut = 0;
static int g_iMaxNumHeapOut = 0;
static int g_iHeapOut = 0;
static DataBlockInfoCache blk_info_cache; 

static int g_iSpecPerAcc = 0;

void __CUDASafeCall(hipError_t iCUDARet,
                               const char* pcFile,
                               const int iLine,
                               void (*pCleanUp)(void));

#define CUDASafeCall(iRet)   __CUDASafeCall(iRet,       \
                                              __FILE__,   \
                                              __LINE__,   \
                                              &cleanup_gpu)



extern "C"
int init_cuda_context(int subbands, int chans, int inBlokSz, int outBlokSz)
{
    int iDevCount = 0;

    /* since CUDASafeCall() calls hipGetErrorString(),
       it should not be used here - will cause crash if no CUDA device is
       found */
    (void) hipGetDeviceCount(&iDevCount);
    if (0 == iDevCount)
    {
        (void) fprintf(stderr, "ERROR: No CUDA-capable device found!\n");
        return EXIT_FAILURE;
    }        

    if (subbands == 0 || chans == 0)
    {

        /* just use the first device */
        printf("pfb_gpu.cu: CUDA_SAFE_CALL(hipSetDevice(0))\n");
        CUDA_SAFE_CALL(hipSetDevice(0));
        printf("pfb_gpu.cu: CUDA_SAFE_CALL(hipFree(0)\n");
        CUDA_SAFE_CALL(hipFree(0));
        printf("#################### GPU CONTEXT INITIALIZED ####################\n");
    }
    else
    {
        // Create a new mode specific set of resources.
        GpuContext *newctx, *oldctx;
        oldctx = gpuCtx;
        newctx = new GpuContext(oldctx, subbands, chans, inBlokSz, outBlokSz);
        gpuCtx = newctx;
        delete oldctx;
    }
    return EXIT_SUCCESS;
}


/* Initialize all necessary memory, etc for doing PFB
 * at the given params.
 */
extern "C" 
int reset_state(size_t input_block_sz, size_t output_block_sz, int num_subbands, int num_chans)
{
    g_iTotHeapOut = 0;
    g_iHeapOut = 0;
    g_iSpecPerAcc = 0;
    
    // Now verify we have the right setup
    if (gpuCtx == 0 || 
        true != gpuCtx->verify_setup(num_subbands, num_chans, input_block_sz, output_block_sz))
    {
        printf("Error: runtime and pre-init GPU setups didn't match\n");
        // For backward compatibility we try to initialize here.
        if (EXIT_SUCCESS != init_cuda_context(num_subbands, num_chans, input_block_sz, output_block_sz))
            return EXIT_FAILURE;
    }
    return EXIT_SUCCESS;
}


struct freq_spead_heap *
frequency_heap(struct vegas_databuf *db, int iblk, int iHeap)
{
    struct freq_spead_heap *freq_heap;
    char *ptr;
    ptr = (char *)(vegas_databuf_data(db, iblk) +
                   sizeof(struct freq_spead_heap) * iHeap);
    freq_heap = (struct freq_spead_heap*)ptr;                
    return freq_heap;
}

/* dump to buffer */
int dump_to_buffer(struct vegas_databuf *db_out,         // Output databuffer
                   int curblk_out,                       // Current output block
                   int iHeapOut,                         // output frequency heap number in current block
                   struct time_spead_heap *firsttimeheap,// first time sample of input 
                   int iTotHeapOut,                      // spectrum number/counter
                   int iSpecPerAcc,                      // GPU accumulations in this heap
                   double heap_mjd,                      // MJD from index_input
                   int first_t_series_status)            // switch state of first accumulation
{
    struct freq_spead_heap *freq_heap_out;
    char * payload_addr_out;
    struct databuf_index *index_out;
    int rtn;
    
    freq_heap_out = vegas_datablock_freq_heap_header(db_out, curblk_out, iHeapOut);
    index_out = (struct databuf_index*)vegas_databuf_index(db_out, curblk_out);

    if (sizeof(struct freq_spead_heap) * MAX_HEAPS_PER_BLK + 
        index_out->heap_size*(index_out->num_heaps+1) > db_out->block_size ||
        iHeapOut >= db_out->index_size)
    {
        printf("DATABUF ERROR: heapsize*nheaps > blocksize!! (%d > %zd) index_size=%d\n",
            index_out->heap_size*index_out->num_heaps, db_out->block_size, db_out->index_size); 
        printf("DATABUF ERROR: blocknum=%d, iHeapOut=%d,iTotHeapOut=%d,iSpecPerAcc=%d\n",
            curblk_out, iHeapOut,iTotHeapOut, iSpecPerAcc);
    } 
        
    payload_addr_out = vegas_datablock_freq_heap_data(db_out, curblk_out, iHeapOut);

    /* Write new heap header fields */
    freq_heap_out->time_cntr_id = 0x20;
    freq_heap_out->time_cntr_top8 = firsttimeheap->time_cntr_top8;
    freq_heap_out->time_cntr = firsttimeheap->time_cntr;
    freq_heap_out->spectrum_cntr_id = 0x21;
    freq_heap_out->spectrum_cntr = iTotHeapOut;
    freq_heap_out->integ_size_id = 0x22;
    freq_heap_out->integ_size = iSpecPerAcc;
    freq_heap_out->mode_id = 0x23;
    freq_heap_out->mode = firsttimeheap->mode;
    freq_heap_out->status_bits_id = 0x24;
    freq_heap_out->status_bits = first_t_series_status;
    freq_heap_out->payload_data_off_addr_mode = 0;
    freq_heap_out->payload_data_off_id = 0x25;
    freq_heap_out->payload_data_off = 0;
    
/////DEBUG
    memset(firsttimeheap, 0, sizeof(struct time_spead_heap));

    /* Update output index */
    index_out->cpu_gpu_buf[iHeapOut].heap_valid = 1;
    index_out->cpu_gpu_buf[iHeapOut].heap_cntr = iTotHeapOut;
    index_out->cpu_gpu_buf[iHeapOut].heap_rcvd_mjd = heap_mjd;

    /* copy out GPU data into buffer */
    rtn = gpuCtx->get_accumulated_spectrum_from_device(payload_addr_out);
    index_out->num_heaps += (rtn == VEGAS_OK ? 1 : 0);
    return rtn;
}
    
/* Actually do the PFB by calling CUDA kernels */
extern "C"
void do_pfb(struct vegas_databuf *db_in,
            int curblock_in,
            struct vegas_databuf *db_out,
            int *curblock_out,
            int first,
            struct vegas_status st,
            int acc_len)
{
    /* Declare local variables */
    char *hdr_out = NULL;
    struct databuf_index *index_in = NULL;
    struct databuf_index *index_out = NULL;
    int heap_in = 0;

    int iProcData = 0;
    hipError_t iCUDARet = hipSuccess;
    int iRet = VEGAS_OK;
    char* payload_addr_in = NULL;
    int pfb_count = 0;
    int iBlockInDataSize;
    int nsubband_x_nchan;
    size_t nsubband_x_nchan_fsize;
    size_t nsubband_x_nchan_csize;
    int num_in_heaps_per_fft = 0;    
    int num_in_heaps_per_pfb;

    nsubband_x_nchan = gpuCtx->_nsubband * gpuCtx->_nchan;
    nsubband_x_nchan_fsize = nsubband_x_nchan * sizeof(float4);
    nsubband_x_nchan_csize = nsubband_x_nchan * sizeof(char4);
    
    /* Setup input and first output data block stuff */
    index_in = (struct databuf_index*)vegas_databuf_index(db_in, curblock_in);
    /* Get the number of heaps per block of data that will be processed by the GPU */
    num_in_heaps_per_fft = nsubband_x_nchan_csize / time_heap_datasize(index_in);
    num_in_heaps_per_pfb = VEGAS_NUM_TAPS * num_in_heaps_per_fft;
    
    iBlockInDataSize = index_in->num_heaps * time_heap_datasize(index_in);

    /* Calculate the maximum number of output heaps per block */
    g_iMaxNumHeapOut = (gpuCtx->_out_block_size - (sizeof(struct freq_spead_heap) * MAX_HEAPS_PER_BLK)) / nsubband_x_nchan_fsize;

    hdr_out = vegas_databuf_header(db_out, *curblock_out);
    index_out = (struct databuf_index*)vegas_databuf_index(db_out, *curblock_out);
    // index_out->num_heaps = 0;
    memcpy(hdr_out, vegas_databuf_header(db_in, curblock_in), VEGAS_STATUS_SIZE);

    /* Set basic params in output index */
    index_out->heap_size = sizeof(struct freq_spead_heap) + (nsubband_x_nchan_fsize);

    /* Here, the payload_addr_in is the start of the contiguous block of data that will be
       copied to the GPU (heap_in = 0) */
    payload_addr_in = vegas_datablock_time_heap_data(db_in, curblock_in, heap_in);
    
    if (iBlockInDataSize == 0)
    {
        fprintf(stderr, "iBlockInDataSize == 0! no data to process\n");
        run = 0;
        return;
    }

    /* Copy data block to GPU */
    if (first)
    {
        // bloksz replaces a calculated value which caused the check below to fail
        // in the presence of dropped packets. We used the blocksize from the data buffer
        // instead here to get things going. Not sure how dropped data at start of
        // scan should be treated.
        
        /* Sanity check for the first iteration */
        if ((iBlockInDataSize % (nsubband_x_nchan_csize)) != 0)
        {
            (void) fprintf(stderr, "ERROR: Data size mismatch on first block!\n  "
                                   "    BlockInDataSize=%d NumSubBands=%d nchan=%d %d heaps\n"
                                   "    skipping the entire block\n",
                                    iBlockInDataSize, gpuCtx->_nsubband, gpuCtx->_nchan,
                                    index_in->num_heaps);
            // run = 0;
            CUDA_SAFE_CALL(hipMemset(&gpuCtx->_pc4Data_d[iBlockInDataSize/sizeof(char4)],
                                      0x2, // something other than exactly zero
                                      MAX_HEAPS_PER_BLK * time_heap_datasize(index_in))); 
            gpuCtx->zero_accumulator();           
            return;
        }
        // Cuda Note: hipMemcpy host to device is asynchronous, be supposedly safe.
        CUDA_SAFE_CALL(hipMemcpy(&gpuCtx->_pc4Data_d[iBlockInDataSize/sizeof(char4)],
                                  payload_addr_in,
                                  iBlockInDataSize,
                                  hipMemcpyHostToDevice));
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        iCUDARet = hipGetLastError();
        if (iCUDARet != hipSuccess)
        {
            (void) fprintf(stderr, hipGetErrorString(iCUDARet));
        }

        /* Load the status data into the upper half for use in the next cycle */
        struct time_spead_heap* time_heap = (struct time_spead_heap*) vegas_databuf_data(db_in, curblock_in);        
        blk_info_cache.input(time_heap, index_in);
        
        // Zero out accumulators for 1st integration
        gpuCtx->zero_accumulator();
        printf("num_heaps per block = %d\n", index_in->num_heaps);
        // We don't do anything yet, we have just primed the pump ....
        return;
    }
    else
    {
        /* If this is not the first run, add copy the previous block to the low area
           for processing
         */
        CUDA_SAFE_CALL(hipMemcpy(&gpuCtx->_pc4Data_d[0],
                                  &gpuCtx->_pc4Data_d[iBlockInDataSize/sizeof(char4)],
                                  iBlockInDataSize,
                                  hipMemcpyDeviceToDevice));
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        iCUDARet = hipGetLastError();
        if (iCUDARet != hipSuccess)
        {
            (void) fprintf(stderr, hipGetErrorString(iCUDARet));
        }
                                
        /* Now add the new incoming data on the high side of the buffer */                                
        CUDA_SAFE_CALL(hipMemcpy(&gpuCtx->_pc4Data_d[iBlockInDataSize/sizeof(char4)],
                                  payload_addr_in,
                                  iBlockInDataSize,
                                  hipMemcpyHostToDevice));
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        iCUDARet = hipGetLastError();
        if (iCUDARet != hipSuccess)
        {
            (void) fprintf(stderr, hipGetErrorString(iCUDARet));
        }
                                
        /* copy the status bits and valid flags for all heaps to arrays separate
           from the index, so that it can be combined with the corresponding
           values from the previous block 
           Again, copy upper to lower half. 
        */
        struct time_spead_heap* time_heap = (struct time_spead_heap*) vegas_databuf_data(db_in, curblock_in);
        
        blk_info_cache.input(time_heap, index_in);
    }
    
    /* now begin processing the 'old' data in the lower half of the buffers */
    gpuCtx->_pc4DataRead_d = gpuCtx->_pc4Data_d;
    iProcData = 0;
    while (iBlockInDataSize > iProcData)  /* loop till (num_heaps * heap_size) of data is processed */
    {
        if (0 == pfb_count)
        {
            /* Check if all heaps necessary for this PFB are valid */
            if (!(blk_info_cache.is_valid(heap_in, num_in_heaps_per_pfb)))
            {
                /* Skip all heaps that go into this PFB if there is an invalid heap */
                iProcData += (VEGAS_NUM_TAPS * nsubband_x_nchan_csize);
                /* update the data read pointer */
                gpuCtx->_pc4DataRead_d += (VEGAS_NUM_TAPS * nsubband_x_nchan);
                if (iProcData >= iBlockInDataSize)
                {
                    break;
                }

                /* Calculate input heap addresses for the next round of processing */
                heap_in += num_in_heaps_per_pfb;
                fprintf(stderr, "Invalid data detected -- stepping to heap %d\n", heap_in);
                
                if (heap_in > 2 * MAX_HEAPS_PER_BLK)
                {
                    /* This is not supposed to happen (but may happen if odd number of pkts are dropped
                       right at the end of the buffer, so we therefore do not exit) */
                    (void) fprintf(stdout,
                                   "WARNING: Heap count %d exceeds available number of heaps %d!\n",
                                   heap_in,
                                   2 * MAX_HEAPS_PER_BLK);
                }
                continue;
            }
        }
        /* Perform polyphase filtering */
        DoPFB<<<gpuCtx->_dimGPFB, gpuCtx->_dimBPFB>>>(gpuCtx->_pc4DataRead_d,
                                                      gpuCtx->_pf4FFTIn_d,
                                                      gpuCtx->_pfPFBCoeff_d);
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        iCUDARet = hipGetLastError();
        if (iCUDARet != hipSuccess)
        {
            (void) fprintf(stdout,
                           "ERROR: File <%s>, Line %d: %s\n",
                           __FILE__,
                           __LINE__,
                           hipGetErrorString(iCUDARet));
            run = 0;
            break;
        }

        iRet = gpuCtx->do_fft();
        if (iRet != VEGAS_OK)
        {
            (void) fprintf(stdout, "ERROR: FFT failed!\n");
            run = 0;
            break;
        }
        // Check for 8 FFT cycles worth of data (the size of the PFB time window) for blanking.
        // Note that this check may access data in the upper half of the buffer (i.e the next block)
        gpuCtx->blanking_inputs(blk_info_cache.is_blanked(heap_in, num_in_heaps_per_pfb));
        
        ++g_iTotHeapOut; // unconditional spectrum counter
                                
        /* Accumulate power x, power y, stokes real and imag, if the blanking
           bit is not set */
        if (!(gpuCtx->blank_current_fft()))
        {
            iRet = gpuCtx->accumulate();
            if (iRet != VEGAS_OK)
            {
                (void) fprintf(stdout, "ERROR: Accumulation failed!\n");
                run = 0;
                break;
            }
            ++g_iSpecPerAcc;
            // record the first unblanked state in this accumulation sequence
            if (1 == g_iSpecPerAcc)
            {
                gpuCtx->_first_time_heap_in_accum_status_bits = blk_info_cache.status(heap_in);
                
                memcpy(&gpuCtx->_first_time_heap_in_accum,
                       &blk_info_cache._heap_hdr[heap_in],
                       sizeof(gpuCtx->_first_time_heap_in_accum));
                gpuCtx->_first_time_heap_mjd = blk_info_cache.mjd(heap_in);               
            }                    
        }
        
        if (g_iSpecPerAcc == acc_len || gpuCtx->needs_flush())
        {
            /* dump to buffer */
            // If no accumulations have occurred, then just clear the accumulator and start again.
            if (g_iSpecPerAcc > 0)
            {
                iRet = dump_to_buffer(db_out,             
                                      *curblock_out,
                                      g_iHeapOut,
                                      &gpuCtx->_first_time_heap_in_accum,
                                      g_iTotHeapOut,
                                      g_iSpecPerAcc,
                                      gpuCtx->_first_time_heap_mjd,
                                      gpuCtx->_first_time_heap_in_accum_status_bits);
            
                if (iRet != VEGAS_OK)
                {
                    (void) fprintf(stdout, "ERROR: Getting accumulated spectrum failed!\n");
                    run = 0;
                    break;
                }                                  
                ++g_iHeapOut;
            }
            else
            {
                // printf("Scanlength: GPU:asked to dump buffer but no accumulations present\n");
            }

            /* zero accumulators */
            gpuCtx->zero_accumulator();
            /* reset time */
            g_iSpecPerAcc = 0;
        }

        iProcData += nsubband_x_nchan_csize;
        /* update the data read pointer */
        gpuCtx->_pc4DataRead_d += (nsubband_x_nchan);

        /* Calculate input heap addresses for the next round of processing */
        heap_in += num_in_heaps_per_fft;

        /* if output block is full */
        if (g_iHeapOut == g_iMaxNumHeapOut)
        {
            /* Set the number of heaps written to this block */
            /* Mark output buffer as filled */
            vegas_databuf_set_filled(db_out, *curblock_out);

            /* Note current output block */
            /* NOTE: vegas_status_lock_safe() and vegas_status_unlock_safe() are macros
               that have been explicitly expanded here, due to compilation issues */
            //vegas_status_lock_safe(&st);
                pthread_cleanup_push((void (*) (void *))&vegas_status_unlock, (void *) &st);
                vegas_status_lock(&st);
            hputi4(st.buf, "PFBBLKOU", *curblock_out);
            //vegas_status_unlock_safe(&st);
                vegas_status_unlock(&st);
                pthread_cleanup_pop(0);

            /*  Wait for next output block */
            *curblock_out = (*curblock_out + 1) % db_out->n_block;
            while ((vegas_databuf_wait_free(db_out, *curblock_out)!=0) && run) {
                //vegas_status_lock_safe(&st);
                    pthread_cleanup_push((void (*)(void *))&vegas_status_unlock, (void *) &st);
                    vegas_status_lock(&st);

                hputs(st.buf, STATUS_KEY, "blocked");
                //vegas_status_unlock_safe(&st);
                    vegas_status_unlock(&st);
                    pthread_cleanup_pop(0);
            }

            g_iHeapOut = 0;

            hdr_out = vegas_databuf_header(db_out, *curblock_out);
            index_out = (struct databuf_index*)vegas_databuf_index(db_out, *curblock_out);
            index_out->num_heaps = 0;
            memcpy(hdr_out, vegas_databuf_header(db_in, curblock_in),
                    VEGAS_STATUS_SIZE);

            /* Set basic params in output index */
            index_out->heap_size = sizeof(struct freq_spead_heap) + (nsubband_x_nchan_fsize);
        }

        pfb_count = (pfb_count + 1) % VEGAS_NUM_TAPS;
    }

    return;
}

/* function that performs the FFT */
int GpuContext::do_fft()
{
    hipfftResult iCUFFTRet = HIPFFT_SUCCESS;
    hipError_t iCUDARet = hipSuccess;

    /* execute plan */
    iCUFFTRet = hipfftExecC2C(_stPlan,
                             (hipfftComplex*) _pf4FFTIn_d,
                             (hipfftComplex*) _pf4FFTOut_d,
                             HIPFFT_FORWARD);
    if (iCUFFTRet != HIPFFT_SUCCESS)
    {
        (void) fprintf(stderr, "ERROR: FFT failed!");
        iCUDARet = hipGetLastError();
        if (iCUDARet != hipSuccess)
        {
            (void) fprintf(stderr, hipGetErrorString(iCUDARet));
            run = 0;
            return VEGAS_ERR_GEN;
        }
    }
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    iCUDARet = hipGetLastError();
    if (iCUDARet != hipSuccess)
    {
        (void) fprintf(stderr, hipGetErrorString(iCUDARet));
        run = 0;
        return VEGAS_ERR_GEN;
    }

    return VEGAS_OK;
}

int GpuContext::accumulate()
{
    hipError_t iCUDARet = hipSuccess;

    Accumulate<<<_dimGAccum, _dimBAccum>>>(_pf4FFTOut_d,
                                           _pf4SumStokes_d);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    iCUDARet = hipGetLastError();
    if (iCUDARet != hipSuccess)
    {
        (void) fprintf(stderr, hipGetErrorString(iCUDARet));
        run = 0;
        return VEGAS_ERR_GEN;
    }

    return VEGAS_OK;
}

void GpuContext::zero_accumulator()
{
    hipError_t iCUDARet = hipSuccess;
    CUDA_SAFE_CALL(hipMemset(_pf4SumStokes_d,
                                       '\0',
                                       (_nsubband
                                       * _nchan
                                       * sizeof(float4))));
    CUDA_SAFE_CALL(hipDeviceSynchronize());                                       
    if (iCUDARet != hipSuccess)
    {
        (void) fprintf(stderr, hipGetErrorString(iCUDARet));
    }

    return;
}

int GpuContext::get_accumulated_spectrum_from_device(char *out)
{
    hipError_t iCUDARet = hipSuccess;
    // Cuda note: Device to host memcpy is always synchronous
    /* copy the negative frequencies out first */
    CUDASafeCall(hipMemcpy(out,
                            _pf4SumStokes_d + (_nsubband * _nchan / 2),
                            (_nsubband
                             * (_nchan / 2)
                             * sizeof(float4)),
                            hipMemcpyDeviceToHost));
    /* copy the positive frequencies out */
    CUDASafeCall(hipMemcpy(out + (_nsubband * (_nchan / 2) * sizeof(float4)),
                            _pf4SumStokes_d,
                            (_nsubband
                             * (_nchan / 2)
                             * sizeof(float4)),
                            hipMemcpyDeviceToHost));
    iCUDARet = hipGetLastError();
    if (iCUDARet != hipSuccess)
    {
        (void) fprintf(stderr, hipGetErrorString(iCUDARet));
    }

#ifdef DEBUG_ZERO_CHANNELS    
    // DEBUG check for near or zero channels X*X and Y*Y should never be =< 0.0                            
    int i, ndata, n_null = 0;
    ndata = _nsubband*_nchan;
    float4 *data = (float4 *)out;
    int first_bad = 0;
    
    for (i=0; i<ndata; ++i)
    {
        if (data[i].x <= 0.0 || data[i].y <= 0.0)
        {
            n_null++;
            first_bad=first_bad == 0 ? i : first_bad;
        }
    }
    if (n_null != 0)
    {
        printf("GPU: %d nil channels starting at %d\n", n_null, first_bad);
    }
#endif
    return VEGAS_OK;
}


void __CUDASafeCall(hipError_t iCUDARet,
                               const char* pcFile,
                               const int iLine,
                               void (*pCleanUp)(void))
{
    if (iCUDARet != hipSuccess)
    {
        (void) fprintf(stderr,
                       "ERROR: File <%s>, Line %d: %s\n",
                       pcFile,
                       iLine,
                       hipGetErrorString(iCUDARet));
        run = 0;
        return;
    }

    return;
}



/*
 * Frees up any allocated memory.
 */
extern "C"
void cleanup_gpu()
{
#if 0
    /* free memory */
    if (gpuCtx._pc4InBuf != NULL)
    {
        free(gpuCtx._pc4InBuf);
        gpuCtx._pc4InBuf = NULL;
    }
    if (gpuCtx._pc4Data_d != NULL)
    {
        (void) hipFree(gpuCtx._pc4Data_d);
        gpuCtx._pc4Data_d = NULL;
    }
    if (gpuCtx._pf4FFTIn_d != NULL)
    {
        (void) hipFree(gpuCtx._pf4FFTIn_d);
        gpuCtx._pf4FFTIn_d = NULL;
    }
    if (gpuCtx._pf4FFTOut_d != NULL)
    {
        (void) hipFree(gpuCtx._pf4FFTOut_d);
        gpuCtx._pf4FFTOut_d = NULL;
    }
    if (gpuCtx._pf4SumStokes_d != NULL)
    {
        (void) hipFree(gpuCtx._pf4SumStokes_d);
        gpuCtx._pf4SumStokes_d = NULL;
    }

    /* destroy plan */
    /* TODO: check if plan exists */
    if (gpuCtx._stPlan)
    {
        (void) hipfftDestroy(gpuCtx._stPlan);
        gpuCtx._stPlan = NULL;
    }
    printf("#################### GPU CONTEXT CLEANED UP ####################\n");
#endif 
    return;
}
