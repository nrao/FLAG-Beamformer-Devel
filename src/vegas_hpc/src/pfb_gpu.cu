#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <pthread.h>
#include <errno.h>
#include <hipfft/hipfft.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>

#include "fitshead.h"
#include "vegas_error.h"
#ifdef __cplusplus
extern "C" {
#endif
#include "vegas_status.h"
#include "vegas_databuf.h"
#ifdef __cplusplus
}
#endif
#include "vegas_defines.h"
#include "pfb_gpu.h"
#include "pfb_gpu_kernels.h"
#include "spead_heap.h"

#define STATUS_KEY "GPUSTAT"

/* ASSUMPTIONS: 1. All blocks contain the same number of heaps. */

extern int run;

/**
 * Global variables: maybe move this to a struct that is passed to each function?
 */
 
class GpuContext
{
public:
    // stuff associated with gpu
    GpuContext();
    GpuContext(GpuContext *, int nchan, int nsubbands, int inblocksz, int outblksz);
    hipfftHandle _stPlan;
    float4* _pf4FFTIn_d;
    float4* _pf4FFTOut_d;
    char4*  _pc4InBuf;
    char4*  _pc4Data_d;              /* raw data starting address */
    char4*  _pc4DataRead_d;          /* raw data read pointer */
    dim3    _dimBPFB;
    dim3    _dimGPFB;
    dim3    _dimBAccum;
    dim3    _dimGAccum;
    float * _pfPFBCoeff;
    float * _pfPFBCoeff_d;
    float4* _pf4SumStokes_d;
    
    int     _nchan;
    int     _nsubband;
    int     _in_block_size;
    int     _out_block_size;
    int     _init_status;
    
    int fft_in_stride()  { return 2*_nsubband; };
    int fft_out_stride() { return 2*_nsubband; };
    int fft_batch()      { return 2*_nsubband; };
    int accumulate();
    int do_fft();
    void zero_accumulator();
    int get_accumulated_spectrum_from_device(char *h_out);
    int init_resources();
    void release_resources();
    int init_status()    { return _init_status; }
    bool verify_setup(int num_subbands, int num_chans, 
                      int input_block_sz, int output_block_sz);

};

GpuContext::GpuContext() :
        _pf4FFTIn_d(0),
        _pf4FFTOut_d(0),
        _pc4InBuf(0),
        _pc4Data_d(0),
        _pc4DataRead_d(0),
        _dimBPFB(),
        _dimGPFB(),
        _dimBAccum(),
        _dimGAccum(),
        _pfPFBCoeff(0),
        _pfPFBCoeff_d(0),
        _pf4SumStokes_d(0),
        _nchan(0),
        _nsubband(0)
{
    memset(&_stPlan, 0, sizeof(_stPlan));    
}

GpuContext::GpuContext(GpuContext *p, int nsubband, int nchan, int in_blok_siz, int out_blok_siz)
{

    if (p != 0)
    {
        // Move resources from p into this object and null out p's reference
        _pf4FFTIn_d    = p->_pf4FFTIn_d;     p->_pf4FFTIn_d = 0;
        _pf4FFTOut_d   = p->_pf4FFTOut_d;    p->_pf4FFTOut_d = 0;
        _pc4InBuf      = p->_pc4InBuf;       p->_pc4InBuf = 0;
        _pc4Data_d     = p->_pc4Data_d;      p->_pc4Data_d = 0;
        _pc4DataRead_d = p->_pc4DataRead_d;  p->_pc4DataRead_d = 0;
        _dimBPFB       = p->_dimBPFB;
        _dimGPFB       = p->_dimGPFB;
        _dimBAccum     = p->_dimBAccum;
        _pfPFBCoeff    = p->_pfPFBCoeff;     p->_pfPFBCoeff = 0;
        _pfPFBCoeff_d  = p->_pfPFBCoeff_d;   p->_pfPFBCoeff_d = 0;
        _pf4SumStokes_d= p->_pf4SumStokes_d; p->_pf4SumStokes_d = 0;
        _stPlan        = p->_stPlan;         p->_stPlan = 0;    
    }
    else
    {
        _pf4FFTIn_d    = 0;
        _pf4FFTOut_d   = 0;
        _pc4InBuf      = 0;
        _pc4Data_d     = 0;
        _pc4DataRead_d = 0;
        _pfPFBCoeff    = 0;
        _pfPFBCoeff_d  = 0;
        _pf4SumStokes_d= 0;
        _stPlan        = 0;
        _nchan         = 0;
        _nsubband      = 0;    
    }
    
    if (_nsubband == nsubband &&
        _nchan    == nchan &&
        _in_block_size == in_blok_siz &&
        _out_block_size == out_blok_siz)
    {
        // We should be done
        return;
    }
    else
    {
        release_resources();
       
        // Now allocate new resources for the new configuration
        _nsubband = nsubband;
        _nchan = nchan;
        _in_block_size = in_blok_siz;
        _out_block_size = out_blok_siz;
        init_resources();
    }
}

bool
GpuContext::verify_setup(int nsubband, int nchan, int in_block_size, int out_block_size)
{
    // Does the setup match?
    if (_nsubband == nsubband &&
        _nchan    == nchan &&
        _in_block_size  == in_block_size &&
        _out_block_size == out_block_size)
        return true;
    return false;
}



// Make the damn object global until we complete refactoring ...
GpuContext *gpuCtx = 0;

static size_t g_buf_out_block_size;
static unsigned int g_iPrevBlankingState = FALSE;
static int g_iTotHeapOut = 0;
static int g_iMaxNumHeapOut = 0;
static int g_iHeapOut = 0;

/* these arrays need to be only a little longer than MAX_HEAPS_PER_BLK, but
   since we don't know the exact length, just allocate twice that value */
static unsigned int g_auiStatusBits[2*MAX_HEAPS_PER_BLK] = {0};
static unsigned int g_auiHeapValid[2*MAX_HEAPS_PER_BLK] = {0};
static int g_iSpecPerAcc = 0;

void __CUDASafeCall(hipError_t iCUDARet,
                               const char* pcFile,
                               const int iLine,
                               void (*pCleanUp)(void));

#define CUDASafeCall(iRet)   __CUDASafeCall(iRet,       \
                                                                  __FILE__,   \
                                                                  __LINE__,   \
                                                                  &cleanup_gpu)

#define CUDA_SAFE_CALL(call) \
do { \
    hipError_t err = call; \
    if (hipSuccess != err) { \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.", \
                 __FILE__, __LINE__, hipGetErrorString(err) ); \
        exit(EXIT_FAILURE); \
    } \
} while (0)


extern "C"
int init_cuda_context(int subbands, int chans, int inBlokSz, int outBlokSz)
{
    int iDevCount = 0;

    /* since CUDASafeCall() calls hipGetErrorString(),
       it should not be used here - will cause crash if no CUDA device is
       found */
    (void) hipGetDeviceCount(&iDevCount);
    if (0 == iDevCount)
    {
        (void) fprintf(stderr, "ERROR: No CUDA-capable device found!\n");
        return EXIT_FAILURE;
    }        

    if (subbands == 0 || chans == 0)
    {

        /* just use the first device */
        printf("pfb_gpu.cu: CUDA_SAFE_CALL(hipSetDevice(0))\n");
        CUDA_SAFE_CALL(hipSetDevice(0));
        printf("pfb_gpu.cu: CUDA_SAFE_CALL(hipFree(0)\n");
        CUDA_SAFE_CALL(hipFree(0));
        printf("#################### GPU CONTEXT INITIALIZED ####################\n");
    }
    else
    {
        // Create a new mode specific set of resources.
        GpuContext *newctx, *oldctx;
        oldctx = gpuCtx;
        newctx = new GpuContext(oldctx, subbands, chans, inBlokSz, outBlokSz);
        gpuCtx = newctx;
        delete oldctx;
    }
    return EXIT_SUCCESS;
}


/* Initialize all necessary memory, etc for doing PFB
 * at the given params.
 */
extern "C" 
int reset_state(size_t input_block_sz, size_t output_block_sz, int num_subbands, int num_chans)
{
    g_iPrevBlankingState = TRUE;
    g_iTotHeapOut = 0;
    g_iHeapOut = 0;
    g_iSpecPerAcc = 0;
    
    // Now verify we have the right setup
    if (gpuCtx == 0 || 
        true != gpuCtx->verify_setup(num_subbands, num_chans, input_block_sz, output_block_sz))
    {
        printf("Error: runtime and pre-init GPU setups didn't match\n");
        // For backward compatibility we try to initialize here.
        if (EXIT_SUCCESS != init_cuda_context(num_subbands, num_chans, input_block_sz, output_block_sz))
            return EXIT_FAILURE;
    }
    return EXIT_SUCCESS;
}

int GpuContext::init_resources()
{
    int iDevCount = 0;
    hipDeviceProp_t stDevProp = {0};
    hipfftResult iCUFFTRet = HIPFFT_SUCCESS;
    int iRet = EXIT_SUCCESS;
    int iMaxThreadsPerBlock = 0;
    size_t buf_in_block_size;
    int iFileCoeff = 0;
    char acFileCoeff[256] = {0};

    
    buf_in_block_size    = _in_block_size;
    g_buf_out_block_size = _out_block_size;

    /* since CUDASafeCall() calls hipGetErrorString(),
       it should not be used here - will cause crash if no CUDA device is
       found */
    (void) hipGetDeviceCount(&iDevCount);
    if (0 == iDevCount)
    {
        (void) fprintf(stderr, "ERROR: No CUDA-capable device found!\n");
        run = 0;
        return EXIT_FAILURE;
    }

    /* just use the first device */
    printf("pfb_gpu.cu: CUDA_SAFE_CALL(hipSetDevice(0))\n");
    CUDA_SAFE_CALL(hipSetDevice(0));

    CUDA_SAFE_CALL(hipGetDeviceProperties(&stDevProp, 0));
    iMaxThreadsPerBlock = stDevProp.maxThreadsPerBlock;
    printf("pfb_gpu.cu: iMaxThreadsPerBlock = %i\n", iMaxThreadsPerBlock);

    _pfPFBCoeff = (float *) malloc(_nsubband
                                   * VEGAS_NUM_TAPS
                                   * _nchan
                                   * sizeof(float));
    if (NULL == _pfPFBCoeff)
    {
        (void) fprintf(stderr,
                       "ERROR: GpuContext Memory allocation failed! %s.\n",
                       strerror(errno));
        return EXIT_FAILURE;
    }

    /* allocate memory for the filter coefficient array on the device */

    printf("pfb_gpu.cu: before CUDA_SAFE_CALL(hipFree(0))\n");
    CUDA_SAFE_CALL(hipFree(0));
    printf("pfb_gpu.cu: after CUDA_SAFE_CALL(hipFree(0))\n");

    printf("pfb_gpu.cu:  before CUDA_SAFE_CALL(hipMalloc((void...\n");
    printf("subbands=%i, taps=%i, nchan=%i, floatsize=%i\n", _nsubband, VEGAS_NUM_TAPS, _nchan, sizeof(float));
    CUDA_SAFE_CALL(hipMalloc((void **) &_pfPFBCoeff_d,
                                       _nsubband
                                       * VEGAS_NUM_TAPS
                                       * _nchan
                                       * sizeof(float)));
    printf("pfb_gpu.cu:  CUDA_SAFE_CALL(hipMalloc((void...\n");

    /* read filter coefficients */
    /* Locate the coefficient directory.  This searches for the configuration
     * directory in one of YGOR_TELESCOPE, VEGAS_DIR or CONFIG_DIR
       If none of the environment variables above are specified
       then we punt and use the current working directory.
     */
     
     
    char *ygor_root = getenv("YGOR_TELESCOPE");
    char *vdir_root = getenv("VEGAS_DIR");
    char *config_root = getenv("CONFIG_DIR");
    char conf_dir_root[128];
    
    if (ygor_root)
    {
        /* Use YGOR_TELESCOPE if available */
        snprintf(conf_dir_root, sizeof(conf_dir_root), "%s/etc/config", ygor_root);
    }
    else if (config_root)
    {
        snprintf(conf_dir_root, sizeof(conf_dir_root), "%s", config_root);
    }
    else if (vdir_root)
    {
        snprintf(conf_dir_root, sizeof(conf_dir_root), "%s", vdir_root);
    }
    else
    {
        snprintf(conf_dir_root, sizeof(conf_dir_root), ".");
    }

    /* build file name */
    (void) snprintf(acFileCoeff, sizeof(acFileCoeff),
                   "%s/%s_%s_%d_%d_%d%s",
                   conf_dir_root,                   
                   FILE_COEFF_PREFIX,
                   FILE_COEFF_DATATYPE,
                   VEGAS_NUM_TAPS,
                   _nchan,
                   _nsubband,
                   FILE_COEFF_SUFFIX);

    iFileCoeff = open(acFileCoeff, O_RDONLY);
    if (iFileCoeff < EXIT_SUCCESS)
    {
        (void) fprintf(stderr,
                       "ERROR: Opening filter coefficients file %s "
                       "failed! %s.\n",
                       acFileCoeff,
                       strerror(errno));
        return EXIT_FAILURE;
    }

    iRet = read(iFileCoeff,
                _pfPFBCoeff,
                _nsubband * VEGAS_NUM_TAPS * _nchan * sizeof(float));
    if (iRet != (_nsubband * VEGAS_NUM_TAPS * _nchan * sizeof(float)))
    {
        (void) fprintf(stderr,
                       "ERROR: Reading filter coefficients failed! %s.\n",
                       strerror(errno));
        return EXIT_FAILURE;
    }
    (void) close(iFileCoeff);

    /* copy filter coefficients to the device */
    CUDA_SAFE_CALL(hipMemcpy(_pfPFBCoeff_d,
                              _pfPFBCoeff,
                              _nsubband * VEGAS_NUM_TAPS * _nchan * sizeof(float),
                              hipMemcpyHostToDevice));

    /* allocate memory for data array - 32MB is the block size for the VEGAS
       input buffer, allocate 32MB + space for (VEGAS_NUM_TAPS - 1) blocks of
       data
       NOTE: the actual data in a 32MB block will be only
       (num_heaps * heap_size), but since we don't know that value until data
       starts flowing, allocate the maximum possible size */
    CUDA_SAFE_CALL(hipMalloc((void **) &_pc4Data_d,
                                       (buf_in_block_size
                                        + ((VEGAS_NUM_TAPS - 1)
                                           * _nsubband
                                           * _nchan
                                           * sizeof(char4)))));
    printf("pfb_gpu.cu: CUDA_SAFE_CALL(hipMalloc((void...)\n");
    _pc4DataRead_d = _pc4Data_d;
    
    /* calculate kernel parameters */
    /* ASSUMPTION: gpuCtx._nchan >= iMaxThreadsPerBlock */
    _dimBPFB.x =   iMaxThreadsPerBlock;
    _dimBAccum.x = iMaxThreadsPerBlock;
    _dimGPFB.x =   (_nsubband * _nchan) / iMaxThreadsPerBlock;
    _dimGAccum.x = (_nsubband * _nchan) / iMaxThreadsPerBlock;

    CUDA_SAFE_CALL(hipMalloc((void **) &_pf4FFTIn_d,
                                 _nsubband * _nchan * sizeof(float4)));
    CUDA_SAFE_CALL(hipMalloc((void **) &_pf4FFTOut_d,
                                 _nsubband * _nchan * sizeof(float4)));
    CUDA_SAFE_CALL(hipMalloc((void **) &_pf4SumStokes_d,
                                 _nsubband * _nchan * sizeof(float4)));
    CUDA_SAFE_CALL(hipMemset(_pf4SumStokes_d,
                              0,
                              _nsubband * _nchan * sizeof(float4)));

    printf("pfb_gpu.cu: 4 CUDA_SAFE_CALL(hipMalloc...) calls\n");

    /* create plan */
    iCUFFTRet = hipfftPlanMany(&_stPlan,
                              FFTPLAN_RANK,
                              &_nchan,
                              &_nchan,
                              fft_in_stride(),
                              FFTPLAN_IDIST,
                              &_nchan,
                              fft_in_stride(),
                              FFTPLAN_ODIST,
                              HIPFFT_C2C,
                              fft_batch() );
    if (iCUFFTRet != HIPFFT_SUCCESS)
    {
        (void) fprintf(stderr, "ERROR: Plan creation failed!\n");
        run = 0;
        return EXIT_FAILURE;
    }
    printf("GPU resources resized for %d subbands and %d channels\n", _nsubband, _nchan);
    printf("#################### GPU RE-INIT COMPLETE ####################\n");
    return EXIT_SUCCESS;
}

void
GpuContext::release_resources()
{
    // Free existing resources
    printf("Releasing GPU resources \n");
    
    if (_pc4InBuf != NULL)
    {
        free(_pc4InBuf);
        _pc4InBuf = NULL;
    }
    if (_pc4Data_d != NULL)
    {
        (void) hipFree(_pc4Data_d);
        _pc4Data_d = NULL;
    }
    if (_pf4FFTIn_d != NULL)
    {
        (void) hipFree(_pf4FFTIn_d);
        _pf4FFTIn_d = NULL;
    }
    if (_pf4FFTOut_d != NULL)
    {
        (void) hipFree(_pf4FFTOut_d);
        _pf4FFTOut_d = NULL;
    }
    if (_pf4SumStokes_d != NULL)
    {
        (void) hipFree(_pf4SumStokes_d);
        _pf4SumStokes_d = NULL;
    }

    /* destroy plan */
    /* TODO: check if plan exists */
    if (_stPlan)
    {
        (void) hipfftDestroy(_stPlan);
        _stPlan = NULL;
    }
}

struct freq_spead_heap *
frequency_heap(struct vegas_databuf *db, int iblk, int iHeap)
{
    struct freq_spead_heap *freq_heap;
    char *ptr;
    ptr = (char *)(vegas_databuf_data(db, iblk) +
                   sizeof(struct freq_spead_heap) * iHeap);
    freq_heap = (struct freq_spead_heap*)ptr;                
    return freq_heap;
}

/* dump to buffer */
int dump_to_buffer(struct vegas_databuf *db_out,         // Output databuffer
                   int curblk_out,                       // Current output block
                   int iHeapOut,                         // output frequency heap number in current block
                   struct time_spead_heap *firsttimeheap,// first time sample of input 
                   int iTotHeapOut,                      // spectrum number/counter
                   int iSpecPerAcc,                      // GPU accumulations in this heap
                   double heap_mjd )                     // MJD from index_input
{
    struct freq_spead_heap *freq_heap_out;
    char * payload_addr_out;
    struct databuf_index *index_out;
    int rtn;
    
    freq_heap_out = frequency_heap(db_out, curblk_out, iHeapOut);
    index_out = (struct databuf_index*)vegas_databuf_index(db_out, curblk_out);
    
    payload_addr_out = (char*)(vegas_databuf_data(db_out, curblk_out) +
                        sizeof(struct freq_spead_heap) * MAX_HEAPS_PER_BLK +
                        (index_out->heap_size - sizeof(struct freq_spead_heap)) * iHeapOut);                        

    /* Write new heap header fields */
    freq_heap_out->time_cntr_id = 0x20;
    freq_heap_out->time_cntr_top8 = firsttimeheap->time_cntr_top8;
    freq_heap_out->time_cntr = firsttimeheap->time_cntr;
    freq_heap_out->spectrum_cntr_id = 0x21;
    freq_heap_out->spectrum_cntr = iTotHeapOut;
    freq_heap_out->integ_size_id = 0x22;
    freq_heap_out->integ_size = iSpecPerAcc;
    freq_heap_out->mode_id = 0x23;
    freq_heap_out->mode = firsttimeheap->mode;
    freq_heap_out->status_bits_id = 0x24;
    freq_heap_out->status_bits = firsttimeheap->status_bits;
    freq_heap_out->payload_data_off_addr_mode = 0;
    freq_heap_out->payload_data_off_id = 0x25;
    freq_heap_out->payload_data_off = 0;
    
/////DEBUG
    memset(firsttimeheap, 0, sizeof(struct time_spead_heap));

    /* Update output index */
    index_out->cpu_gpu_buf[iHeapOut].heap_valid = 1;
    index_out->cpu_gpu_buf[iHeapOut].heap_cntr = iTotHeapOut;
    index_out->cpu_gpu_buf[iHeapOut].heap_rcvd_mjd = heap_mjd;

    /* copy out GPU data into buffer */
    rtn = gpuCtx->get_accumulated_spectrum_from_device(payload_addr_out);
    index_out->num_heaps += (rtn == VEGAS_OK ? 1 : 0);
    return rtn;
}

/* Actually do the PFB by calling CUDA kernels */
extern "C"
void do_pfb(struct vegas_databuf *db_in,
            int curblock_in,
            struct vegas_databuf *db_out,
            int *curblock_out,
            int first,
            struct vegas_status st,
            int acc_len)
{
    /* Declare local variables */
    char *hdr_out = NULL;
    struct databuf_index *index_in = NULL;
    struct databuf_index *index_out = NULL;
    int heap_in = 0;
    char *heap_addr_in = NULL;
    struct time_spead_heap first_time_heap_in_accum;
    int iProcData = 0;
    hipError_t iCUDARet = hipSuccess;
    int iRet = VEGAS_OK;
    char* payload_addr_in = NULL;
    int num_in_heaps_per_proc = 0;
    int pfb_count = 0;
    int num_in_heaps_gpu_buffer = 0;
    int num_in_heaps_tail = 0;
    int i = 0;
    int iBlockInDataSize;
    double first_time_heap_mjd;

    /* Setup input and first output data block stuff */
    index_in = (struct databuf_index*)vegas_databuf_index(db_in, curblock_in);
    /* Get the number of heaps per block of data that will be processed by the GPU */
    num_in_heaps_per_proc = (gpuCtx->_nsubband * gpuCtx->_nchan * sizeof(char4)) / (index_in->heap_size - sizeof(struct time_spead_heap));
    iBlockInDataSize = (index_in->num_heaps * index_in->heap_size) - (index_in->num_heaps * sizeof(struct time_spead_heap));

    num_in_heaps_tail = (((VEGAS_NUM_TAPS - 1) * gpuCtx->_nsubband * gpuCtx->_nchan * sizeof(char4))
                         / (index_in->heap_size - sizeof(struct time_spead_heap)));
    num_in_heaps_gpu_buffer = index_in->num_heaps + num_in_heaps_tail;

    /* Calculate the maximum number of output heaps per block */
    g_iMaxNumHeapOut = (g_buf_out_block_size - (sizeof(struct time_spead_heap) * MAX_HEAPS_PER_BLK)) / (gpuCtx->_nsubband * gpuCtx->_nchan * sizeof(float4));

    hdr_out = vegas_databuf_header(db_out, *curblock_out);
    index_out = (struct databuf_index*)vegas_databuf_index(db_out, *curblock_out);
    // index_out->num_heaps = 0;
    memcpy(hdr_out, vegas_databuf_header(db_in, curblock_in),
            VEGAS_STATUS_SIZE);

    /* Set basic params in output index */
    index_out->heap_size = sizeof(struct freq_spead_heap) + (gpuCtx->_nsubband * gpuCtx->_nchan * sizeof(float4));
    /* Read in heap from buffer */
    heap_addr_in = (char*)(vegas_databuf_data(db_in, curblock_in) +
                        sizeof(struct time_spead_heap) * heap_in);
    // first_time_heap_in_accum = (struct time_spead_heap*)(heap_addr_in);
    memcpy(&first_time_heap_in_accum, heap_addr_in, sizeof(first_time_heap_in_accum));
    first_time_heap_mjd = index_in->cpu_gpu_buf[heap_in].heap_rcvd_mjd;
    /* Here, the payload_addr_in is the start of the contiguous block of data that will be
       copied to the GPU (heap_in = 0) */
    payload_addr_in = (char*)(vegas_databuf_data(db_in, curblock_in) +
                        sizeof(struct time_spead_heap) * MAX_HEAPS_PER_BLK +
                        (index_in->heap_size - sizeof(struct time_spead_heap)) * heap_in );

    /* Copy data block to GPU */
    if (first)
    {
        // bloksz replaces a calculated value which caused the check below to fail
        // in the presence of dropped packets. We used the blocksize from the data buffer
        // instead here to get things going. Not sure how dropped data at start of
        // scan should be treated.
        int bloksz = db_in->block_size;
        /* Sanity check for the first iteration */
        if ((bloksz % (gpuCtx->_nsubband * gpuCtx->_nchan * sizeof(char4))) != 0)
        {
            (void) fprintf(stderr, "ERROR: Data size mismatch! BlockInDataSize=%d NumSubBands=%d nchan=%d\n",
                                    bloksz, gpuCtx->_nsubband, gpuCtx->_nchan);
            run = 0;
            return;
        }
        CUDA_SAFE_CALL(hipMemcpy(gpuCtx->_pc4Data_d,
                                payload_addr_in,
                                bloksz,
                                hipMemcpyHostToDevice));
        /* duplicate the last (VEGAS_NUM_TAPS - 1) segm                payload_addr_out = (char*)(vegas_databuf_data(db_out, g_iPFBCurBlockOut) +
                                sizeof(struct freq_spead_heap) * MAX_HEAPS_PER_BLK +
                                (index_out->heap_size - sizeof(struct freq_spead_heap)) * g_iHeapOut);
ents at the end for
           the next iteration */
        CUDA_SAFE_CALL(hipMemcpy(gpuCtx->_pc4Data_d + (bloksz / sizeof(char4)),
                                gpuCtx->_pc4Data_d + (bloksz / sizeof(char4)) - ((VEGAS_NUM_TAPS - 1) * gpuCtx->_nsubband * gpuCtx->_nchan),
                                ((VEGAS_NUM_TAPS - 1) * gpuCtx->_nsubband * gpuCtx->_nchan * sizeof(char4)),
                                hipMemcpyDeviceToDevice));

        /* copy the status bits and valid flags for all heaps to arrays separate
           from the index, so that it can be combined with the corresponding
           values from the previous block */
        struct time_spead_heap* time_heap = (struct time_spead_heap*) vegas_databuf_data(db_in, curblock_in);
        for (i = 0; i < index_in->num_heaps; ++i)
        {
            g_auiStatusBits[i] = time_heap->status_bits;
            g_auiHeapValid[i] = index_in->cpu_gpu_buf[i].heap_valid;
            ++time_heap;
        }
        /* duplicate the last (VEGAS_NUM_TAPS - 1) segments at the end for the
           next iteration */
        for ( ; i < index_in->num_heaps + num_in_heaps_tail; ++i)
        {
            g_auiStatusBits[i] = g_auiStatusBits[i-num_in_heaps_tail];
            g_auiHeapValid[i] = g_auiHeapValid[i-num_in_heaps_tail];
        }
    }
    else
    {
        /* If this is not the first run, need to handle block boundary, while doing the PFB */
        CUDA_SAFE_CALL(hipMemcpy(gpuCtx->_pc4Data_d,
                                gpuCtx->_pc4Data_d + (iBlockInDataSize / sizeof(char4)),
                                ((VEGAS_NUM_TAPS - 1) * gpuCtx->_nsubband * gpuCtx->_nchan * sizeof(char4)),
                                hipMemcpyDeviceToDevice));
        CUDA_SAFE_CALL(hipMemcpy(gpuCtx->_pc4Data_d + ((VEGAS_NUM_TAPS - 1) * gpuCtx->_nsubband * gpuCtx->_nchan),
                                payload_addr_in,
                                iBlockInDataSize,
                                hipMemcpyHostToDevice));
        /* copy the status bits and valid flags for all heaps to arrays separate
           from the index, so that it can be combined with the corresponding
           values from the previous block */
        for (i = 0; i < num_in_heaps_tail; ++i)
        {
            g_auiStatusBits[i] = g_auiStatusBits[index_in->num_heaps+i];
            g_auiHeapValid[i] = g_auiHeapValid[index_in->num_heaps+i];
        }
        struct time_spead_heap* time_heap = (struct time_spead_heap*) vegas_databuf_data(db_in, curblock_in);
        for ( ; i < num_in_heaps_tail + index_in->num_heaps; ++i)
        {
            g_auiStatusBits[i] = time_heap->status_bits;
            g_auiHeapValid[i] = index_in->cpu_gpu_buf[i-num_in_heaps_tail].heap_valid;
            ++time_heap;
        }
    }

    gpuCtx->_pc4DataRead_d = gpuCtx->_pc4Data_d;
    iProcData = 0;
    while (iBlockInDataSize > iProcData)  /* loop till (num_heaps * heap_size) of data is processed */
    {
        if (0 == pfb_count)
        {
            /* Check if all heaps necessary for this PFB are valid */
            if (!(is_valid(heap_in, (VEGAS_NUM_TAPS * num_in_heaps_per_proc))))
            {
                /* Skip all heaps that go into this PFB if there is an invalid heap */
                iProcData += (VEGAS_NUM_TAPS * gpuCtx->_nsubband * gpuCtx->_nchan * sizeof(char4));
                /* update the data read pointer */
                gpuCtx->_pc4DataRead_d += (VEGAS_NUM_TAPS * gpuCtx->_nsubband * gpuCtx->_nchan);
                if (iProcData >= iBlockInDataSize)
                {
                    break;
                }

                /* Calculate input heap addresses for the next round of processing */
                heap_in += (VEGAS_NUM_TAPS * num_in_heaps_per_proc);
                if (heap_in > num_in_heaps_gpu_buffer)
                {
                    /* This is not supposed to happen (but may happen if odd number of pkts are dropped
                       right at the end of the buffer, so we therefore do not exit) */
                    (void) fprintf(stdout,
                                   "WARNING: Heap count %d exceeds available number of heaps %d!\n",
                                   heap_in,
                                   num_in_heaps_gpu_buffer);
                }
                heap_addr_in = (char*)(vegas_databuf_data(db_in, curblock_in) +
                                    sizeof(struct time_spead_heap) * heap_in);
                // The check above indicates this is an invalid heap, therefore we don't treat
                // it as the 'first heap'                    
                // memcpy(&first_time_heap_in_accum, heap_addr_in, sizeof(first_time_heap_in_accum));
                continue;
            }
        }

        /* Perform polyphase filtering */
        DoPFB<<<gpuCtx->_dimGPFB, gpuCtx->_dimBPFB>>>(gpuCtx->_pc4DataRead_d,
                                                    gpuCtx->_pf4FFTIn_d,
                                                    gpuCtx->_pfPFBCoeff_d);
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        iCUDARet = hipGetLastError();
        if (iCUDARet != hipSuccess)
        {
            (void) fprintf(stdout,
                           "ERROR: File <%s>, Line %d: %s\n",
                           __FILE__,
                           __LINE__,
                           hipGetErrorString(iCUDARet));
            run = 0;
            break;
        }

        iRet = gpuCtx->do_fft();
        if (iRet != VEGAS_OK)
        {
            (void) fprintf(stdout, "ERROR: FFT failed!\n");
            run = 0;
            break;
        }

        /* Accumulate power x, power y, stokes real and imag, if the blanking
           bit is not set */
        if (!(is_blanked(heap_in, num_in_heaps_per_proc)))
        {
            iRet = gpuCtx->accumulate();
            if (iRet != VEGAS_OK)
            {
                (void) fprintf(stdout, "ERROR: Accumulation failed!\n");
                run = 0;
                break;
            }
            ++g_iSpecPerAcc;
            g_iPrevBlankingState = FALSE;
        }
        else
        {
            /* state just changed */
            if (FALSE == g_iPrevBlankingState)
            {
                /* dump to buffer */
                iRet = dump_to_buffer(db_out,             // Output databuffer
                                      *curblock_out,                       // Current output block
                                      g_iHeapOut,                         // output frequency heap number in current block
                                      &first_time_heap_in_accum,// first time sample of input 
                                      g_iTotHeapOut,                      // spectrum number/counter
                                      g_iSpecPerAcc,                      // GPU accumulations in this heap
                                      first_time_heap_mjd); // MJD from index_input
                                                                       
                if (iRet != VEGAS_OK)
                {
                    (void) fprintf(stdout, "ERROR: Getting accumulated spectrum failed (blank state changed)!\n");
                    run = 0;
                    break;
                }

                ++g_iHeapOut;
                ++g_iTotHeapOut;

                /* zero accumulators */
                gpuCtx->zero_accumulator();
                /* reset time */
                g_iSpecPerAcc = 0;
                g_iPrevBlankingState = TRUE;
            }
        }
        if (g_iSpecPerAcc == acc_len)
        {
            /* dump to buffer */
            iRet = dump_to_buffer(db_out,             
                                  *curblock_out,
                                  g_iHeapOut,
                                  &first_time_heap_in_accum,
                                  g_iTotHeapOut,
                                  g_iSpecPerAcc,
                                  first_time_heap_mjd);
                                  
            if (iRet != VEGAS_OK)
            {
                (void) fprintf(stdout, "ERROR: Getting accumulated spectrum failed!\n");
                run = 0;
                break;
            }                                  
            ++g_iHeapOut;
            ++g_iTotHeapOut;

            /* zero accumulators */
            gpuCtx->zero_accumulator();
            /* reset time */
            g_iSpecPerAcc = 0;
        }

        iProcData += (gpuCtx->_nsubband * gpuCtx->_nchan * sizeof(char4));
        /* update the data read pointer */
        gpuCtx->_pc4DataRead_d += (gpuCtx->_nsubband * gpuCtx->_nchan);

        /* Calculate input heap addresses for the next round of processing */
        heap_in += num_in_heaps_per_proc;
        heap_addr_in = (char*)(vegas_databuf_data(db_in, curblock_in) +
                            sizeof(struct time_spead_heap) * heap_in);
        if (0 == g_iSpecPerAcc)
        {
            // first_time_heap_in_accum = (struct time_spead_heap*)(heap_addr_in);
            memcpy(&first_time_heap_in_accum, heap_addr_in, sizeof(first_time_heap_in_accum));
            first_time_heap_mjd = index_in->cpu_gpu_buf[heap_in].heap_rcvd_mjd;
        }

        /* if output block is full */
        if (g_iHeapOut == g_iMaxNumHeapOut)
        {
            /* Set the number of heaps written to this block */
            // JJB index_out->num_heaps = g_iHeapOut;
            // printf("gpu filled num_heaps=%d snum=%d\n", index_out->num_heaps, g_iTotHeapOut);

            /* Mark output buffer as filled */
            vegas_databuf_set_filled(db_out, *curblock_out);

            // printf("Debug: vegas_pfb_thread going to next output block\n");

            /* Note current output block */
            /* NOTE: vegas_status_lock_safe() and vegas_status_unlock_safe() are macros
               that have been explicitly expanded here, due to compilation issues */
            //vegas_status_lock_safe(&st);
                pthread_cleanup_push((void (*) (void *))&vegas_status_unlock, (void *) &st);
                vegas_status_lock(&st);
            hputi4(st.buf, "PFBBLKOU", *curblock_out);
            //vegas_status_unlock_safe(&st);
                vegas_status_unlock(&st);
                pthread_cleanup_pop(0);

            /*  Wait for next output block */
            *curblock_out = (*curblock_out + 1) % db_out->n_block;
            while ((vegas_databuf_wait_free(db_out, *curblock_out)!=0) && run) {
                //vegas_status_lock_safe(&st);
                    pthread_cleanup_push((void (*)(void *))&vegas_status_unlock, (void *) &st);
                    vegas_status_lock(&st);

                hputs(st.buf, STATUS_KEY, "blocked");
                //vegas_status_unlock_safe(&st);
                    vegas_status_unlock(&st);
                    pthread_cleanup_pop(0);
            }

            g_iHeapOut = 0;

            hdr_out = vegas_databuf_header(db_out, *curblock_out);
            index_out = (struct databuf_index*)vegas_databuf_index(db_out, *curblock_out);
            index_out->num_heaps = 0;
            memcpy(hdr_out, vegas_databuf_header(db_in, curblock_in),
                    VEGAS_STATUS_SIZE);

            /* Set basic params in output index */
            index_out->heap_size = sizeof(struct freq_spead_heap) + (gpuCtx->_nsubband * gpuCtx->_nchan * sizeof(float4));
        }

        pfb_count = (pfb_count + 1) % VEGAS_NUM_TAPS;
    }

    return;
}

/* function that performs the FFT */
int GpuContext::do_fft()
{
    hipfftResult iCUFFTRet = HIPFFT_SUCCESS;

    /* execute plan */
    iCUFFTRet = hipfftExecC2C(_stPlan,
                             (hipfftComplex*) _pf4FFTIn_d,
                             (hipfftComplex*) _pf4FFTOut_d,
                             HIPFFT_FORWARD);
    if (iCUFFTRet != HIPFFT_SUCCESS)
    {
        (void) fprintf(stderr, "ERROR: FFT failed!");
        run = 0;
        return VEGAS_ERR_GEN;
    }

    return VEGAS_OK;
}

int GpuContext::accumulate()
{
    hipError_t iCUDARet = hipSuccess;

    Accumulate<<<_dimGAccum, _dimBAccum>>>(_pf4FFTOut_d,
                                           _pf4SumStokes_d);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    iCUDARet = hipGetLastError();
    if (iCUDARet != hipSuccess)
    {
        (void) fprintf(stderr, hipGetErrorString(iCUDARet));
        run = 0;
        return VEGAS_ERR_GEN;
    }

    return VEGAS_OK;
}

void GpuContext::zero_accumulator()
{
    CUDA_SAFE_CALL(hipMemset(_pf4SumStokes_d,
                                       '\0',
                                       (_nsubband
                                       * _nchan
                                       * sizeof(float4))));

    return;
}

int GpuContext::get_accumulated_spectrum_from_device(char *out)
{
    /* copy the negative frequencies out first */
    CUDASafeCall(hipMemcpy(out,
                            _pf4SumStokes_d + (_nsubband * _nchan / 2),
                            (_nsubband
                             * (_nchan / 2)
                             * sizeof(float4)),
                            hipMemcpyDeviceToHost));
    /* copy the positive frequencies out */
    CUDASafeCall(hipMemcpy(out + (_nsubband * (_nchan / 2) * sizeof(float4)),
                            _pf4SumStokes_d,
                            (_nsubband
                             * (_nchan / 2)
                             * sizeof(float4)),
                            hipMemcpyDeviceToHost));
    return VEGAS_OK;
}

/*
 * function to be used to check if any heap within the current PFB is invalid,
 * in which case, the entire PFB should be discarded.
 * NOTE: this function does not check ALL heaps - it returns at the first
 * invalid heap.
 */
int is_valid(int heap_start, int num_heaps)
{
    for (int i = heap_start; i < (heap_start + num_heaps); ++i)
    {
        if (!g_auiHeapValid[i])
        {
            return FALSE;
        }
    }

    return TRUE;
}

/*
 * function that checks if blanking has started within this accumulation.
 * ASSUMPTION: the blanking bit does not toggle within this time interval.
 */
int is_blanked(int heap_start, int num_heaps)
{
    for (int i = heap_start; i < (heap_start + num_heaps); ++i)
    {
        if (g_auiStatusBits[i] & 0x08)
        {
            return TRUE;
        }
    }

    return FALSE;
}

void __CUDASafeCall(hipError_t iCUDARet,
                               const char* pcFile,
                               const int iLine,
                               void (*pCleanUp)(void))
{
    if (iCUDARet != hipSuccess)
    {
        (void) fprintf(stderr,
                       "ERROR: File <%s>, Line %d: %s\n",
                       pcFile,
                       iLine,
                       hipGetErrorString(iCUDARet));
        run = 0;
        return;
    }

    return;
}



/*
 * Frees up any allocated memory.
 */
extern "C"
void cleanup_gpu()
{
#if 0
    /* free memory */
    if (gpuCtx._pc4InBuf != NULL)
    {
        free(gpuCtx._pc4InBuf);
        gpuCtx._pc4InBuf = NULL;
    }
    if (gpuCtx._pc4Data_d != NULL)
    {
        (void) hipFree(gpuCtx._pc4Data_d);
        gpuCtx._pc4Data_d = NULL;
    }
    if (gpuCtx._pf4FFTIn_d != NULL)
    {
        (void) hipFree(gpuCtx._pf4FFTIn_d);
        gpuCtx._pf4FFTIn_d = NULL;
    }
    if (gpuCtx._pf4FFTOut_d != NULL)
    {
        (void) hipFree(gpuCtx._pf4FFTOut_d);
        gpuCtx._pf4FFTOut_d = NULL;
    }
    if (gpuCtx._pf4SumStokes_d != NULL)
    {
        (void) hipFree(gpuCtx._pf4SumStokes_d);
        gpuCtx._pf4SumStokes_d = NULL;
    }

    /* destroy plan */
    /* TODO: check if plan exists */
    if (gpuCtx._stPlan)
    {
        (void) hipfftDestroy(gpuCtx._stPlan);
        gpuCtx._stPlan = NULL;
    }
    printf("#################### GPU CONTEXT CLEANED UP ####################\n");
#endif 
    return;
}
