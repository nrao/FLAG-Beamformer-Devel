#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <string.h>
#include <pthread.h>
#include <errno.h>
#include <hipfft/hipfft.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>

#include "fitshead.h"
#include "vegas_error.h"
#ifdef __cplusplus
extern "C" {
#endif
#include "vegas_status.h"
#include "vegas_databuf.h"
#ifdef __cplusplus
}
#endif
#include "vegas_defines.h"
#include "pfb_gpu.h"
#include "pfb_gpu_kernels.h"
#include "spead_heap.h"

#include "BlankingStateMachine.h"
#include "gpu_context.h"

#define STATUS_KEY "GPUSTAT"

/* ASSUMPTIONS: 1. All blocks contain the same number of heaps. */

extern int run;

/**
 * Global variables: maybe move this to a struct that is passed to each function?
 */
#include "gpu_context.h"


// Make the damn object global until we complete refactoring ...
GpuContext *gpuCtx = 0;

// static size_t g_buf_out_block_size;
static int g_iTotHeapOut = 0;
static int g_iMaxNumHeapOut = 0;
static int g_iHeapOut = 0;

/* these arrays need to be only a little longer than MAX_HEAPS_PER_BLK, but
   since we don't know the exact length, just allocate twice that value */
static unsigned int g_auiStatusBits[2*MAX_HEAPS_PER_BLK] = {0};
static unsigned int g_auiHeapValid[2*MAX_HEAPS_PER_BLK] = {0};
static int g_iSpecPerAcc = 0;

void __CUDASafeCall(hipError_t iCUDARet,
                               const char* pcFile,
                               const int iLine,
                               void (*pCleanUp)(void));

#define CUDASafeCall(iRet)   __CUDASafeCall(iRet,       \
                                                                  __FILE__,   \
                                                                  __LINE__,   \
                                                                  &cleanup_gpu)



extern "C"
int init_cuda_context(int subbands, int chans, int inBlokSz, int outBlokSz)
{
    int iDevCount = 0;

    /* since CUDASafeCall() calls hipGetErrorString(),
       it should not be used here - will cause crash if no CUDA device is
       found */
    (void) hipGetDeviceCount(&iDevCount);
    if (0 == iDevCount)
    {
        (void) fprintf(stderr, "ERROR: No CUDA-capable device found!\n");
        return EXIT_FAILURE;
    }        

    if (subbands == 0 || chans == 0)
    {

        /* just use the first device */
        printf("pfb_gpu.cu: CUDA_SAFE_CALL(hipSetDevice(0))\n");
        CUDA_SAFE_CALL(hipSetDevice(0));
        printf("pfb_gpu.cu: CUDA_SAFE_CALL(hipFree(0)\n");
        CUDA_SAFE_CALL(hipFree(0));
        printf("#################### GPU CONTEXT INITIALIZED ####################\n");
    }
    else
    {
        // Create a new mode specific set of resources.
        GpuContext *newctx, *oldctx;
        oldctx = gpuCtx;
        newctx = new GpuContext(oldctx, subbands, chans, inBlokSz, outBlokSz);
        gpuCtx = newctx;
        delete oldctx;
    }
    return EXIT_SUCCESS;
}


/* Initialize all necessary memory, etc for doing PFB
 * at the given params.
 */
extern "C" 
int reset_state(size_t input_block_sz, size_t output_block_sz, int num_subbands, int num_chans)
{
    g_iTotHeapOut = 0;
    g_iHeapOut = 0;
    g_iSpecPerAcc = 0;
    
    // Now verify we have the right setup
    if (gpuCtx == 0 || 
        true != gpuCtx->verify_setup(num_subbands, num_chans, input_block_sz, output_block_sz))
    {
        printf("Error: runtime and pre-init GPU setups didn't match\n");
        // For backward compatibility we try to initialize here.
        if (EXIT_SUCCESS != init_cuda_context(num_subbands, num_chans, input_block_sz, output_block_sz))
            return EXIT_FAILURE;
    }
    return EXIT_SUCCESS;
}


struct freq_spead_heap *
frequency_heap(struct vegas_databuf *db, int iblk, int iHeap)
{
    struct freq_spead_heap *freq_heap;
    char *ptr;
    ptr = (char *)(vegas_databuf_data(db, iblk) +
                   sizeof(struct freq_spead_heap) * iHeap);
    freq_heap = (struct freq_spead_heap*)ptr;                
    return freq_heap;
}

/* dump to buffer */
int dump_to_buffer(struct vegas_databuf *db_out,         // Output databuffer
                   int curblk_out,                       // Current output block
                   int iHeapOut,                         // output frequency heap number in current block
                   struct time_spead_heap *firsttimeheap,// first time sample of input 
                   int iTotHeapOut,                      // spectrum number/counter
                   int iSpecPerAcc,                      // GPU accumulations in this heap
                   double heap_mjd,                      // MJD from index_input
                   int first_t_series_status)            // switch state of first accumulation
{
    struct freq_spead_heap *freq_heap_out;
    char * payload_addr_out;
    struct databuf_index *index_out;
    int rtn;
    
    freq_heap_out = vegas_datablock_freq_heap_header(db_out, curblk_out, iHeapOut);
    index_out = (struct databuf_index*)vegas_databuf_index(db_out, curblk_out);

    if (sizeof(struct freq_spead_heap) * MAX_HEAPS_PER_BLK + 
        index_out->heap_size*(index_out->num_heaps+1) > db_out->block_size ||
        iHeapOut >= db_out->index_size)
    {
        printf("DATABUF ERROR: heapsize*nheaps > blocksize!! (%d > %zd) index_size=%d\n",
            index_out->heap_size*index_out->num_heaps, db_out->block_size, db_out->index_size); 
        printf("DATABUF ERROR: blocknum=%d, iHeapOut=%d,iTotHeapOut=%d,iSpecPerAcc=%d\n",
            curblk_out, iHeapOut,iTotHeapOut, iSpecPerAcc);
    } 
        
    payload_addr_out = vegas_datablock_freq_heap_data(db_out, curblk_out, iHeapOut);

    /* Write new heap header fields */
    freq_heap_out->time_cntr_id = 0x20;
    freq_heap_out->time_cntr_top8 = firsttimeheap->time_cntr_top8;
    freq_heap_out->time_cntr = firsttimeheap->time_cntr;
    freq_heap_out->spectrum_cntr_id = 0x21;
    freq_heap_out->spectrum_cntr = iTotHeapOut;
    freq_heap_out->integ_size_id = 0x22;
    freq_heap_out->integ_size = iSpecPerAcc;
    freq_heap_out->mode_id = 0x23;
    freq_heap_out->mode = firsttimeheap->mode;
    freq_heap_out->status_bits_id = 0x24;
    freq_heap_out->status_bits = first_t_series_status;
    freq_heap_out->payload_data_off_addr_mode = 0;
    freq_heap_out->payload_data_off_id = 0x25;
    freq_heap_out->payload_data_off = 0;
    
/////DEBUG
    memset(firsttimeheap, 0, sizeof(struct time_spead_heap));

    /* Update output index */
    index_out->cpu_gpu_buf[iHeapOut].heap_valid = 1;
    index_out->cpu_gpu_buf[iHeapOut].heap_cntr = iTotHeapOut;
    index_out->cpu_gpu_buf[iHeapOut].heap_rcvd_mjd = heap_mjd;

    /* copy out GPU data into buffer */
    rtn = gpuCtx->get_accumulated_spectrum_from_device(payload_addr_out);
    index_out->num_heaps += (rtn == VEGAS_OK ? 1 : 0);
    return rtn;
}
    
/* Actually do the PFB by calling CUDA kernels */
extern "C"
void do_pfb(struct vegas_databuf *db_in,
            int curblock_in,
            struct vegas_databuf *db_out,
            int *curblock_out,
            int first,
            struct vegas_status st,
            int acc_len)
{
    /* Declare local variables */
    char *hdr_out = NULL;
    struct databuf_index *index_in = NULL;
    struct databuf_index *index_out = NULL;
    int heap_in = 0;
    char *heap_addr_in = NULL;

    int iProcData = 0;
    hipError_t iCUDARet = hipSuccess;
    int iRet = VEGAS_OK;
    char* payload_addr_in = NULL;
    int num_in_heaps_per_proc = 0;
    int pfb_count = 0;
    int num_in_heaps_gpu_buffer = 0;
    int num_in_heaps_tail = 0;
    int i = 0;
    int iBlockInDataSize;
    int nsubband_x_nchan;
    size_t nsubband_x_nchan_fsize;
    size_t nsubband_x_nchan_csize;

    nsubband_x_nchan = gpuCtx->_nsubband * gpuCtx->_nchan;
    nsubband_x_nchan_fsize = nsubband_x_nchan * sizeof(float4);
    nsubband_x_nchan_csize = nsubband_x_nchan * sizeof(char4);
    
    /* Setup input and first output data block stuff */
    index_in = (struct databuf_index*)vegas_databuf_index(db_in, curblock_in);
    /* Get the number of heaps per block of data that will be processed by the GPU */
    num_in_heaps_per_proc = nsubband_x_nchan_csize / time_heap_datasize(index_in);
    iBlockInDataSize = index_in->num_heaps * time_heap_datasize(index_in);

    num_in_heaps_tail = ((VEGAS_NUM_TAPS - 1) * nsubband_x_nchan_csize)
                         / time_heap_datasize(index_in);
    num_in_heaps_gpu_buffer = index_in->num_heaps + num_in_heaps_tail;

    /* Calculate the maximum number of output heaps per block */
    g_iMaxNumHeapOut = (gpuCtx->_out_block_size - (sizeof(struct freq_spead_heap) * MAX_HEAPS_PER_BLK)) / nsubband_x_nchan_fsize;

    hdr_out = vegas_databuf_header(db_out, *curblock_out);
    index_out = (struct databuf_index*)vegas_databuf_index(db_out, *curblock_out);
    // index_out->num_heaps = 0;
    memcpy(hdr_out, vegas_databuf_header(db_in, curblock_in), VEGAS_STATUS_SIZE);

    /* Set basic params in output index */
    index_out->heap_size = sizeof(struct freq_spead_heap) + (nsubband_x_nchan_fsize);
    /* Read in heap from buffer */
    heap_addr_in = (char*)vegas_datablock_time_heap_header(db_in, curblock_in, heap_in);

    /* Here, the payload_addr_in is the start of the contiguous block of data that will be
       copied to the GPU (heap_in = 0) */
    payload_addr_in = vegas_datablock_time_heap_data(db_in, curblock_in, heap_in);

    /* Copy data block to GPU */
    if (first)
    {
        // bloksz replaces a calculated value which caused the check below to fail
        // in the presence of dropped packets. We used the blocksize from the data buffer
        // instead here to get things going. Not sure how dropped data at start of
        // scan should be treated.
        int bloksz;
        bloksz = iBlockInDataSize; // calculated above
        /* Sanity check for the first iteration */
        if ((bloksz % (nsubband_x_nchan_csize)) != 0)
        {
            (void) fprintf(stderr, "ERROR: Data size mismatch! BlockInDataSize=%d NumSubBands=%d nchan=%d\n",
                                    bloksz, gpuCtx->_nsubband, gpuCtx->_nchan);
            run = 0;
            return;
        }
        // Cuda Note: hipMemcpy host to device is asynchronous, be supposedly safe.
        CUDA_SAFE_CALL(hipMemcpy(gpuCtx->_pc4Data_d,
                                payload_addr_in,
                                bloksz,
                                hipMemcpyHostToDevice));
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        iCUDARet = hipGetLastError();
        if (iCUDARet != hipSuccess)
        {
            (void) fprintf(stderr, hipGetErrorString(iCUDARet));
        }
                                
        /* duplicate the last (VEGAS_NUM_TAPS - 1) segments at the end for 
           the next iteration */
        CUDA_SAFE_CALL(hipMemcpy(gpuCtx->_pc4Data_d + (bloksz / sizeof(char4)),
                                  gpuCtx->_pc4Data_d + (bloksz - ((VEGAS_NUM_TAPS - 1) * nsubband_x_nchan_csize))/sizeof(char4),
                                  ((VEGAS_NUM_TAPS - 1) * nsubband_x_nchan_csize),
                                  hipMemcpyDeviceToDevice));
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        iCUDARet = hipGetLastError();
        if (iCUDARet != hipSuccess)
        {
            (void) fprintf(stderr, hipGetErrorString(iCUDARet));
        }

        /* copy the status bits and valid flags for all heaps to arrays separate
           from the index, so that it can be combined with the corresponding
           values from the previous block */
        struct time_spead_heap* time_heap = (struct time_spead_heap*) vegas_databuf_data(db_in, curblock_in);
        for (i = 0; i < index_in->num_heaps; ++i)
        {
            g_auiStatusBits[i] = time_heap->status_bits;
            g_auiHeapValid[i] = index_in->cpu_gpu_buf[i].heap_valid;
            ++time_heap;
        }
        /* duplicate the last (VEGAS_NUM_TAPS - 1) segments at the end for the
           next iteration */
        for ( ; i < index_in->num_heaps + num_in_heaps_tail; ++i)
        {
            g_auiStatusBits[i] = g_auiStatusBits[i-num_in_heaps_tail];
            g_auiHeapValid[i] = g_auiHeapValid[i-num_in_heaps_tail];
        }
        // Zero out accumulators for 1st integration
        gpuCtx->zero_accumulator();
        printf("num_heaps=%d num_in_tail=%d\n", index_in->num_heaps, num_in_heaps_tail);
    }
    else
    {
        /* If this is not the first run, need to handle block boundary, while doing the PFB */
        CUDA_SAFE_CALL(hipMemcpy(gpuCtx->_pc4Data_d,
                                gpuCtx->_pc4Data_d + (iBlockInDataSize / sizeof(char4)),
                                ((VEGAS_NUM_TAPS - 1) * nsubband_x_nchan_csize),
                                hipMemcpyDeviceToDevice));
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        iCUDARet = hipGetLastError();
        if (iCUDARet != hipSuccess)
        {
            (void) fprintf(stderr, hipGetErrorString(iCUDARet));
        }
                                
        // Cuda Note: hipMemcpy host to device is asynchronous, be supposedly safe.                                
        CUDA_SAFE_CALL(hipMemcpy(gpuCtx->_pc4Data_d + ((VEGAS_NUM_TAPS - 1) * nsubband_x_nchan),
                                payload_addr_in,
                                iBlockInDataSize,
                                hipMemcpyHostToDevice));
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        iCUDARet = hipGetLastError();
        if (iCUDARet != hipSuccess)
        {
            (void) fprintf(stderr, hipGetErrorString(iCUDARet));
        }
                                
        /* copy the status bits and valid flags for all heaps to arrays separate
           from the index, so that it can be combined with the corresponding
           values from the previous block */
        for (i = 0; i < num_in_heaps_tail; ++i)
        {
            g_auiStatusBits[i] = g_auiStatusBits[index_in->num_heaps+i];
            g_auiHeapValid[i] = g_auiHeapValid[index_in->num_heaps+i];
        }
        struct time_spead_heap* time_heap = (struct time_spead_heap*) vegas_databuf_data(db_in, curblock_in);
        for ( ; i < num_in_heaps_tail + index_in->num_heaps; ++i)
        {
            g_auiStatusBits[i] = time_heap->status_bits;           
            g_auiHeapValid[i] = index_in->cpu_gpu_buf[i-num_in_heaps_tail].heap_valid;
            ++time_heap;
        }
    }

    gpuCtx->_pc4DataRead_d = gpuCtx->_pc4Data_d;
    iProcData = 0;
    while (iBlockInDataSize > iProcData)  /* loop till (num_heaps * heap_size) of data is processed */
    {
        if (0 == pfb_count)
        {
            /* Check if all heaps necessary for this PFB are valid */
            if (!(is_valid(heap_in, (VEGAS_NUM_TAPS * num_in_heaps_per_proc))))
            {
                /* Skip all heaps that go into this PFB if there is an invalid heap */
                iProcData += (VEGAS_NUM_TAPS * nsubband_x_nchan_csize);
                /* update the data read pointer */
                gpuCtx->_pc4DataRead_d += (VEGAS_NUM_TAPS * nsubband_x_nchan);
                if (iProcData >= iBlockInDataSize)
                {
                    break;
                }

                /* Calculate input heap addresses for the next round of processing */
                heap_in += (VEGAS_NUM_TAPS * num_in_heaps_per_proc);
                if (heap_in > num_in_heaps_gpu_buffer)
                {
                    /* This is not supposed to happen (but may happen if odd number of pkts are dropped
                       right at the end of the buffer, so we therefore do not exit) */
                    (void) fprintf(stdout,
                                   "WARNING: Heap count %d exceeds available number of heaps %d!\n",
                                   heap_in,
                                   num_in_heaps_gpu_buffer);
                }
                heap_addr_in = (char*)vegas_datablock_time_heap_header(db_in, curblock_in, heap_in);
                continue;
            }
        }
        /* Perform polyphase filtering */
        DoPFB<<<gpuCtx->_dimGPFB, gpuCtx->_dimBPFB>>>(gpuCtx->_pc4DataRead_d,
                                                      gpuCtx->_pf4FFTIn_d,
                                                      gpuCtx->_pfPFBCoeff_d);
        CUDA_SAFE_CALL(hipDeviceSynchronize());
        iCUDARet = hipGetLastError();
        if (iCUDARet != hipSuccess)
        {
            (void) fprintf(stdout,
                           "ERROR: File <%s>, Line %d: %s\n",
                           __FILE__,
                           __LINE__,
                           hipGetErrorString(iCUDARet));
            run = 0;
            break;
        }

        iRet = gpuCtx->do_fft();
        if (iRet != VEGAS_OK)
        {
            (void) fprintf(stdout, "ERROR: FFT failed!\n");
            run = 0;
            break;
        }

        gpuCtx->blanking_inputs(is_blanked(heap_in, num_in_heaps_per_proc));
        ++g_iTotHeapOut; // unconditional spectrum counter
                                
        /* Accumulate power x, power y, stokes real and imag, if the blanking
           bit is not set */
        if (!(gpuCtx->blank_current_fft()))
        {
            iRet = gpuCtx->accumulate();
            if (iRet != VEGAS_OK)
            {
                (void) fprintf(stdout, "ERROR: Accumulation failed!\n");
                run = 0;
                break;
            }
            ++g_iSpecPerAcc;
            // record the first unblanked state in this accumulation sequence
            if (1 == g_iSpecPerAcc)
            {
                gpuCtx->_first_time_heap_in_accum_status_bits = g_auiStatusBits[heap_in];
                memcpy(&gpuCtx->_first_time_heap_in_accum, heap_addr_in, sizeof(gpuCtx->_first_time_heap_in_accum));
                gpuCtx->_first_time_heap_mjd = index_in->cpu_gpu_buf[heap_in].heap_rcvd_mjd;               
            }                    
        }
        
        if (g_iSpecPerAcc == acc_len || gpuCtx->needs_flush())
        {
            /* dump to buffer */
            // If no accumulations have occurred, then just clear the accumulator and start again.
            if (g_iSpecPerAcc > 0)
            {
                iRet = dump_to_buffer(db_out,             
                                      *curblock_out,
                                      g_iHeapOut,
                                      &gpuCtx->_first_time_heap_in_accum,
                                      g_iTotHeapOut,
                                      g_iSpecPerAcc,
                                      gpuCtx->_first_time_heap_mjd,
                                      gpuCtx->_first_time_heap_in_accum_status_bits);
            
                if (iRet != VEGAS_OK)
                {
                    (void) fprintf(stdout, "ERROR: Getting accumulated spectrum failed!\n");
                    run = 0;
                    break;
                }                                  
                ++g_iHeapOut;
            }
            else
            {
                printf("Scanlength: GPU:asked to dump buffer but no accumulations present\n");
            }

            /* zero accumulators */
            gpuCtx->zero_accumulator();
            /* reset time */
            g_iSpecPerAcc = 0;
        }

        iProcData += nsubband_x_nchan_csize;
        /* update the data read pointer */
        gpuCtx->_pc4DataRead_d += (nsubband_x_nchan);

        /* Calculate input heap addresses for the next round of processing */
        heap_in += num_in_heaps_per_proc;
        heap_addr_in = (char*)vegas_datablock_time_heap_header(db_in, curblock_in, heap_in);

        /* if output block is full */
        if (g_iHeapOut == g_iMaxNumHeapOut)
        {
            /* Set the number of heaps written to this block */
            /* Mark output buffer as filled */
            vegas_databuf_set_filled(db_out, *curblock_out);

            // printf("Debug: vegas_pfb_thread going to next output block\n");

            /* Note current output block */
            /* NOTE: vegas_status_lock_safe() and vegas_status_unlock_safe() are macros
               that have been explicitly expanded here, due to compilation issues */
            //vegas_status_lock_safe(&st);
                pthread_cleanup_push((void (*) (void *))&vegas_status_unlock, (void *) &st);
                vegas_status_lock(&st);
            hputi4(st.buf, "PFBBLKOU", *curblock_out);
            //vegas_status_unlock_safe(&st);
                vegas_status_unlock(&st);
                pthread_cleanup_pop(0);

            /*  Wait for next output block */
            *curblock_out = (*curblock_out + 1) % db_out->n_block;
            while ((vegas_databuf_wait_free(db_out, *curblock_out)!=0) && run) {
                //vegas_status_lock_safe(&st);
                    pthread_cleanup_push((void (*)(void *))&vegas_status_unlock, (void *) &st);
                    vegas_status_lock(&st);

                hputs(st.buf, STATUS_KEY, "blocked");
                //vegas_status_unlock_safe(&st);
                    vegas_status_unlock(&st);
                    pthread_cleanup_pop(0);
            }

            g_iHeapOut = 0;

            hdr_out = vegas_databuf_header(db_out, *curblock_out);
            index_out = (struct databuf_index*)vegas_databuf_index(db_out, *curblock_out);
            index_out->num_heaps = 0;
            memcpy(hdr_out, vegas_databuf_header(db_in, curblock_in),
                    VEGAS_STATUS_SIZE);

            /* Set basic params in output index */
            index_out->heap_size = sizeof(struct freq_spead_heap) + (nsubband_x_nchan_fsize);
        }

        pfb_count = (pfb_count + 1) % VEGAS_NUM_TAPS;
    }

    return;
}

/* function that performs the FFT */
int GpuContext::do_fft()
{
    hipfftResult iCUFFTRet = HIPFFT_SUCCESS;
    hipError_t iCUDARet = hipSuccess;

    /* execute plan */
    iCUFFTRet = hipfftExecC2C(_stPlan,
                             (hipfftComplex*) _pf4FFTIn_d,
                             (hipfftComplex*) _pf4FFTOut_d,
                             HIPFFT_FORWARD);
    if (iCUFFTRet != HIPFFT_SUCCESS)
    {
        (void) fprintf(stderr, "ERROR: FFT failed!");
        iCUDARet = hipGetLastError();
        if (iCUDARet != hipSuccess)
        {
            (void) fprintf(stderr, hipGetErrorString(iCUDARet));
            run = 0;
            return VEGAS_ERR_GEN;
        }
    }
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    iCUDARet = hipGetLastError();
    if (iCUDARet != hipSuccess)
    {
        (void) fprintf(stderr, hipGetErrorString(iCUDARet));
        run = 0;
        return VEGAS_ERR_GEN;
    }

    return VEGAS_OK;
}

int GpuContext::accumulate()
{
    hipError_t iCUDARet = hipSuccess;

    Accumulate<<<_dimGAccum, _dimBAccum>>>(_pf4FFTOut_d,
                                           _pf4SumStokes_d);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    iCUDARet = hipGetLastError();
    if (iCUDARet != hipSuccess)
    {
        (void) fprintf(stderr, hipGetErrorString(iCUDARet));
        run = 0;
        return VEGAS_ERR_GEN;
    }

    return VEGAS_OK;
}

void GpuContext::zero_accumulator()
{
    hipError_t iCUDARet = hipSuccess;
    CUDA_SAFE_CALL(hipMemset(_pf4SumStokes_d,
                                       '\0',
                                       (_nsubband
                                       * _nchan
                                       * sizeof(float4))));
    CUDA_SAFE_CALL(hipDeviceSynchronize());                                       
    if (iCUDARet != hipSuccess)
    {
        (void) fprintf(stderr, hipGetErrorString(iCUDARet));
    }

    return;
}

int GpuContext::get_accumulated_spectrum_from_device(char *out)
{
    hipError_t iCUDARet = hipSuccess;
    // Cuda note: Device to host memcpy is always synchronous
    /* copy the negative frequencies out first */
    CUDASafeCall(hipMemcpy(out,
                            _pf4SumStokes_d + (_nsubband * _nchan / 2),
                            (_nsubband
                             * (_nchan / 2)
                             * sizeof(float4)),
                            hipMemcpyDeviceToHost));
    /* copy the positive frequencies out */
    CUDASafeCall(hipMemcpy(out + (_nsubband * (_nchan / 2) * sizeof(float4)),
                            _pf4SumStokes_d,
                            (_nsubband
                             * (_nchan / 2)
                             * sizeof(float4)),
                            hipMemcpyDeviceToHost));
    iCUDARet = hipGetLastError();
    if (iCUDARet != hipSuccess)
    {
        (void) fprintf(stderr, hipGetErrorString(iCUDARet));
    }

#ifdef DEBUG_ZERO_CHANNELS    
    // DEBUG check for near or zero channels X*X and Y*Y should never be =< 0.0                            
    int i, ndata, n_null = 0;
    ndata = _nsubband*_nchan;
    float4 *data = (float4 *)out;
    int first_bad = 0;
    
    for (i=0; i<ndata; ++i)
    {
        if (data[i].x <= 0.0 || data[i].y <= 0.0)
        {
            n_null++;
            first_bad=first_bad == 0 ? i : first_bad;
        }
    }
    if (n_null != 0)
    {
        printf("GPU: %d nil channels starting at %d\n", n_null, first_bad);
    }
#endif
    return VEGAS_OK;
}

/*
 * function to be used to check if any heap within the current PFB is invalid,
 * in which case, the entire PFB should be discarded.
 * NOTE: this function does not check ALL heaps - it returns at the first
 * invalid heap.
 */
int is_valid(int heap_start, int num_heaps)
{
    for (int i = heap_start; i < (heap_start + num_heaps); ++i)
    {
        if (!g_auiHeapValid[i])
        {
            return FALSE;
        }
    }

    return TRUE;
}

/*
A note about blanking:
The blanking status is copied from the time series input into the array
g_auiStatusBits, with time acending with index, like so:
    g_auiStatusBits[32] = t0
    g_auiStatusBits[33] = t0 + dt
    g_auiStatusBits[34] = t0 + dt + dt
    
So when we think about labeling the frequency heap outputs, the convention
is to use the '1st' non-blanked time-series (e.g. index 32 above) to fill 
in the timestamp, counter, mjd etc.

However, when we think about how to process blanking, we need to use the
most recent(e.g index 34 above), status to drive the blanking state machine.
Below, the check which sets 0x2 is taken from the most current time-series status.

 is_blanked(tail, length)
 * Check the input time series for blanking and encode
 * the result.
 * Return value: 
 *  - bit 0x4 -- indicates cal or sig/ref state changed during input
 *  - bit 0x2 -- indicates if the most recent time sample had blanking asserted
 *  - bit 0x1 -- indicates if any of the time samples had blanking asserted
 */
int is_blanked(int heap_start, int num_heaps)
{
    int state_changed = 0;
    int banked_at_start = (g_auiStatusBits[heap_start + num_heaps- 1] & 0x8)  ? 0x2 : 0x0;
    int is_blanked = (banked_at_start || (g_auiStatusBits[heap_start] & 0x8)) ? 0x1 : 0x0;
   
    for (int i = heap_start + 1; i < (heap_start + num_heaps); ++i)
    {
        if ((g_auiStatusBits[i] & 0x3) != (g_auiStatusBits[i-1] & 0x3))
        {
            state_changed = 0x4;
        }
        if (g_auiStatusBits[i] & 0x08)
        {
            is_blanked = 0x1;
        }
    }
    return (banked_at_start | state_changed | is_blanked);
}

void __CUDASafeCall(hipError_t iCUDARet,
                               const char* pcFile,
                               const int iLine,
                               void (*pCleanUp)(void))
{
    if (iCUDARet != hipSuccess)
    {
        (void) fprintf(stderr,
                       "ERROR: File <%s>, Line %d: %s\n",
                       pcFile,
                       iLine,
                       hipGetErrorString(iCUDARet));
        run = 0;
        return;
    }

    return;
}



/*
 * Frees up any allocated memory.
 */
extern "C"
void cleanup_gpu()
{
#if 0
    /* free memory */
    if (gpuCtx._pc4InBuf != NULL)
    {
        free(gpuCtx._pc4InBuf);
        gpuCtx._pc4InBuf = NULL;
    }
    if (gpuCtx._pc4Data_d != NULL)
    {
        (void) hipFree(gpuCtx._pc4Data_d);
        gpuCtx._pc4Data_d = NULL;
    }
    if (gpuCtx._pf4FFTIn_d != NULL)
    {
        (void) hipFree(gpuCtx._pf4FFTIn_d);
        gpuCtx._pf4FFTIn_d = NULL;
    }
    if (gpuCtx._pf4FFTOut_d != NULL)
    {
        (void) hipFree(gpuCtx._pf4FFTOut_d);
        gpuCtx._pf4FFTOut_d = NULL;
    }
    if (gpuCtx._pf4SumStokes_d != NULL)
    {
        (void) hipFree(gpuCtx._pf4SumStokes_d);
        gpuCtx._pf4SumStokes_d = NULL;
    }

    /* destroy plan */
    /* TODO: check if plan exists */
    if (gpuCtx._stPlan)
    {
        (void) hipfftDestroy(gpuCtx._stPlan);
        gpuCtx._stPlan = NULL;
    }
    printf("#################### GPU CONTEXT CLEANED UP ####################\n");
#endif 
    return;
}
