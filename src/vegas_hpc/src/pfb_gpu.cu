#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <pthread.h>
#include <errno.h>
#include <hipfft/hipfft.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>

#include "fitshead.h"
#include "vegas_error.h"
#ifdef __cplusplus
extern "C" {
#endif
#include "vegas_status.h"
#include "vegas_databuf.h"
#ifdef __cplusplus
}
#endif
#include "vegas_defines.h"
#include "pfb_gpu.h"
#include "pfb_gpu_kernels.h"
#include "spead_heap.h"

#define STATUS_KEY "GPUSTAT"

/* ASSUMPTIONS: 1. All blocks contain the same number of heaps. */

extern int run;

/**
 * Global variables: maybe move this to a struct that is passed to each function?
 */
size_t g_buf_in_block_size;
size_t g_buf_out_block_size;
int g_nchan;

hipfftHandle g_stPlan = {0};
float4* g_pf4FFTIn_d = NULL;
float4* g_pf4FFTOut_d = NULL;
char4* g_pc4InBuf = NULL;
char4* g_pc4InBufRead = NULL;
char4* g_pc4Data_d = NULL;              /* raw data starting address */
char4* g_pc4DataRead_d = NULL;          /* raw data read pointer */
dim3 g_dimBPFB(1, 1, 1);
dim3 g_dimGPFB(1, 1);
dim3 g_dimBAccum(1, 1, 1);
dim3 g_dimGAccum(1, 1);
float4* g_pf4SumStokes_d = NULL;
int g_iNumSubBands = 0;
int g_iFileCoeff = 0;
char g_acFileCoeff[256] = {0};
float *g_pfPFBCoeff = NULL;
float *g_pfPFBCoeff_d = NULL;
unsigned int g_iPrevBlankingState = FALSE;
int g_iTotHeapOut = 0;
int g_iMaxNumHeapOut = 0;
int g_iPFBCurBlockOut = 0;
int g_iHeapOut = 0;
int g_iBlockInDataSize = 0;
/* these arrays need to be only a little longer than MAX_HEAPS_PER_BLK, but
   since we don't know the exact length, just allocate twice that value */
unsigned int g_auiStatusBits[2*MAX_HEAPS_PER_BLK] = {0};
unsigned int g_auiHeapValid[2*MAX_HEAPS_PER_BLK] = {0};
int g_iFirstHeapIn = 0;
double g_dFirstHeapRcvdMJD = 0.0;
int g_iSpecPerAcc = 0;

void __CUDASafeCall(hipError_t iCUDARet,
                               const char* pcFile,
                               const int iLine,
                               void (*pCleanUp)(void));

#define CUDASafeCall(iRet)   __CUDASafeCall(iRet,       \
                                                                  __FILE__,   \
                                                                  __LINE__,   \
                                                                  &cleanup_gpu)

/* Initialize all necessary memory, etc for doing PFB 
 * at the given params.
 */
extern "C"
int init_gpu(size_t input_block_sz, size_t output_block_sz, int num_subbands, int num_chans)
{
    int iDevCount = 0;
    hipDeviceProp_t stDevProp = {0};
    hipfftResult iCUFFTRet = HIPFFT_SUCCESS;
    int iRet = EXIT_SUCCESS;
    int iMaxThreadsPerBlock = 0;

    g_buf_in_block_size = input_block_sz;
    g_buf_out_block_size = output_block_sz;
    g_nchan = num_chans;
    g_iNumSubBands = num_subbands;

    /* since CUDASafeCall() calls hipGetErrorString(),
       it should not be used here - will cause crash if no CUDA device is
       found */
    (void) hipGetDeviceCount(&iDevCount);
    if (0 == iDevCount)
    {
        (void) fprintf(stderr, "ERROR: No CUDA-capable device found!\n");
        run = 0;
        return EXIT_FAILURE;
    }

    /* just use the first device */
    CUDASafeCall(hipSetDevice(0));

    CUDASafeCall(hipGetDeviceProperties(&stDevProp, 0));
    iMaxThreadsPerBlock = stDevProp.maxThreadsPerBlock;

    g_pfPFBCoeff = (float *) malloc(g_iNumSubBands
                                          * VEGAS_NUM_TAPS
                                          * g_nchan
                                          * sizeof(float));
    if (NULL == g_pfPFBCoeff)
    {
        (void) fprintf(stderr,
                       "ERROR: Memory allocation failed! %s.\n",
                       strerror(errno));
        return EXIT_FAILURE;
    }

    /* allocate memory for the filter coefficient array on the device */
    CUDASafeCall(hipMalloc((void **) &g_pfPFBCoeff_d,
                                       g_iNumSubBands
                                       * VEGAS_NUM_TAPS
                                       * g_nchan
                                       * sizeof(float)));

    /* read filter coefficients */
    /* build file name */
    (void) sprintf(g_acFileCoeff,
                   "%s_%s_%d_%d_%d%s",
                   FILE_COEFF_PREFIX,
                   FILE_COEFF_DATATYPE,
                   VEGAS_NUM_TAPS,
                   g_nchan,
                   g_iNumSubBands,
                   FILE_COEFF_SUFFIX);
    g_iFileCoeff = open(g_acFileCoeff, O_RDONLY);
    if (g_iFileCoeff < EXIT_SUCCESS)
    {
        (void) fprintf(stderr,
                       "ERROR: Opening filter coefficients file %s "
                       "failed! %s.\n",
                       g_acFileCoeff,
                       strerror(errno));
        return EXIT_FAILURE;
    }

    iRet = read(g_iFileCoeff,
                g_pfPFBCoeff,
                g_iNumSubBands * VEGAS_NUM_TAPS * g_nchan * sizeof(float));
    if (iRet != (g_iNumSubBands * VEGAS_NUM_TAPS * g_nchan * sizeof(float)))
    {
        (void) fprintf(stderr,
                       "ERROR: Reading filter coefficients failed! %s.\n",
                       strerror(errno));
        return EXIT_FAILURE;
    }
    (void) close(g_iFileCoeff);

    /* copy filter coefficients to the device */
    CUDASafeCall(hipMemcpy(g_pfPFBCoeff_d,
               g_pfPFBCoeff,
               g_iNumSubBands * VEGAS_NUM_TAPS * g_nchan * sizeof(float),
               hipMemcpyHostToDevice));

    /* allocate memory for data array - 32MB is the block size for the VEGAS
       input buffer, allocate 32MB + space for (VEGAS_NUM_TAPS - 1) blocks of
       data
       NOTE: the actual data in a 32MB block will be only
       (num_heaps * heap_size), but since we don't know that value until data
       starts flowing, allocate the maximum possible size */
    CUDASafeCall(hipMalloc((void **) &g_pc4Data_d,
                                       (g_buf_in_block_size
                                        + ((VEGAS_NUM_TAPS - 1)
                                           * g_iNumSubBands
                                           * g_nchan
                                           * sizeof(char4)))));
    g_pc4DataRead_d = g_pc4Data_d;

    /* calculate kernel parameters */
    /* ASSUMPTION: g_nchan >= iMaxThreadsPerBlock */
    g_dimBPFB.x = iMaxThreadsPerBlock;
    g_dimBAccum.x = iMaxThreadsPerBlock;
    g_dimGPFB.x = (g_iNumSubBands * g_nchan) / iMaxThreadsPerBlock;
    g_dimGAccum.x = (g_iNumSubBands * g_nchan) / iMaxThreadsPerBlock;

    CUDASafeCall(hipMalloc((void **) &g_pf4FFTIn_d,
                                 g_iNumSubBands * g_nchan * sizeof(float4)));
    CUDASafeCall(hipMalloc((void **) &g_pf4FFTOut_d,
                                 g_iNumSubBands * g_nchan * sizeof(float4)));
    CUDASafeCall(hipMalloc((void **) &g_pf4SumStokes_d,
                                 g_iNumSubBands * g_nchan * sizeof(float4)));
    CUDASafeCall(hipMemset(g_pf4SumStokes_d,
                                 '\0',
                                 g_iNumSubBands * g_nchan * sizeof(float4)));

    /* create plan */
    iCUFFTRet = hipfftPlanMany(&g_stPlan,
                              FFTPLAN_RANK,
                              &g_nchan,
                              &g_nchan,
                              FFTPLAN_ISTRIDE,
                              FFTPLAN_IDIST,
                              &g_nchan,
                              FFTPLAN_OSTRIDE,
                              FFTPLAN_ODIST,
                              HIPFFT_C2C,
                              FFTPLAN_BATCH);
    if (iCUFFTRet != HIPFFT_SUCCESS)
    {
        (void) fprintf(stderr, "ERROR: Plan creation failed!\n");
        run = 0;
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}

/* Actually do the PFB by calling CUDA kernels */
extern "C"
void do_pfb(struct vegas_databuf *db_in,
            int curblock_in,
            struct vegas_databuf *db_out,
            int first,
            struct vegas_status st,
            int acc_len)
{
    /* Declare local variables */
    char *hdr_out = NULL;
    struct databuf_index *index_in = NULL;
    struct databuf_index *index_out = NULL;
    int heap_in = 0;
    char *heap_addr_in = NULL;
    char *heap_addr_out = NULL;
    struct time_spead_heap* first_time_heap_in_accum = NULL;
    struct freq_spead_heap* freq_heap_out = NULL;
    int iProcData = 0;
    hipError_t iCUDARet = hipSuccess;
    int iRet = VEGAS_OK;
    char* payload_addr_in = NULL;
    char* payload_addr_out = NULL;
    int num_in_heaps_per_proc = 0;
    int pfb_count = 0;
    int num_in_heaps_gpu_buffer = 0;
    int num_in_heaps_tail = 0;
    int i = 0;

    /* Setup input and first output data block stuff */
    index_in = (struct databuf_index*)vegas_databuf_index(db_in, curblock_in);
    /* Get the number of heaps per block of data that will be processed by the GPU */
    num_in_heaps_per_proc = (g_iNumSubBands * g_nchan * sizeof(char4)) / (index_in->heap_size - sizeof(struct time_spead_heap));
    g_iBlockInDataSize = (index_in->num_heaps * index_in->heap_size) - (index_in->num_heaps * sizeof(struct time_spead_heap));

    num_in_heaps_tail = (((VEGAS_NUM_TAPS - 1) * g_iNumSubBands * g_nchan * sizeof(char4))
                         / (index_in->heap_size - sizeof(struct time_spead_heap)));
    num_in_heaps_gpu_buffer = index_in->num_heaps + num_in_heaps_tail;

    /* Calculate the maximum number of output heaps per block */
    g_iMaxNumHeapOut = (g_buf_out_block_size - (sizeof(struct time_spead_heap) * MAX_HEAPS_PER_BLK)) / (g_iNumSubBands * g_nchan * sizeof(float4)); 

    hdr_out = vegas_databuf_header(db_out, g_iPFBCurBlockOut);
    index_out = (struct databuf_index*)vegas_databuf_index(db_out, g_iPFBCurBlockOut);
    memcpy(hdr_out, vegas_databuf_header(db_in, curblock_in),
            VEGAS_STATUS_SIZE);

    /* Set basic params in output index */
    index_out->heap_size = sizeof(struct freq_spead_heap) + (g_iNumSubBands * g_nchan * sizeof(float4));
    /* Read in heap from buffer */
    heap_addr_in = (char*)(vegas_databuf_data(db_in, curblock_in) +
                        sizeof(struct time_spead_heap) * heap_in);
    first_time_heap_in_accum = (struct time_spead_heap*)(heap_addr_in);
    if (first)
    {
        g_iFirstHeapIn = heap_in;
        g_dFirstHeapRcvdMJD = index_in->cpu_gpu_buf[g_iFirstHeapIn].heap_rcvd_mjd;
    }
    /* Here, the payload_addr_in is the start of the contiguous block of data that will be
       copied to the GPU (heap_in = 0) */
    payload_addr_in = (char*)(vegas_databuf_data(db_in, curblock_in) +
                        sizeof(struct time_spead_heap) * MAX_HEAPS_PER_BLK +
                        (index_in->heap_size - sizeof(struct time_spead_heap)) * heap_in );

    /* Copy data block to GPU */
    if (first)
    {
        /* Sanity check for the first iteration */
        if ((g_iBlockInDataSize % (g_iNumSubBands * g_nchan * sizeof(char4))) != 0)
        {
            (void) fprintf(stderr, "ERROR: Data size mismatch! BlockInDataSize=%d NumSubBands=%d nchan=%d\n",
                                    g_iBlockInDataSize, g_iNumSubBands, g_nchan);
            run = 0;
            return;
        }
        CUDASafeCall(hipMemcpy(g_pc4Data_d,
                                payload_addr_in,
                                g_iBlockInDataSize,
                                hipMemcpyHostToDevice));
        /* duplicate the last (VEGAS_NUM_TAPS - 1) segments at the end for
           the next iteration */
        CUDASafeCall(hipMemcpy(g_pc4Data_d + (g_iBlockInDataSize / sizeof(char4)),
                                g_pc4Data_d + (g_iBlockInDataSize / sizeof(char4)) - ((VEGAS_NUM_TAPS - 1) * g_iNumSubBands * g_nchan),
                                ((VEGAS_NUM_TAPS - 1) * g_iNumSubBands * g_nchan * sizeof(char4)),
                                hipMemcpyDeviceToDevice));

        /* copy the status bits and valid flags for all heaps to arrays separate
           from the index, so that it can be combined with the corresponding
           values from the previous block */
        struct time_spead_heap* time_heap = (struct time_spead_heap*) vegas_databuf_data(db_in, curblock_in);
        for (i = 0; i < index_in->num_heaps; ++i)
        {
            g_auiStatusBits[i] = time_heap->status_bits;
            g_auiHeapValid[i] = index_in->cpu_gpu_buf[i].heap_valid;
            ++time_heap;
        }
        /* duplicate the last (VEGAS_NUM_TAPS - 1) segments at the end for the
           next iteration */
        for ( ; i < index_in->num_heaps + num_in_heaps_tail; ++i)
        {
            g_auiStatusBits[i] = g_auiStatusBits[i-num_in_heaps_tail];
            g_auiHeapValid[i] = g_auiHeapValid[i-num_in_heaps_tail];
        }
    }
    else
    {
        /* If this is not the first run, need to handle block boundary, while doing the PFB */
        CUDASafeCall(hipMemcpy(g_pc4Data_d,
                                g_pc4Data_d + (g_iBlockInDataSize / sizeof(char4)),
                                ((VEGAS_NUM_TAPS - 1) * g_iNumSubBands * g_nchan * sizeof(char4)),
                                hipMemcpyDeviceToDevice));
        CUDASafeCall(hipMemcpy(g_pc4Data_d + ((VEGAS_NUM_TAPS - 1) * g_iNumSubBands * g_nchan),
                                payload_addr_in,
                                g_iBlockInDataSize,
                                hipMemcpyHostToDevice));
        /* copy the status bits and valid flags for all heaps to arrays separate
           from the index, so that it can be combined with the corresponding
           values from the previous block */
        for (i = 0; i < num_in_heaps_tail; ++i)
        {
            g_auiStatusBits[i] = g_auiStatusBits[index_in->num_heaps+i];
            g_auiHeapValid[i] = g_auiHeapValid[index_in->num_heaps+i];
        }
        struct time_spead_heap* time_heap = (struct time_spead_heap*) vegas_databuf_data(db_in, curblock_in);
        for ( ; i < num_in_heaps_tail + index_in->num_heaps; ++i)
        {
            g_auiStatusBits[i] = time_heap->status_bits;
            g_auiHeapValid[i] = index_in->cpu_gpu_buf[i-num_in_heaps_tail].heap_valid;
            ++time_heap;
        }
    }

    g_pc4DataRead_d = g_pc4Data_d;
    iProcData = 0;
    while (g_iBlockInDataSize > iProcData)  /* loop till (num_heaps * heap_size) of data is processed */
    {
        if (0 == pfb_count)
        {
            /* Check if all heaps necessary for this PFB are valid */
            if (!(is_valid(heap_in, (VEGAS_NUM_TAPS * num_in_heaps_per_proc))))
            {
                /* Skip all heaps that go into this PFB if there is an invalid heap */
                iProcData += (VEGAS_NUM_TAPS * g_iNumSubBands * g_nchan * sizeof(char4));
                /* update the data read pointer */
                g_pc4DataRead_d += (VEGAS_NUM_TAPS * g_iNumSubBands * g_nchan);
                if (iProcData == g_iBlockInDataSize)
                {
                    break;
                }

                /* Calculate input heap addresses for the next round of processing */
                heap_in += (VEGAS_NUM_TAPS * num_in_heaps_per_proc);
                if (heap_in > num_in_heaps_gpu_buffer)
                {
                    /* This is not supposed to happen (but may happen if odd number of pkts are dropped
                       right at the end of the buffer, so we therefore do not exit) */
                    (void) fprintf(stderr,
                                   "WARNING: Heap count %d exceeds available number of heaps %d!\n",
                                   heap_in,
                                   num_in_heaps_gpu_buffer);
                }
                heap_addr_in = (char*)(vegas_databuf_data(db_in, curblock_in) +
                                    sizeof(struct time_spead_heap) * heap_in);
                continue;
            }
        }

        /* Perform polyphase filtering */
        DoPFB<<<g_dimGPFB, g_dimBPFB>>>(g_pc4DataRead_d,
                                        g_pf4FFTIn_d,
                                        g_pfPFBCoeff_d);
        CUDASafeCall(hipDeviceSynchronize());
        iCUDARet = hipGetLastError();
        if (iCUDARet != hipSuccess)
        {
            (void) fprintf(stderr,
                           "ERROR: File <%s>, Line %d: %s\n",
                           __FILE__,
                           __LINE__,
                           hipGetErrorString(iCUDARet));
            run = 0;
            break;
        }
         
        iRet = do_fft();
        if (iRet != VEGAS_OK)
        {
            (void) fprintf(stderr, "ERROR: FFT failed!\n");
            run = 0;
            break;
        }

        /* Accumulate power x, power y, stokes real and imag, if the blanking
           bit is not set */
        if (!(is_blanked(heap_in, num_in_heaps_per_proc)))
        {
            iRet = accumulate();
            if (iRet != VEGAS_OK)
            {
                (void) fprintf(stderr, "ERROR: Accumulation failed!\n");
                run = 0;
                break;
            }
            ++g_iSpecPerAcc;
            g_iPrevBlankingState = FALSE;
        }
        else
        {
            /* state just changed */
            if (FALSE == g_iPrevBlankingState)
            {
                /* dump to buffer */
                heap_addr_out = (char*)(vegas_databuf_data(db_out, g_iPFBCurBlockOut) +
                                    sizeof(struct freq_spead_heap) * g_iHeapOut);
                freq_heap_out = (struct freq_spead_heap*)(heap_addr_out);
                payload_addr_out = (char*)(vegas_databuf_data(db_out, g_iPFBCurBlockOut) +
                                    sizeof(struct freq_spead_heap) * MAX_HEAPS_PER_BLK +
                                    (index_out->heap_size - sizeof(struct freq_spead_heap)) * g_iHeapOut);
         
                /* Write new heap header fields */
                freq_heap_out->time_cntr_id = 0x20;
                freq_heap_out->time_cntr_top8 = first_time_heap_in_accum->time_cntr_top8;
                freq_heap_out->time_cntr = first_time_heap_in_accum->time_cntr;
                freq_heap_out->spectrum_cntr_id = 0x21;
                freq_heap_out->spectrum_cntr = g_iTotHeapOut;
                freq_heap_out->integ_size_id = 0x22;
                freq_heap_out->integ_size = g_iSpecPerAcc;
                freq_heap_out->mode_id = 0x23;
                freq_heap_out->mode = first_time_heap_in_accum->mode;
                freq_heap_out->status_bits_id = 0x24;
                freq_heap_out->status_bits = first_time_heap_in_accum->status_bits;
                freq_heap_out->payload_data_off_addr_mode = 0;
                freq_heap_out->payload_data_off_id = 0x25;
                freq_heap_out->payload_data_off = 0;

                /* Update output index */
                index_out->cpu_gpu_buf[g_iHeapOut].heap_valid = 1;
                index_out->cpu_gpu_buf[g_iHeapOut].heap_cntr = g_iTotHeapOut;
                index_out->cpu_gpu_buf[g_iHeapOut].heap_rcvd_mjd =
                         index_in->cpu_gpu_buf[g_iFirstHeapIn].heap_rcvd_mjd ;

                iRet = get_accumulated_spectrum_from_device(payload_addr_out);
                if (iRet != VEGAS_OK)
                {
                    (void) fprintf(stderr, "ERROR: Getting accumulated spectrum failed!\n");
                    run = 0;
                    break;
                }

                ++g_iHeapOut;
                ++g_iTotHeapOut;

                /* zero accumulators */
                zero_accumulator();
                /* reset time */
                g_iSpecPerAcc = 0;
                g_iPrevBlankingState = TRUE;
            }
        }

        if (g_iSpecPerAcc == acc_len)
        {
            /* dump to buffer */
            heap_addr_out = (char*)(vegas_databuf_data(db_out, g_iPFBCurBlockOut) +
                                sizeof(struct freq_spead_heap) * g_iHeapOut);
            freq_heap_out = (struct freq_spead_heap*)(heap_addr_out);
            payload_addr_out = (char*)(vegas_databuf_data(db_out, g_iPFBCurBlockOut) +
                                sizeof(struct freq_spead_heap) * MAX_HEAPS_PER_BLK +
                                (index_out->heap_size - sizeof(struct freq_spead_heap)) * g_iHeapOut);
     
            /* Write new heap header fields */
            freq_heap_out->time_cntr_id = 0x20;
            freq_heap_out->time_cntr_top8 = first_time_heap_in_accum->time_cntr_top8;
            freq_heap_out->time_cntr = first_time_heap_in_accum->time_cntr;
            freq_heap_out->spectrum_cntr_id = 0x21;
            freq_heap_out->spectrum_cntr = g_iTotHeapOut;
            freq_heap_out->integ_size_id = 0x22;
            freq_heap_out->integ_size = g_iSpecPerAcc;
            freq_heap_out->mode_id = 0x23;
            freq_heap_out->mode = first_time_heap_in_accum->mode;
            freq_heap_out->status_bits_id = 0x24;
            freq_heap_out->status_bits = first_time_heap_in_accum->status_bits;
            freq_heap_out->payload_data_off_addr_mode = 0;
            freq_heap_out->payload_data_off_id = 0x25;
            freq_heap_out->payload_data_off = 0;

            /* Update output index */
            index_out->cpu_gpu_buf[g_iHeapOut].heap_valid = 1;
            index_out->cpu_gpu_buf[g_iHeapOut].heap_cntr = g_iTotHeapOut;
            index_out->cpu_gpu_buf[g_iHeapOut].heap_rcvd_mjd =
                     index_in->cpu_gpu_buf[g_iFirstHeapIn].heap_rcvd_mjd ;

            iRet = get_accumulated_spectrum_from_device(payload_addr_out);
            if (iRet != VEGAS_OK)
            {
                (void) fprintf(stderr, "ERROR: Getting accumulated spectrum failed!\n");
                run = 0;
                break;
            }

            ++g_iHeapOut;
            ++g_iTotHeapOut;

            /* zero accumulators */
            zero_accumulator();
            /* reset time */
            g_iSpecPerAcc = 0;
        }

        iProcData += (g_iNumSubBands * g_nchan * sizeof(char4));
        /* update the data read pointer */
        g_pc4DataRead_d += (g_iNumSubBands * g_nchan);

        /* Calculate input heap addresses for the next round of processing */
        heap_in += num_in_heaps_per_proc;
        heap_addr_in = (char*)(vegas_databuf_data(db_in, curblock_in) +
                            sizeof(struct time_spead_heap) * heap_in);
        if (0 == g_iSpecPerAcc)
        {
            first_time_heap_in_accum = (struct time_spead_heap*)(heap_addr_in);
            g_iFirstHeapIn = heap_in;
            g_dFirstHeapRcvdMJD = index_in->cpu_gpu_buf[g_iFirstHeapIn].heap_rcvd_mjd;
        }

        /* if output block is full */
        if (g_iHeapOut == g_iMaxNumHeapOut)
        {
            /* Set the number of heaps written to this block */
            index_out->num_heaps = g_iHeapOut;

            /* Mark output buffer as filled */
            vegas_databuf_set_filled(db_out, g_iPFBCurBlockOut);

            printf("Debug: vegas_pfb_thread going to next output block\n");

            /* Note current output block */
            /* NOTE: vegas_status_lock_safe() and vegas_status_unlock_safe() are macros
               that have been explicitly expanded here, due to compilation issues */
            //vegas_status_lock_safe(&st);
                pthread_cleanup_push((void (*) (void *))&vegas_status_unlock, (void *) &st);
                vegas_status_lock(&st);
            hputi4(st.buf, "PFBBLKOU", g_iPFBCurBlockOut);
            //vegas_status_unlock_safe(&st);
                vegas_status_unlock(&st);
                pthread_cleanup_pop(0);

            /*  Wait for next output block */
            g_iPFBCurBlockOut = (g_iPFBCurBlockOut + 1) % db_out->n_block;
            while ((vegas_databuf_wait_free(db_out, g_iPFBCurBlockOut)!=0) && run) {
                //vegas_status_lock_safe(&st);
                    pthread_cleanup_push((void (*)(void *))&vegas_status_unlock, (void *) &st);
                    vegas_status_lock(&st);

                hputs(st.buf, STATUS_KEY, "blocked");
                //vegas_status_unlock_safe(&st);
                    vegas_status_unlock(&st);
                    pthread_cleanup_pop(0);
            }

            g_iHeapOut = 0;

            hdr_out = vegas_databuf_header(db_out, g_iPFBCurBlockOut);
            index_out = (struct databuf_index*)vegas_databuf_index(db_out, g_iPFBCurBlockOut);
            memcpy(hdr_out, vegas_databuf_header(db_in, curblock_in),
                    VEGAS_STATUS_SIZE);

            /* Set basic params in output index */
            index_out->heap_size = sizeof(struct freq_spead_heap) + (g_iNumSubBands * g_nchan * sizeof(float4));
        }

        pfb_count = (pfb_count + 1) % VEGAS_NUM_TAPS;
    }

    return;
}

/* function that performs the FFT */
int do_fft()
{
    hipfftResult iCUFFTRet = HIPFFT_SUCCESS;

    /* execute plan */
    iCUFFTRet = hipfftExecC2C(g_stPlan,
                             (hipfftComplex*) g_pf4FFTIn_d,
                             (hipfftComplex*) g_pf4FFTOut_d,
                             HIPFFT_FORWARD);
    if (iCUFFTRet != HIPFFT_SUCCESS)
    {
        (void) fprintf(stderr, "ERROR: FFT failed!");
        run = 0;
        return VEGAS_ERR_GEN;
    }

    return VEGAS_OK;
}

int accumulate()
{
    hipError_t iCUDARet = hipSuccess;

    Accumulate<<<g_dimGAccum, g_dimBAccum>>>(g_pf4FFTOut_d,
                                             g_pf4SumStokes_d);
    CUDASafeCall(hipDeviceSynchronize());
    iCUDARet = hipGetLastError();
    if (iCUDARet != hipSuccess)
    {
        (void) fprintf(stderr, hipGetErrorString(iCUDARet));
        run = 0;
        return VEGAS_ERR_GEN;
    }

    return VEGAS_OK;
}

void zero_accumulator()
{
    CUDASafeCall(hipMemset(g_pf4SumStokes_d,
                                       '\0',
                                       (g_iNumSubBands
                                       * g_nchan
                                       * sizeof(float4))));

    return;
}

int get_accumulated_spectrum_from_device(char *out)
{
    /* copy the negative frequencies out first */
    CUDASafeCall(hipMemcpy(out,
                            g_pf4SumStokes_d + (g_iNumSubBands * g_nchan / 2),
                            (g_iNumSubBands
                             * (g_nchan / 2)
                             * sizeof(float4)),
                            hipMemcpyDeviceToHost));
    /* copy the positive frequencies out */
    CUDASafeCall(hipMemcpy(out + (g_iNumSubBands * (g_nchan / 2) * sizeof(float4)),
                            g_pf4SumStokes_d,
                            (g_iNumSubBands
                             * (g_nchan / 2)
                             * sizeof(float4)),
                            hipMemcpyDeviceToHost));

    return VEGAS_OK;
}

/*
 * function to be used to check if any heap within the current PFB is invalid,
 * in which case, the entire PFB should be discarded.
 * NOTE: this function does not check ALL heaps - it returns at the first
 * invalid heap.
 */
int is_valid(int heap_start, int num_heaps)
{
    for (int i = heap_start; i < (heap_start + num_heaps); ++i)
    {
        if (!g_auiHeapValid[i])
        {
            return FALSE;
        }
    }

    return TRUE;
}

/*
 * function that checks if blanking has started within this accumulation.
 * ASSUMPTION: the blanking bit does not toggle within this time interval.
 */
int is_blanked(int heap_start, int num_heaps)
{
    for (int i = heap_start; i < (heap_start + num_heaps); ++i)
    {
        if (g_auiStatusBits[i] & 0x08)
        {
            return TRUE;
        }
    }

    return FALSE;
}

void __CUDASafeCall(hipError_t iCUDARet,
                               const char* pcFile,
                               const int iLine,
                               void (*pCleanUp)(void))
{
    if (iCUDARet != hipSuccess)
    {
        (void) fprintf(stderr,
                       "ERROR: File <%s>, Line %d: %s\n",
                       pcFile,
                       iLine,
                       hipGetErrorString(iCUDARet));
        run = 0;
        return;
    }

    return;
}

/* 
 * Frees up any allocated memory.
 */
void cleanup_gpu()
{
    /* free memory */
    if (g_pc4InBuf != NULL)
    {
        free(g_pc4InBuf);
        g_pc4InBuf = NULL;
    }
    if (g_pc4Data_d != NULL)
    {
        (void) hipFree(g_pc4Data_d);
        g_pc4Data_d = NULL;
    }
    if (g_pf4FFTIn_d != NULL)
    {
        (void) hipFree(g_pf4FFTIn_d);
        g_pf4FFTIn_d = NULL;
    }
    if (g_pf4FFTOut_d != NULL)
    {
        (void) hipFree(g_pf4FFTOut_d);
        g_pf4FFTOut_d = NULL;
    }
    if (g_pf4SumStokes_d != NULL)
    {
        (void) hipFree(g_pf4SumStokes_d);
        g_pf4SumStokes_d = NULL;
    }

    /* destroy plan */
    /* TODO: check if plan exists */
    (void) hipfftDestroy(g_stPlan);

    return;
}

