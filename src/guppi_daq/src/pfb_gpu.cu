#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "guppi_databuf.h"
#include "pfb_gpu_kernels.h"

/**
 * Global variables: maybe move this to a struct that is passed to each function?
 */
size_t buf_block_size;
size_t buf_index_size;

char *device_in_buf;
char *device_out_buf;


/* Initialize all necessary memory, etc for doing PFB 
 * at the given params.
 */
extern "C"
void init_pfb(size_t block_size, size_t index_size)
{
	buf_block_size = block_size;
	buf_index_size = index_size;

    /* Allocate GPU memory */
    hipMalloc((void**)&device_in_buf, buf_block_size);
    hipMalloc((void**)&device_out_buf, buf_block_size);
}


/* Actually do the PFB by calling CUDA kernels */
extern "C"
void do_pfb(const char *in, char *out, struct databuf_index* index_in,
                struct databuf_index* index_out)
{
    /* Copy data to GPU */
    hipMemcpy(device_in_buf, in, buf_block_size, hipMemcpyHostToDevice);

    /* Dummy processing */
    int numBlocks = 32;
    int threadsPerBlock = 64;
    mult_by_1<<<numBlocks, threadsPerBlock>>>(device_in_buf, device_out_buf, buf_block_size);
     
    /* Transfer data back to host*/
    hipMemcpy(out, device_out_buf, buf_block_size, hipMemcpyDeviceToHost);

    /* Copy input index to output index: for initial testing only */
	memcpy(index_out, index_in, buf_index_size);
}
